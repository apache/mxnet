#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2017 by Contributors
 * \file random_generator.cu
 * \brief gpu implements for parallel random number generator.
 */

#include <mxnet/random_generator.h>
#include <algorithm>
#include "../operator/mxnet_op.h"

namespace mxnet {
namespace common {
namespace random {

template<>
const int RandGenerator<gpu, float>::kMinNumRandomPerThread = 64;

template<>
const int RandGenerator<gpu, float>::kNumRandomStates = 32768;

__global__ void rand_generator_seed_kernel(hiprandStatePhilox4_32_10_t *states_,
                                           const int size,
                                           uint32_t seed) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < size) hiprand_init(seed, id, 0, states_ + id);
}

template<>
void RandGenerator<gpu, float>::Seed(mshadow::Stream<gpu> *s, uint32_t seed) {
  using namespace mshadow::cuda;
  int ngrid = std::min(kMaxGridNum,
                       (RandGenerator<gpu, float>::kNumRandomStates + kBaseThreadNum - 1) /
                         kBaseThreadNum);
  rand_generator_seed_kernel
      <<<ngrid, kBaseThreadNum, 0, mshadow::Stream<gpu>::GetStream(s)>>>(
          states_,
          RandGenerator<gpu, float>::kNumRandomStates,
          seed);
  MSHADOW_CUDA_POST_KERNEL_CHECK(rand_generator_seed_kernel);
  s->Wait();
}

template<>
void RandGenerator<gpu, float>::AllocState(RandGenerator<gpu> *inst) {
  CUDA_CALL(hipMalloc(&inst->states_,
                       kNumRandomStates * sizeof(hiprandStatePhilox4_32_10_t)));
}

template<>
void RandGenerator<gpu, float>::FreeState(RandGenerator<gpu> *inst) {
  CUDA_CALL(hipFree(inst->states_));
}

template<>
void* RandGenerator<gpu, float>::GetStates() {
  return static_cast<void*>(states_);
}

}  // namespace random
}  // namespace common
}  // namespace mxnet
