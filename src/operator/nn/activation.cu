#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2015 by Contributors
 * \file activation.cu
 * \brief
 * \author Bing Xu
*/
#include "./activation-inl.h"
#include "../mshadow_op.h"
#if MXNET_USE_CUDNN == 1
#include "./cudnn/cudnn_activation-inl.h"
#endif

namespace mxnet {
namespace op {

#if MXNET_USE_CUDNN == 1

template<typename DType>
static CuDNNActivationOp<DType> &get_cudnn_op(const ActivationParam& param) {
#if DMLC_CXX11_THREAD_LOCAL
  static thread_local CuDNNActivationOp<DType> cudnn_op;
#else
  static MX_THREAD_LOCAL CuDNNActivationOp<DType> cudnn_op;
#endif
  cudnn_op.Init(param);
  return cudnn_op;
}

template<>
void ActivationCompute<gpu>(const nnvm::NodeAttrs& attrs,
    const OpContext& ctx,
    const std::vector<TBlob>& inputs,
    const std::vector<OpReqType>& req,
    const std::vector<TBlob>& outputs) {
  CHECK_EQ(inputs.size(), 1U);
  CHECK_EQ(outputs.size(), 1U);
  const ActivationParam& param = nnvm::get<ActivationParam>(attrs.parsed);
  const int act_type = param.act_type;

  // SoftReLU, SoftSign, Log_Sigmoid and Mish are not supported by CUDNN yet
  if (act_type == activation::kSoftReLU) {
    ActivationForward<gpu, mshadow_op::softrelu, mshadow_op::softrelu_grad>(ctx,
      inputs[0], req[0], outputs[0]);
  } else if (act_type == activation::kSoftSign) {
    ActivationForward<gpu, mshadow_op::softsign, mshadow_op::softsign_grad>(ctx,
      inputs[0], req[0], outputs[0]);
  } else if (act_type == activation::kLogSigmoid) {
    ActivationForward<gpu, mshadow_op::log_sigmoid, mshadow_op::log_sigmoid_grad>(ctx,
      inputs[0], req[0], outputs[0]);
  } else if (act_type == activation::kMish) {
    ActivationForward<gpu, mshadow_op::mish, mshadow_op::mish_grad>(ctx,
      inputs[0], req[0], outputs[0]);
  } else {
    MSHADOW_REAL_TYPE_SWITCH(inputs[0].type_flag_, DType, {
      get_cudnn_op<DType>(param).Forward(ctx, inputs[0], req[0], outputs[0]);
    });
  }
}

template<>
void ActivationGradCompute<gpu>(const nnvm::NodeAttrs& attrs,
                                const OpContext& ctx,
                                const std::vector<TBlob>& inputs,
                                const std::vector<OpReqType>& req,
                                const std::vector<TBlob>& outputs) {
  const ActivationParam& param = nnvm::get<ActivationParam>(attrs.parsed);
  const int act_type = param.act_type;
  CHECK_EQ(inputs.size(), activation::GradNumInputs(act_type));
  CHECK_EQ(outputs.size(), 1U);
  CHECK_EQ(req.size(), 1U);

  bool do_memory_opt = dmlc::GetEnv("MXNET_MEMORY_OPT", 0);

  // SoftReLU, SoftSign, Log_Sigmoid and Mish not supported by CUDNN yet
  if (act_type == activation::kSoftReLU) {
    ActivationBackward<gpu, mshadow_op::softrelu, mshadow_op::softrelu_grad>(
      ctx, inputs.at(0), inputs.at(1), req[0], outputs[0]);
  } else if (act_type == activation::kLogSigmoid) {
    ActivationBackward<gpu, mshadow_op::log_sigmoid, mshadow_op::log_sigmoid_grad>(
      ctx, inputs.at(0), inputs.at(1), req[0], outputs[0]);
  } else if (act_type == activation::kMish) {
    ActivationBackward<gpu, mshadow_op::mish, mshadow_op::mish_grad>(
      ctx, inputs.at(0), inputs.at(2), req[0], outputs[0]);
  } else if (act_type == activation::kSoftSign) {
    if (do_memory_opt) {
      ActivationBackward<gpu, mshadow_op::softsign, mshadow_op::softsign_grad>(
        ctx, inputs.at(0), inputs.at(1), req[0], outputs[0]);
    } else {
      ActivationBackward<gpu, mshadow_op::softsign, mshadow_op::softsign_grad>(
        ctx, inputs.at(0), inputs.at(2), req[0], outputs[0]);
    }
  } else if (act_type == activation::kReLU) {
    if (do_memory_opt) {
      ActivationBackward<gpu, mshadow_op::relu, mshadow_op::relu_grad>(
        ctx, inputs.at(0), inputs.at(1), req[0], outputs[0]);
    } else {
      MSHADOW_REAL_TYPE_SWITCH(inputs.at(0).type_flag_, DType, {
        // XXX: for y = relu(x), y is passed as "in_data" to Backward()
        get_cudnn_op<DType>(param).Backward(ctx, inputs.at(0), inputs.at(1),
                                            inputs.at(1), req[0], outputs[0]);
      });
    }
  } else {
    if (do_memory_opt) {
      if (act_type == activation::kTanh) {
        ActivationBackward<gpu, mshadow_op::tanh, mshadow_op::tanh_grad>(
          ctx, inputs.at(0), inputs.at(1), req[0], outputs[0]);
      } else if (act_type == activation::kSigmoid) {
        ActivationBackward<gpu, mshadow_op::sigmoid, mshadow_op::sigmoid_grad>(
          ctx, inputs.at(0), inputs.at(1), req[0], outputs[0]);
      } else {
        LOG(FATAL) << "unknown activation type";
      }
    } else {
      MSHADOW_REAL_TYPE_SWITCH(inputs.at(0).type_flag_, DType, {
        get_cudnn_op<DType>(param).Backward(ctx, inputs.at(0), inputs.at(2),
                                            inputs.at(1), req[0], outputs[0]);
      });
    }  // if (do_memory_opt)
  }
}
#endif

NNVM_REGISTER_OP(Activation)
.set_attr<FCompute>("FCompute<gpu>", ActivationCompute<gpu>);

NNVM_REGISTER_OP(_backward_Activation)
.set_attr<FCompute>("FCompute<gpu>", ActivationGradCompute<gpu>);

}  // namespace op
}  // namespace mxnet
