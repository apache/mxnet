/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2017 by Contributors
 * \file pooling.cu
 * \brief
 * \author Bing Xu, Jun Wu, Da Zheng
*/
#include <vector>
#include "./pooling-inl.h"
#if MXNET_USE_CUDNN == 1
#include "./cudnn/cudnn_pooling-inl.h"
#endif  // MXNET_USE_CUDNN

namespace mxnet {
namespace op {

#if MXNET_USE_CUDNN == 1
template<typename DType>
static CuDNNPoolingOp<DType> &GetCuDNNPoolingOp(const PoolingParam &param) {
#if DMLC_CXX11_THREAD_LOCAL
  static thread_local CuDNNPoolingOp<DType> op;
#else
  static MX_THREAD_LOCAL CuDNNPoolingOp<DType> op;
#endif
  op.Init(param);
  return op;
}
#endif

template<>
void PoolingCompute<gpu>(const nnvm::NodeAttrs& attrs,
                         const OpContext& ctx,
                         const std::vector<TBlob>& inputs,
                         const std::vector<OpReqType>& req,
                         const std::vector<TBlob>& outputs) {
  const PoolingParam& param = nnvm::get<PoolingParam>(attrs.parsed);
  CHECK_EQ(inputs.size(), 1U);
  CHECK_EQ(outputs.size(), GetNumOutputs(param));

#if MXNET_USE_CUDNN == 1
  if (!param.cudnn_off) {
    MSHADOW_REAL_TYPE_SWITCH(inputs[0].type_flag_, DType, {
      if (CuDNNPoolingOp<DType>::Supports(param, inputs[0])) {
        GetCuDNNPoolingOp<DType>(param).Forward(ctx, inputs[0], req[0], outputs[0]);
        return;
      }
    });
  }
#endif  // MXNET_USE_CUDNN

  MSHADOW_REAL_TYPE_SWITCH(inputs[0].type_flag_, DType, {
    if (pool_enum::kMaxPooling == param.pool_type
        || pool_enum::kAvgPooling == param.pool_type
        || pool_enum::kSumPooling == param.pool_type
        || pool_enum::kLpPooling == param.pool_type) {
      PoolingOp<gpu, DType> op;
      op.Init(param);
      op.Forward(ctx, inputs[0], req[0], outputs[0]);
    } else {
      LOG(FATAL) << "unknown pooling type";
    }
  });
}

template<>
void PoolingGradCompute<gpu>(const nnvm::NodeAttrs& attrs,
                             const OpContext& ctx,
                             const std::vector<TBlob>& inputs,
                             const std::vector<OpReqType>& req,
                             const std::vector<TBlob>& outputs) {
  const PoolingParam& param = nnvm::get<PoolingParam>(attrs.parsed);
  CHECK_EQ(inputs.size(), GetNumBackInputs(param));
  CHECK_EQ(outputs.size(), 1U);
  CHECK_EQ(req.size(), 1U);
  off_t ograd_idx, in_data_idx, out_data_idx;
  // When MKLDNN is enabled, the input data may contains arrays for workspace.
  if (GetNumBackInputs(param) == 5) {
    ograd_idx = 0;
    in_data_idx = 2;
    out_data_idx = 3;
  } else {
    ograd_idx = 0;
    in_data_idx = 1;
    out_data_idx = 2;
  }

#if MXNET_USE_CUDNN == 1
  if (!param.cudnn_off) {
    MSHADOW_REAL_TYPE_SWITCH(inputs[0].type_flag_, DType, {
      if (CuDNNPoolingOp<DType>::Supports(param, inputs[in_data_idx])) {
          GetCuDNNPoolingOp<DType>(param).Backward(ctx, inputs[ograd_idx],
                                                   inputs[in_data_idx], inputs[out_data_idx],
                                                   req[0], outputs[0]);
          return;
      }
    });
  }
#endif  // MXNET_USE_CUDNN

  MSHADOW_REAL_TYPE_SWITCH(inputs[0].type_flag_, DType, {
    if (pool_enum::kMaxPooling == param.pool_type
        || pool_enum::kAvgPooling == param.pool_type
        || pool_enum::kSumPooling == param.pool_type
        || pool_enum::kLpPooling == param.pool_type) {
      PoolingOp<gpu, DType> op;
      op.Init(param);
      op.Backward(ctx, inputs[ograd_idx], inputs[in_data_idx],
                  inputs[out_data_idx], req[0], outputs[0]);
    } else {
      LOG(FATAL) << "unknown pooling type";
    }
  });
}

NNVM_REGISTER_OP(Pooling)
.set_attr<FCompute>("FCompute<gpu>", PoolingCompute<gpu>);

NNVM_REGISTER_OP(_backward_Pooling)
.set_attr<FCompute>("FCompute<gpu>", PoolingGradCompute<gpu>);

}  // namespace op
}  // namespace mxnet
