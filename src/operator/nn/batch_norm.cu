#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2017 by Contributors
 * \file batch_norm.cu
 * \brief CUDA Batch Normalization code
 * \author Chris Olivier, Bing Xu, Da Zheng
 * Adapted from Torch
*/
#include <hip/hip_runtime_api.h>
#include <algorithm>
#include "batch_norm-inl.h"

#define WRITE_DATA_FLAG       1
#define WRITE_GAMMA_FLAG      2
#define WRITE_BETA_FLAG       4
#define FIX_GAMMA_FLAG        8
#define IS_TRAINING_FLAG      16
#define USE_GLOBAL_STATS_FLAG 32
#define ADDTO_DATA_FLAG       (1 << 6)
#define ADDTO_GAMMA_FLAG      (1 << 7)
#define ADDTO_BETA_FLAG       (1 << 8)

#if MXNET_USE_CUDNN == 1
#include "./cudnn/cudnn_batch_norm-inl.h"
#endif

#include "../../common/cuda_utils.h"
#include "../../../include/mxnet/tensor_blob.h"

using namespace mxnet;

namespace mxnet {
namespace op {
namespace batchnorm {
namespace cuda {

static const unsigned WARP_SIZE = 32;

// The maximum number of threads in a block
static const unsigned MAX_BLOCK_SIZE = 512U;

template<typename In, typename Out>
struct ScalarConvert {
  static __host__ __device__ __forceinline__ Out to(const In v) { return (Out) v; }
};

// Number of threads in a block given an input size up to MAX_BLOCK_SIZE
static unsigned getNumThreads(int nElem, const bool smaller) {
  unsigned threadSizes[5] = {32, 64, 128, 256, MAX_BLOCK_SIZE};
  const int maxi = smaller ? 4 : 5;
  for (int i = 0; i != maxi; ++i) {
    if (static_cast<unsigned>(nElem) <= threadSizes[i]) {
      return threadSizes[i];
    }
  }
  return smaller ? (MAX_BLOCK_SIZE >> 1) : MAX_BLOCK_SIZE;
}

// Returns the index of the most significant 1 bit in `val`.
__device__ __forceinline__ int getMSB(int val) {
  return 31 - __clz(val);
}

template<typename DType, typename AccReal>
struct Float2 {
  AccReal v1, v2;
  __device__ Float2() {}
  __device__ Float2(DType v1, DType v2)
    : v1(ScalarConvert<DType, AccReal>::to(v1))
      , v2(ScalarConvert<DType, AccReal>::to(v2)) {}
  __device__ Float2(DType v)
    : v1(ScalarConvert<DType, AccReal>::to(v))
      , v2(ScalarConvert<DType, AccReal>::to(v)) {}
  __device__ Float2(int v)
    : v1(ScalarConvert<int, AccReal>::to(v))
      , v2(ScalarConvert<int, AccReal>::to(v)) {}
  __device__ Float2 &operator+=(const Float2 &a) {
    v1 += a.v1;
    v2 += a.v2;
    return *this;
  }
};

template<typename DType, typename AccReal, typename DeviceTensor>
struct SumOp {
  __device__ SumOp(const DeviceTensor t) : tensor(t) {}
  __device__ __forceinline__ AccReal operator()(int batch, int plane, int n) {
    return ScalarConvert<DType, AccReal>::to(tensor.get_ref(batch, plane, n));
  }
  const DeviceTensor tensor;
};

template<typename DType, typename AccReal, typename DeviceTensor>
struct VarOp {
  __device__ VarOp(AccReal m, const DeviceTensor t)
    : mean(m)
      , tensor(t) {
  }
  __device__ __forceinline__ AccReal operator()(int batch, int plane, int n) {
    DType val = tensor.get_ref(batch, plane, n);
    return (val - mean) * (val - mean);
  }
  const AccReal mean;
  const DeviceTensor tensor;
};

template<typename DType, typename AccReal, typename DeviceTensor>
struct GradOp {
  __device__ GradOp(AccReal m, const DeviceTensor i, const DeviceTensor g)
    : mean(m), input(i), gradOutput(g) {}
  __device__ __forceinline__ Float2<DType, AccReal> operator()(int batch, int plane, int n) {
    const DType g = gradOutput.get_ref(batch, plane, n);
    const DType c = ScalarConvert<AccReal, DType>::to(input.get_ref(batch, plane, n) - mean);
    return Float2<DType, AccReal>(g, g * c);
  }
  const AccReal mean;
  const DeviceTensor input;
  const DeviceTensor gradOutput;
};

#if CUDA_VERSION >= 9000
#define FULLMASK 0xFFFFFFFF
#define __shfl_xor(...) __shfl_xor_sync(FULLMASK, __VA_ARGS__)
#endif

// Sum across all threads within a warp
template<typename T>
static __device__ __forceinline__ T warpSum(T val) {
#if __CUDA_ARCH__ >= 300
for (int i = 0; i < getMSB(WARP_SIZE); ++i) {
    val += __shfl_xor(val, 1 << i, WARP_SIZE);
  }
#else
__shared__ T values[MAX_BLOCK_SIZE];
values[threadIdx.x] = val;
__threadfence_block();
const int base = (threadIdx.x / WARP_SIZE) * WARP_SIZE;
for (int i = 1; i < WARP_SIZE; i++) {
val += values[base + ((i + threadIdx.x) % WARP_SIZE)];
}
#endif
return val;
}

template<typename DType, typename AccReal>
static __device__ __forceinline__ Float2<DType, AccReal> warpSum(Float2<DType, AccReal> value) {
  value.v1 = warpSum(value.v1);
  value.v2 = warpSum(value.v2);
  return value;
}

// Sum across (batch, x/y/z) applying Op() pointwise
template<typename T, typename Op, typename DeviceTensor>
static __device__ T reduce(Op op, DeviceTensor tensor, int plane) {
  T sum = (T) 0;
  for (int batch = 0; batch < tensor.OuterSize(); ++batch) {
    for (int x = threadIdx.x; x < tensor.InnerSize(); x += blockDim.x) {
      sum += op(batch, plane, x);
    }
  }

  // sum over NumThreads within a warp
  sum = warpSum(sum);

  // 'transpose', and reduce within warp again
  __shared__ T shared[32];
  __syncthreads();
  if (threadIdx.x % WARP_SIZE == 0) {
    shared[threadIdx.x / WARP_SIZE] = sum;
  }
  if (threadIdx.x >= blockDim.x / WARP_SIZE && threadIdx.x < WARP_SIZE) {
    // zero out the other entries in shared
    shared[threadIdx.x] = (T) 0;
  }
  __syncthreads();
  if (threadIdx.x / WARP_SIZE == 0) {
    sum = warpSum(shared[threadIdx.x]);
    if (threadIdx.x == 0) {
      shared[0] = sum;
    }
  }
  __syncthreads();

  // Everyone picks it up, should be broadcast into the whole gradInput
  return shared[0];
}

template <typename DType, typename AccReal, typename DeviceTensor1, typename DeviceTensor>
__global__ void BatchNormalizationUpdateOutputInferenceKernel(
  DeviceTensor input,
  DeviceTensor output,
  DeviceTensor1 runningMean,
  DeviceTensor1 runningVar,
  DeviceTensor1 saveMean,
  DeviceTensor1 saveInvStd,
  DeviceTensor1 weight,
  DeviceTensor1 bias,
  const DType epsilon,
  const uint32_t flags) {
  int plane = blockIdx.x;

  AccReal invstd = VARIANCE_TO_INVSTD(runningVar[plane], epsilon);
  AccReal mean = ScalarConvert<DType, AccReal>::to(runningMean[plane]);
  AccReal gamma = ((flags & FIX_GAMMA_FLAG) == 0 && weight.numElements() > 0)
                  ? ScalarConvert<DType, AccReal>::to(weight[plane])
                  : ScalarConvert<int, AccReal>::to(1);
  AccReal beta = bias.numElements() > 0 ? ScalarConvert<DType, AccReal>::to(bias[plane])
                                        : ScalarConvert<int, AccReal>::to(0);
  if (threadIdx.x == 0) {
    saveMean[plane] = runningMean[plane];
    saveInvStd[plane] = VARIANCE_TO_INVSTD(runningVar[plane], epsilon);
    if ((flags & WRITE_GAMMA_FLAG) != 0 && (flags & FIX_GAMMA_FLAG) != 0
        && weight.numElements() > 0) {
      weight[plane] = AccReal(1);
    }
  }
  // Write normalized and update the output
  for (int batch = 0, nbatch = input.OuterSize(); batch < nbatch; ++batch) {
    for (int x = threadIdx.x, nx = input.InnerSize(); x < nx; x += blockDim.x) {
      const DType inp = input.get_ref(batch, plane, x);
      output.get_ref(batch, plane, x) =
        ScalarConvert<AccReal, DType>::to(gamma * (inp - mean) * invstd + beta);
    }
  }
}

template<typename DType, typename AccReal, typename DeviceTensor1, typename DeviceTensor>
__global__ void BatchNormalizationUpdateOutputKernel(
  DeviceTensor input,
  DeviceTensor output,
  DeviceTensor1 weight,
  DeviceTensor1 bias,
  const AccReal epsilon,
  const AccReal momentum,
  DeviceTensor1 runningMean,
  DeviceTensor1 runningVar,
  DeviceTensor1 saveMean,
  DeviceTensor1 saveInvStd,
  const uint32_t flags) {
  const int plane = blockIdx.x;
  const int N = input.OuterSize() * input.InnerSize();

  const AccReal norm = AccReal(1) / N;

  // Compute the mean and variance across (batch, x/y/z)
  const AccReal mean = reduce<AccReal>(
    SumOp<DType, AccReal, DeviceTensor>(input), input, plane) * norm;
  __syncthreads();
  const AccReal varN = reduce<AccReal>(VarOp<DType, AccReal, DeviceTensor>(mean, input),
                                       input, plane);
  AccReal invStd = 0;
  if (varN != AccReal(0) || epsilon != AccReal(0)) {
    invStd = AccReal(1.0) / sqrt(varN * norm + epsilon);
  }

  // Save the mean, variance, and moving averages
  if (threadIdx.x == 0) {
    // For one item (0th) per plane (channel), write the per-channel data (ie mean, variance, etc)
    // Momentum based writeback
    saveMean[plane] = ScalarConvert<AccReal, DType>::to(mean);
    saveInvStd[plane] = invStd;
    if ((flags & WRITE_GAMMA_FLAG) != 0 && (flags & FIX_GAMMA_FLAG) != 0
        && weight.numElements() > 0) {
      weight[plane] = AccReal(1);
    }
  }

  // Write normalized and update the output
  const AccReal gamma = ((flags & FIX_GAMMA_FLAG) == 0 && weight.numElements() > 0)
                        ? ScalarConvert<DType, AccReal>::to(weight[plane])
                        : ScalarConvert<int, AccReal>::to(1);
  const AccReal beta = bias.numElements() > 0 ? ScalarConvert<DType, AccReal>::to(bias[plane])
                                              : ScalarConvert<int, AccReal>::to(0);
  for (int batch = 0, nbatch = input.OuterSize(); batch < nbatch; ++batch) {
    for (int x = threadIdx.x, nx = input.InnerSize(); x < nx; x += blockDim.x) {
      const DType inp = input.get_ref(batch, plane, x);
      output.get_ref(batch, plane, x) =
        ScalarConvert<AccReal, DType>::to(gamma * (inp - mean) * invStd + beta);
    }
  }
}

template<typename DeviceTensor1>
struct CUDATensors {
  DeviceTensor1 gradWeight;
  DeviceTensor1 gradBias;
  DeviceTensor1 weight;
  DeviceTensor1 runningMean;
  DeviceTensor1 runningVar;
  DeviceTensor1 saveMean;
  DeviceTensor1 saveInvStd;
};

template<typename DType, typename AccReal, typename DeviceTensor1, typename DeviceTensor>
static __global__ void BatchNormalizationBackwardKernel(
  const DeviceTensor input,
  const DeviceTensor gradOutput,
  DeviceTensor gradInput,
  CUDATensors<DeviceTensor1> tensors,
  const uint32_t flags,
  const AccReal momentum,
  const double eps) {
  int plane = blockIdx.x;
  int N = gradOutput.OuterSize() * gradOutput.InnerSize();

  const bool is_train_and_not_global_stats =
    (flags & IS_TRAINING_FLAG) != 0 && (flags & USE_GLOBAL_STATS_FLAG) == 0;

  AccReal mean, invstd;
  if (is_train_and_not_global_stats) {
    mean = ScalarConvert<DType, AccReal>::to(tensors.saveMean[plane]);
    invstd = tensors.saveInvStd[plane];
  } else {
    mean = ScalarConvert<DType, AccReal>::to(tensors.runningMean[plane]);
    invstd = VARIANCE_TO_INVSTD(tensors.runningVar[plane], eps);
  }

  const AccReal weightVal = ((flags & FIX_GAMMA_FLAG) == 0 && tensors.weight.numElements() > 0) ?
                      ScalarConvert<DType, AccReal>::to(tensors.weight[plane]) : AccReal(1);
  const AccReal norm = AccReal(1) / N;

  // Compute two values across (batch, x/y/z) in one pass:
  // 1. Sum(gradOutput)
  // 2. DotProduct(input - mean, gradOutput)
  GradOp<DType, AccReal, DeviceTensor> g(mean, input, gradOutput);
  Float2< DType, AccReal > res = reduce < Float2 < DType, AccReal >,
    GradOp< DType, AccReal, DeviceTensor >, DeviceTensor > (g, gradOutput, plane);
  const AccReal gradOutputSum = res.v1;
  const AccReal dotP = res.v2;

  const AccReal gradMean = gradOutputSum * norm;
  const AccReal projScale = dotP * norm * invstd * invstd;
  const AccReal gradScale = invstd * weightVal;

  if (threadIdx.x == 0 && is_train_and_not_global_stats) {
    const AccReal localVariance = INVSTD_TO_VARIANCE(tensors.saveInvStd[plane], eps);
    const AccReal localMean = tensors.saveMean[plane];

    // update running averages
    tensors.runningMean[plane] = tensors.runningMean[plane]
                                 * momentum + localMean * (AccReal(1) - momentum);
    tensors.runningVar[plane] = tensors.runningVar[plane]
                                * momentum + localVariance * (AccReal(1) - momentum);
  }

  if (gradInput.Size() > 0 && (flags & (WRITE_DATA_FLAG | ADDTO_DATA_FLAG)) != 0) {
    const bool grad_write = flags & WRITE_DATA_FLAG;
    if (grad_write) {
      for (int batch = 0, nbatch = gradOutput.OuterSize(); batch < nbatch; ++batch) {
        for (int x = threadIdx.x, nx = gradOutput.InnerSize(); x < nx; x += blockDim.x) {
          const DType gradOut = gradOutput.get_ref(batch, plane, x);
          if (is_train_and_not_global_stats) {
            const DType inp = input.get_ref(batch, plane, x);
            const AccReal proj = (inp - mean) * projScale;
            gradInput.get_ref(batch, plane, x) =
              ScalarConvert<AccReal, DType>::to((gradOut - proj - gradMean) * gradScale);
          } else {
            gradInput.get_ref(batch, plane, x) = ScalarConvert<AccReal, DType>::to(
              gradOut * gradScale);
          }
        }
      }
    } else {
      // grad addto
      for (int batch = 0, nbatch = gradOutput.OuterSize(); batch < nbatch; ++batch) {
        for (int x = threadIdx.x, nx = gradOutput.InnerSize(); x < nx; x += blockDim.x) {
          const DType gradOut = gradOutput.get_ref(batch, plane, x);
          if (is_train_and_not_global_stats) {
            const DType inp = input.get_ref(batch, plane, x);
            const AccReal proj = (inp - mean) * projScale;
            gradInput.get_ref(batch, plane, x) +=
              ScalarConvert<AccReal, DType>::to((gradOut - proj - gradMean) * gradScale);
          } else {
            gradInput.get_ref(batch, plane, x) += ScalarConvert<AccReal, DType>::to(
              gradOut * gradScale);
          }
        }
      }
    }
  }

  if (tensors.gradWeight.numElements() > 0 && threadIdx.x == 0 &&
      (flags & (WRITE_GAMMA_FLAG | ADDTO_GAMMA_FLAG)) != 0) {
    if ((flags & FIX_GAMMA_FLAG) == 0) {
      if (flags & WRITE_GAMMA_FLAG)
        tensors.gradWeight[plane] = ScalarConvert<AccReal, DType>::to(dotP * invstd);
      else
        tensors.gradWeight[plane] += ScalarConvert<AccReal, DType>::to(dotP * invstd);
    } else {
      tensors.gradWeight[plane] = DType(0);
    }
  }

  if (tensors.gradBias.numElements() > 0 && threadIdx.x == 0 &&
      (flags & (WRITE_BETA_FLAG | ADDTO_BETA_FLAG)) != 0) {
    if (flags & WRITE_BETA_FLAG)
      tensors.gradBias[plane] = ScalarConvert<AccReal, DType>::to(gradOutputSum);
    else
      tensors.gradBias[plane] += ScalarConvert<AccReal, DType>::to(gradOutputSum);
  }
}

template<typename DType, int Dim>
struct DeviceTensor {
 public:
  inline DeviceTensor() {}
  inline DeviceTensor(DType *p, const int *size)
    : dptr_(p) {
    for (int i = 0; i < Dim; ++i) {
      size_[i] = size ? size[i] : 0;
    }
  }

  MSHADOW_XINLINE unsigned getSize(const int i) const {
    return size_[i];
  }

  MSHADOW_XINLINE int numElements() const {
    int n = 1;
    for (int i = 0; i < Dim; ++i) {
      n *= size_[i];
    }
    return n;
  }

  MSHADOW_XINLINE DType &operator()(const size_t batch,
                                    const size_t plane,
                                    const size_t x) const {
    int offset = 0;

    offset *= size_[0];
    offset += batch;

    offset *= size_[1];
    offset += plane;

    offset *= size_[2];
    offset += x;

    return *(const_cast<DType *>(dptr_ + offset));
  }

  MSHADOW_XINLINE DType &operator[](const size_t x) const {
    return *(dptr_ + x);
  }

  MSHADOW_XINLINE size_t InnerSize() const {
    size_t sz = 1;
    for (size_t i = 2; i < Dim; ++i) {
      sz *= size_[i];
    }
    return sz;
  }

  MSHADOW_XINLINE size_t ChannelCount() const {
    return size_[1];
  }

  DType *dptr_;
  int size_[Dim];
};

template<typename DType, int Dim>
static DeviceTensor<DType, Dim> devicetensor(const TBlob &blob) {
  CHECK_EQ(blob.type_flag_, mshadow::DataType<DType>::kFlag);
  DType *data = blob.dptr<DType>();
  const int inDim = blob.shape_.ndim();
  if (inDim == Dim) {
    DeviceTensor<DType, Dim> tensor(data, nullptr);
    for (int i = 0; i < Dim; ++i) {
      tensor.size_[i] = blob.size(i);
    }
    return tensor;
  }

  // View in which the last dimensions are collapsed or expanded as needed
  int size[Dim];
  for (int i = 0; i < Dim || i < inDim; ++i) {
    if (i < Dim && i < inDim) {
      size[i] = blob.size(i);
    } else if (i < Dim) {
      size[i] = 1;
    } else {
      size[Dim - 1] *= blob.size(i);
    }
  }
  return DeviceTensor<DType, Dim>(data, &size[0]);
}


#define DeviceTensor1 DeviceTensor<AccReal, 1>

using namespace mxnet::op;

template<typename DType, typename AccReal>
static void BatchNormalizationUpdateOutput(mshadow::Stream<gpu> *s,
                                           const OpContext &ctx,
                                           const BatchNormParam& param,
                                           const std::vector<TBlob> &in_data,
                                           const std::vector<TBlob> &out_data,
                                           const std::vector<TBlob> &aux_states,
                                           const uint32_t flags,
                                           double momentum,
                                           double eps) {
  batchnorm::BNTensor3<DType> input  = batchnorm::BNTensor3<DType>(
    in_data[batchnorm::kData], param.axis);
  batchnorm::BNTensor3<DType> output = batchnorm::BNTensor3<DType>(
    out_data[batchnorm::kOut], param.axis);
  DeviceTensor1 weight = devicetensor<AccReal, 1>(in_data[batchnorm::kGamma]);
  DeviceTensor1 bias = devicetensor<AccReal, 1>(in_data[batchnorm::kBeta]);
  DeviceTensor1 runningMean = devicetensor<AccReal, 1>(aux_states[batchnorm::kMovingMean]);
  DeviceTensor1 runningVar = devicetensor<AccReal, 1>(aux_states[batchnorm::kMovingVar]);
  DeviceTensor1 saveMean = devicetensor<AccReal, 1>(out_data[batchnorm::kMean]);
  DeviceTensor1 saveInvStd = devicetensor<AccReal, 1>(out_data[batchnorm::kVar]);

  DCHECK_GT(weight.numElements(), 0);

  if ((flags & IS_TRAINING_FLAG) == 0 || (flags & USE_GLOBAL_STATS_FLAG) != 0) {
    dim3 blocks(input.ChannelCount());
    dim3 threads(batchnorm::cuda::getNumThreads(input.InnerSize(), false));
    BatchNormalizationUpdateOutputInferenceKernel<DType, AccReal, DeviceTensor1,
      batchnorm::BNTensor3<DType>>
      <<< blocks, threads, 0, mshadow::Stream<gpu>::GetStream(s) >>> (
      input, output, runningMean, runningVar, saveMean,
        saveInvStd, weight, bias, eps, flags);
  } else {
    dim3 blocks(input.ChannelCount());
    dim3 threads(batchnorm::cuda::getNumThreads(input.InnerSize(), false));
    BatchNormalizationUpdateOutputKernel<DType, AccReal, DeviceTensor1,
      batchnorm::BNTensor3<DType>>
      << < blocks, threads, 0, mshadow::Stream<gpu>::GetStream(s) >> > (
      input, output, weight, bias, eps, momentum, runningMean, runningVar,
        saveMean, saveInvStd, flags);
  }
  MSHADOW_CUDA_POST_KERNEL_CHECK(BatchNormalizationUpdateOutput);
}

template<typename DType, typename AccReal>
static void BatchNormalizationBackward(mshadow::Stream<gpu> *s,
                                       const OpContext &ctx,
                                       const BatchNormParam& param,
                                       const std::vector<TBlob> &out_grad,
                                       const std::vector<TBlob> &in_data,
                                       const std::vector<TBlob> &out_data,
                                       const std::vector<TBlob> &in_grad,
                                       const std::vector<TBlob> &aux_states,
                                       const uint32_t flags,
                                       double momentum,
                                       double eps) {
  batchnorm::BNTensor3<DType> input = batchnorm::BNTensor3<DType>(
    in_data[batchnorm::kData], param.axis);
  batchnorm::BNTensor3<DType>gradOutput = batchnorm::BNTensor3<DType>(
    out_grad[batchnorm::kOut], param.axis);
  batchnorm::BNTensor3<DType>gradInput = batchnorm::BNTensor3<DType>(
    in_grad[batchnorm::kData], param.axis);

  CHECK_EQ(gradOutput.Size(), gradInput.Size());

  CUDATensors<DeviceTensor1> tensors;

  tensors.gradWeight = devicetensor<AccReal, 1>(in_grad[batchnorm::kGamma]);
  tensors.gradBias = devicetensor<AccReal, 1>(in_grad[batchnorm::kBeta]);
  tensors.weight = devicetensor<AccReal, 1>(in_data[batchnorm::kGamma]);
  tensors.runningMean = devicetensor<AccReal, 1>(aux_states[batchnorm::kMovingMean]);
  tensors.runningVar = devicetensor<AccReal, 1>(aux_states[batchnorm::kMovingVar]);
  tensors.saveMean = devicetensor<AccReal, 1>(out_data[batchnorm::kMean]);
  tensors.saveInvStd = devicetensor<AccReal, 1>(out_data[batchnorm::kVar]);

  DCHECK_GT(tensors.weight.numElements(), 0);
#ifdef NDEBUG
  constexpr bool SMALLER_THREADS = false;
#else
  constexpr bool SMALLER_THREADS = true;
#endif
  dim3 blocks(gradOutput.ChannelCount());
  dim3 threads(batchnorm::cuda::getNumThreads(gradOutput.InnerSize(), SMALLER_THREADS));
  BatchNormalizationBackwardKernel<DType, AccReal, DeviceTensor1, batchnorm::BNTensor3<DType>>
    <<< blocks, threads, 0, mshadow::Stream<gpu>::GetStream(s) >>> (
    input, gradOutput, gradInput, tensors, flags, momentum, eps);
  MSHADOW_CUDA_POST_KERNEL_CHECK(BatchNormalizationBackward);
}

}  // namespace cuda
}  // namespace batchnorm

template<typename xpu, typename DType, typename AccReal>
static inline uint32_t SetupFlags(const OpContext &ctx,
                                  const BatchNormParam& params,
                                  const std::vector<OpReqType> &req) {
  uint32_t flags = 0;
  flags |= ctx.is_train ? IS_TRAINING_FLAG : 0;
  flags |= params.fix_gamma ? FIX_GAMMA_FLAG : 0;
  flags |= params.use_global_stats ? USE_GLOBAL_STATS_FLAG : 0;
  if (IsBNWriting(req[batchnorm::kData])) {
    flags |= WRITE_DATA_FLAG;
  } else if (req[batchnorm::kData] == kAddTo) {
    flags |= ADDTO_DATA_FLAG;
  }
  if (IsBNWriting(req[batchnorm::kGamma])) {
    flags |= WRITE_GAMMA_FLAG;
  } else if (req[batchnorm::kGamma] == kAddTo) {
    flags |= ADDTO_GAMMA_FLAG;
  }
  if (IsBNWriting(req[batchnorm::kBeta])) {
    flags |= WRITE_BETA_FLAG;
  } else if (req[batchnorm::kBeta] == kAddTo) {
    flags |= ADDTO_BETA_FLAG;
  }
  return flags;
}

/*! \brief Forward batch-norm pass on GPU */
template<typename xpu, typename DType, typename AccReal>
void BatchNormForwardImpl(mshadow::Stream<gpu> *stream,
                          const OpContext &ctx, const BatchNormParam& param_,
                          const std::vector<TBlob> &in_data,
                          const std::vector<OpReqType> &req,
                          const std::vector<TBlob> &out_data,
                          const std::vector<TBlob> &aux_states) {
  batchnorm::cuda::BatchNormalizationUpdateOutput<DType, AccReal>(
    stream,
    ctx,
    param_,
    in_data,
    out_data,
    aux_states,
    SetupFlags<xpu, DType, AccReal>(ctx, param_, req),
    param_.momentum,
    param_.eps);
  MSHADOW_CUDA_POST_KERNEL_CHECK(BatchNormOp_DoForward_gpu);
}

/*! \brief Backward batch-norm pass on GPU */
template<typename xpu, typename DType, typename AccReal>
void BatchNormBackwardImpl(mshadow::Stream<gpu> *stream,
                           const OpContext &ctx, const BatchNormParam& param_,
                           const std::vector<TBlob> &out_grad,
                           const std::vector<TBlob> &in_data,
                           const std::vector<TBlob> &out_data,
                           const std::vector<OpReqType> &req,
                           const std::vector<TBlob> &in_grad,
                           const std::vector<TBlob> &aux_states) {
  batchnorm::cuda::BatchNormalizationBackward<DType, AccReal>(
    stream,
    ctx,
    param_,
    out_grad,
    in_data,
    out_data,
    in_grad,
    aux_states,
    SetupFlags<xpu, DType, AccReal>(ctx, param_, req),
    param_.momentum,
    param_.eps);
  MSHADOW_CUDA_POST_KERNEL_CHECK(BatchNormOp_DoBackward_gpu);
}

#if MXNET_USE_CUDNN == 1
template<typename DType>
static CuDNNBatchNormOp<DType> &GetCuDNNOp(const BatchNormParam& param) {
#if DMLC_CXX11_THREAD_LOCAL
  static thread_local CuDNNBatchNormOp<DType> op;
#else
  static MX_THREAD_LOCAL CuDNNBatchNormOp<DType> op;
#endif
  op.Init(param);
  return op;
}
#endif

template<>
void BatchNormCompute<gpu>(const nnvm::NodeAttrs& attrs,
                           const OpContext& ctx, const std::vector<TBlob>& inputs,
                           const std::vector<OpReqType>& req,
                           const std::vector<TBlob>& outputs) {
  BatchNormParam param = nnvm::get<BatchNormParam>(attrs.parsed);
  CHECK_EQ(inputs.size(), 5U);
  std::vector<TBlob> in_data(inputs.begin(), inputs.begin() + 3);
  std::vector<TBlob> aux_states(inputs.begin() + 3, inputs.end());
  int dtype = inputs[0].type_flag_;
  mxnet::TShape shape = inputs[0].shape_;

  param.axis = mxnet::op::batchnorm::GetRealAxis(shape, param.axis);
#if MXNET_USE_CUDNN == 1
  if (!param.use_global_stats && !param.cudnn_off) {
    MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
      GetCuDNNOp<DType>(param).Forward(ctx, in_data, req, outputs, aux_states);
    })
  } else {
    MSHADOW_REAL_TYPE_SWITCH_EX(dtype, DType, AccReal, {
      BatchNormForward<gpu, DType, AccReal>(ctx, param, in_data, req, outputs, aux_states);
    })
  }
#else
  MSHADOW_REAL_TYPE_SWITCH_EX(inputs[0].type_flag_, DType, AccReal, {
    BatchNormForward<gpu, DType, AccReal>(ctx, param, in_data, req, outputs, aux_states);
  });
#endif
}

template<>
void BatchNormGradCompute<gpu>(const nnvm::NodeAttrs& attrs,
                               const OpContext& ctx, const std::vector<TBlob>& inputs,
                               const std::vector<OpReqType>& req,
                               const std::vector<TBlob>& outputs) {
  CHECK_EQ(inputs.size(), 8U);
  BatchNormParam param = nnvm::get<BatchNormParam>(attrs.parsed);
  int dtype = inputs[0].type_flag_;
  mxnet::TShape shape = inputs[0].shape_;

  param.axis = mxnet::op::batchnorm::GetRealAxis(shape, param.axis);
#if MXNET_USE_CUDNN == 1
  if (!param.use_global_stats && !param.cudnn_off) {
    MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
      GetCuDNNOp<DType>(param).Backward(ctx, inputs, req, outputs);
    })
  } else {
    MSHADOW_REAL_TYPE_SWITCH_EX(dtype, DType, AccReal, {
      BatchNormBackward<gpu, DType, AccReal>(ctx, param, inputs, req, outputs);
    })
  }
#else
  MSHADOW_REAL_TYPE_SWITCH_EX(dtype, DType, AccReal, {
    BatchNormBackward<gpu, DType, AccReal>(ctx, param, inputs, req, outputs);
  });
#endif
}

NNVM_REGISTER_OP(BatchNorm)
.set_attr<FCompute>("FCompute<gpu>", BatchNormCompute<gpu>);

NNVM_REGISTER_OP(_backward_BatchNorm)
.set_attr<FCompute>("FCompute<gpu>", BatchNormGradCompute<gpu>);

}  // namespace op
}  // namespace mxnet
