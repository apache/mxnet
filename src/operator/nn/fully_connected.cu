/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2015 by Contributors
 * \file fully_connected.cu
 * \brief fully connect operator
*/
#include "./fully_connected-inl.h"
namespace mxnet {
namespace op {

template<>
void FullyConnectedCompute<gpu>(const nnvm::NodeAttrs& attrs,
                                const OpContext& ctx,
                                const std::vector<TBlob>& inputs,
                                const std::vector<OpReqType>& req,
                                const std::vector<TBlob>& outputs) {
  const FullyConnectedParam& param = nnvm::get<FullyConnectedParam>(attrs.parsed);
  uint32_t in_expected = param.no_bias ? 2 : 3;
  CHECK_EQ(inputs.size(), in_expected);
  CHECK_EQ(outputs.size(), 1U);
  int dtype = inputs[0].type_flag_;

  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    FCForward<gpu, DType>(ctx, param, inputs, req, outputs);
  });
}

template<>
void FullyConnectedGradCompute<gpu>(const nnvm::NodeAttrs& attrs,
                                    const OpContext& ctx,
                                    const std::vector<TBlob>& inputs,
                                    const std::vector<OpReqType>& req,
                                    const std::vector<TBlob>& outputs) {
  const FullyConnectedParam& param = nnvm::get<FullyConnectedParam>(attrs.parsed);
  uint32_t out_expected = param.no_bias ? 2 : 3;
  CHECK_EQ(inputs.size(), 3U);
  CHECK_EQ(outputs.size(), out_expected);
  CHECK_EQ(req.size(), out_expected);

  std::vector<TBlob> out_grad{inputs[0]};
  std::vector<TBlob> in_data(inputs.begin() + 1, inputs.end());
  int dtype = inputs[0].type_flag_;

  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    FCBackward<gpu, DType>(ctx, param, out_grad, in_data, req, outputs);
  });
}

NNVM_REGISTER_OP(FullyConnected)
.set_attr<FCompute>("FCompute<gpu>", FullyConnectedCompute<gpu>);

NNVM_REGISTER_OP(_backward_FullyConnected)
.set_attr<FCompute>("FCompute<gpu>", FullyConnectedGradCompute<gpu>);

}  // namespace op
}  // namespace mxnet
