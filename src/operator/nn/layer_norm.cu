#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file layer_norm.cu
 * \brief Implements Ba et. al, Layer Normalization (https://arxiv.org/abs/1607.06450).
 */
#include "./layer_norm-inl.h"

using namespace mshadow::cuda;

namespace mxnet {
namespace op {

template <>
void LayerNormGradComputeGeneralImpl<gpu>(const nnvm::NodeAttrs& attrs,
                                          const OpContext& ctx,
                                          const TBlob& ograd,
                                          const TBlob& data,
                                          const TBlob& gamma,
                                          const TBlob& mean,
                                          const TBlob& std,
                                          const TBlob& normalized_data,
                                          const TBlob& ograd_mult,
                                          const TBlob& red_out,
                                          const std::vector<OpReqType>& req,
                                          const std::vector<TBlob>& outputs,
                                          const mshadow::Tensor<gpu, 1, char>& workspace,
                                          const mxnet::TShape& red_dst_shape,
                                          const mxnet::TShape& red_src_shape,
                                          const mxnet::TShape& red_exclude_dst_shape,
                                          const mxnet::TShape& red_exclude_src_shape,
                                          const int channel_size) {
  using namespace mshadow;
  using namespace mshadow::expr;
  Stream<gpu>* s = ctx.get_stream<gpu>();
  // Compute normalized_data = (data - mean) / std
  BinaryBroadcastRTCCompute{"sub"}(  // NOLINT
      attrs,
      ctx,
      {data, mean},
      {kWriteTo},
      {normalized_data});
  BinaryBroadcastRTCCompute{"div"}(  // NOLINT
      attrs,
      ctx,
      {normalized_data, std},
      {kWriteTo},
      {normalized_data});
  // Calculate grad_beta
  if (req[2] != kNullOp) {
    BROADCAST_NDIM_SWITCH(red_exclude_dst_shape.ndim(), NDim, {
      broadcast::RTCReduce(ctx,
                           outputs[2].reshape(red_exclude_dst_shape),
                           req[2],
                           workspace,
                           ograd.reshape(red_exclude_src_shape),
                           "red::sum{}",
                           NDim,
                           "identity");
    });
  }
  // Calculate grad_gamma, it will be sum(ograd * normalized_data, exclude_axis)
  ElemwiseBinaryRTCCompute{"mul"}(  // NOLINT
      attrs,
      ctx,
      {normalized_data, ograd},
      {kWriteTo},
      {ograd_mult});
  if (req[1] != kNullOp) {
    BROADCAST_NDIM_SWITCH(red_exclude_dst_shape.ndim(), NDim, {
      broadcast::RTCReduce(ctx,
                           outputs[1].reshape(red_exclude_dst_shape),
                           req[1],
                           workspace,
                           ograd_mult.reshape(red_exclude_src_shape),
                           "red::sum{}",
                           NDim,
                           "identity");
    });
  }
  // Calculate grad_data:
  //   ograd_mult = ograd * gamma / std
  //   grad_data = ograd_mult - mean(ograd_mult, axis)
  //               + normalized_data * (-mean(normalized_data * ograd_mult, axis))
  if (req[0] != kNullOp) {
    BinaryBroadcastRTCCompute{"mul"}(  // NOLINT
        attrs,
        ctx,
        {ograd, gamma},
        {kWriteTo},
        {ograd_mult});
    BinaryBroadcastRTCCompute{"div"}(  // NOLINT
        attrs,
        ctx,
        {ograd_mult, std},
        {kWriteTo},
        {ograd_mult});
    BROADCAST_NDIM_SWITCH(red_dst_shape.ndim(), NDim, {
      broadcast::RTCReduce(ctx,
                           red_out.reshape(red_dst_shape),
                           kWriteTo,
                           workspace,
                           ograd_mult.reshape(red_src_shape),
                           "red::sum{}",
                           NDim,
                           "identity");
    });
    MSHADOW_REAL_TYPE_SWITCH(outputs[0].type_flag_, DType, {
      Tensor<gpu, 1, DType> red_out_tensor = red_out.FlatTo1D<gpu, DType>(s);
      red_out_tensor /= scalar<DType>(channel_size);
    });
    BinaryBroadcastRTCCompute{"sub"}(  // NOLINT
        attrs,
        ctx,
        {ograd_mult, red_out},
        {req[0]},
        {outputs[0]});
    ElemwiseBinaryRTCCompute{"mul"}(  // NOLINT
        attrs,
        ctx,
        {ograd_mult, normalized_data},
        {kWriteTo},
        {ograd_mult});
    BROADCAST_NDIM_SWITCH(red_dst_shape.ndim(), NDim, {
      broadcast::RTCReduce(ctx,
                           red_out.reshape(red_dst_shape),
                           kWriteTo,
                           workspace,
                           ograd_mult.reshape(red_src_shape),
                           "red::sum{}",
                           NDim,
                           "identity");
    });
    MSHADOW_REAL_TYPE_SWITCH(outputs[0].type_flag_, DType, {
      Tensor<gpu, 1, DType> red_out_tensor = red_out.FlatTo1D<gpu, DType>(s);
      red_out_tensor /= scalar<DType>(-channel_size);
    });
    BinaryBroadcastRTCCompute{"mul"}(  // NOLINT
        attrs,
        ctx,
        {normalized_data, red_out},
        {kAddTo},
        {outputs[0]});
  }
}
template <typename DType>
__device__ __forceinline__ DType
warp_shfl(DType value, int src_lane, int width = 32, unsigned int mask = 0xffffffff) {
#if CUDA_VERSION >= 9000
  return __shfl_sync(mask, value, src_lane, width);
#else
  return __shfl(value, src_lane, width);
#endif
}

template <typename DType>
__device__ __forceinline__ DType
warp_shfl_xor(DType value, int laneMask, int width = 32, unsigned int mask = 0xffffffff) {
#if CUDA_VERSION >= 9000
  return __shfl_xor_sync(mask, value, laneMask, width);
#else
  return __shfl_xor(value, laneMask, width);
#endif
}

/* A single updating step of the Welford's online algorithm to calculate the mean and variance.
 * The value 'curr' will be accumulated to the (mean, sigma2, count) triplet.
 *
 */
template <typename DType, typename IType>
__device__ __forceinline__ void StepWelfordOnlineSum(const DType curr,
                                                     DType& mean,     // NOLINT
                                                     DType& sigma2,   // NOLINT
                                                     IType& count) {  // NOLINT
  count += IType(1);
  DType delta = curr - mean;
  mean += delta / count;
  sigma2 += delta * (curr - mean);
}

/* Merge the mean/variance of two partitions. It's the key step of the Chan's parallel algorithm.
 * The (lhs_mean, lhs_sigma2, lhs_count) will be merged into (rhs_mean, rhs_sigma2, rhs_count)
 *
 * See https://en.wikipedia.org/wiki/Algorithms_for_calculating_variance for more details.
 *
 *  TODO(sxjscience) Explore the possibility of int lhs_count and rhs_count
 */
template <typename DType, typename IType>
__device__ __inline__ void ChanMergePartition(const DType lhs_mean,
                                              const DType lhs_sigma2,
                                              const IType lhs_count,
                                              DType& rhs_mean,     // NOLINT
                                              DType& rhs_sigma2,   // NOLINT
                                              IType& rhs_count) {  // NOLINT
  DType delta = rhs_mean - lhs_mean;
  DType nA    = static_cast<DType>(lhs_count);
  DType nB    = static_cast<DType>(rhs_count);
  rhs_count   = nA + nB;
  if (rhs_count > DType(0)) {
    nA         = nA / rhs_count;
    nB         = nB / rhs_count;
    rhs_mean   = nA * lhs_mean + nB * rhs_mean;
    rhs_sigma2 = rhs_sigma2 + lhs_sigma2 + delta * delta * nA * nB * rhs_count;
  } else {
    rhs_mean   = DType(0);
    rhs_sigma2 = DType(0);
  }
}

/* Split the input column into multiple partitions and compute the mean/sigma of each partition.
 * Each thread will keep a mean/sigma2. The mean/sigma2 can be further merged to get the mean and
 * sigma2 of the column.
 */
template <typename AType, typename DType, typename IType>
__device__ __forceinline__ void BlockWelfordOnlineSum(const DType* __restrict__ col_vals,
                                                      const int nchannel,
                                                      AType& mean,     // NOLINT
                                                      AType& sigma2,   // NOLINT
                                                      IType& count) {  // NOLINT
  int tid           = threadIdx.x + threadIdx.y * blockDim.x;
  const int nthread = blockDim.x * blockDim.y;
  // Each thread takes charge of 4 consecutive numbers. This should optimize the loading speed using
  // vectorized types like float4.
  // Also, to minimize branch divergence, we split the for-loop into two parts.
  int l = 4 * tid;
  for (; l + 3 < nchannel; l += 4 * nthread) {
#pragma unroll
    for (int i = 0; i < 4; ++i) {
      StepWelfordOnlineSum(static_cast<AType>(col_vals[l + i]), mean, sigma2, count);
    }
  }
  for (; l < nchannel; ++l) {
    StepWelfordOnlineSum(static_cast<AType>(col_vals[l]), mean, sigma2, count);
  }
}

template <>
__device__ __forceinline__ void BlockWelfordOnlineSum<float, mshadow::half::half_t, int>(
    const mshadow::half::half_t* __restrict__ col_vals,
    const int nchannel,
    float& mean,    // NOLINT
    float& sigma2,  // NOLINT
    int& count) {   // NOLINT
  int tid           = threadIdx.x + threadIdx.y * blockDim.x;
  const int nthread = blockDim.x * blockDim.y;
  // We cast the input half pointer to half2 to optimize the loading speed.
  // Here, we need to notice that CUDA forces memory alignment, i.e.,
  // ASSERT static_cast<size_t>(ptr) % sizeof(dtype) == 0.
  // Thus, we need to shift the address of the half pointer to be aligned by half2.
  int align_shift               = (reinterpret_cast<size_t>(col_vals) % 4) != 0;
  int padding                   = (nchannel - align_shift) % 2;
  int half2_size                = (nchannel - align_shift) / 2;
  const __half2* half2_col_vals = reinterpret_cast<const __half2*>(col_vals + align_shift);
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    if (align_shift) {
      StepWelfordOnlineSum(__half2float(col_vals[0].cuhalf_), mean, sigma2, count);
    }
    if (padding) {
      StepWelfordOnlineSum(__half2float(col_vals[nchannel - 1].cuhalf_), mean, sigma2, count);
    }
  }

  for (int l = tid; l < half2_size; l += nthread) {
    float2 ele_val = __half22float2(half2_col_vals[l]);
    StepWelfordOnlineSum(ele_val.x, mean, sigma2, count);
    StepWelfordOnlineSum(ele_val.y, mean, sigma2, count);
  }
}

/* Fused CUDA kernel for the forward pass of layer normalization.
 * It computes the LayerNorm when axis=-1, i.e., contiguous reduction scenario.
 * Shape of the input tensors:
 *      in_data = (nbatch, nchannel)
 *      gamma = (nchannel,)
 *      beta = (nchannel,)
 *      out_data = (nchannel,)
 *      mean_data = (nbatch,)
 *      var_data = (nbatch,)
 *  It's always launched with (blockDim.x, blockDim.y) = (WARP_SIZE, blockDim.y)
 *  Also, when blockDim.y > 1, it requires shared memory that has size:
 *      sizeof(AType) * blockDim.y + sizeof(int) * blockDim.y / 2
 */
template <typename AType, typename DType, typename IType>
__global__ void LayerNormFusedForwardKernelContig(const int nbatch,
                                                  const int nchannel,
                                                  const AType eps,
                                                  const DType* __restrict__ in_data,
                                                  const DType* __restrict__ gamma,
                                                  const DType* __restrict__ beta,
                                                  DType* __restrict__ out_data,
                                                  DType* __restrict__ mean_data,
                                                  DType* __restrict__ std_data) {
  int bid           = blockIdx.x + blockIdx.y * gridDim.x;
  const int tid     = threadIdx.y * blockDim.x + threadIdx.x;
  const int nthread = blockDim.x * blockDim.y;
  IType count       = 0;
  AType mean        = 0;
  AType sigma2      = 0;

  if (bid < nbatch) {
    extern __shared__ char buf[];  // Shared memory
    const DType* col_vals = in_data + bid * nchannel;
    BlockWelfordOnlineSum(col_vals, nchannel, mean, sigma2, count);

    // Merge the mean/sigma2 within a warp
    // Use the Chan's Parallel Algorithm to merge all (mean, sigma2, counts)
    // within a warp of threads.
    // After calling the function, threadIdx.x == 0 will store the result of
    // the aggregated (mean, sigma2, counts).
    for (int mask = blockDim.x / 2; mask > 0; mask >>= 1) {
      AType meanB   = warp_shfl_xor(mean, mask);
      AType sigma2B = warp_shfl_xor(sigma2, mask);
      IType countB  = warp_shfl_xor(count, mask);
      ChanMergePartition(meanB, sigma2B, countB, mean, sigma2, count);
    }
    if (blockDim.y > 1) {
      // Inter-warp reduction. Copy the upper-half of the warps to shared memory
      // and merge with the lower-half warp
      AType* mean_buf = reinterpret_cast<AType*>(buf);
      AType* sigma2_buf =
          reinterpret_cast<AType*>(buf + sizeof(AType) * blockDim.y / 2 * blockDim.x);
      IType* count_buf = reinterpret_cast<IType*>(buf + sizeof(AType) * blockDim.y * blockDim.x);
      for (int offset = blockDim.y / 2; offset > 0; offset >>= 1) {
        if (threadIdx.y >= offset && threadIdx.y < 2 * offset) {
          const int idx   = (threadIdx.y - offset) * blockDim.x + threadIdx.x;
          mean_buf[idx]   = mean;
          sigma2_buf[idx] = sigma2;
          count_buf[idx]  = count;
        }
        __syncthreads();
        if (threadIdx.y < offset) {
          const int idx = threadIdx.y * blockDim.x + threadIdx.x;
          ChanMergePartition(mean_buf[idx], sigma2_buf[idx], count_buf[idx], mean, sigma2, count);
        }
        __syncthreads();
      }
      // Broadcast the result to all threads
      if (threadIdx.y == 0) {
        mean_buf[threadIdx.x]   = mean;
        sigma2_buf[threadIdx.x] = sigma2;
      }
      __syncthreads();
      mean   = mean_buf[threadIdx.x];
      sigma2 = sigma2_buf[threadIdx.x] / nchannel;
    } else {
      sigma2 /= nchannel;
    }
    // Calculate the out_data: gamma * (x - mean) / sqrt(var + eps) + beta
    AType std_eps      = sqrt(sigma2 + eps);
    AType invstd_eps   = DType(1.0) / std_eps;
    DType* out_col_val = out_data + bid * nchannel;

    if (gamma != nullptr && beta != nullptr) {
      for (int i = tid; i < nchannel; i += nthread) {
        out_col_val[i] =
            gamma[i] * static_cast<DType>(invstd_eps * (static_cast<AType>(col_vals[i]) - mean)) +
            beta[i];
      }
    } else if (gamma == nullptr && beta != nullptr) {
      for (int i = tid; i < nchannel; i += nthread) {
        out_col_val[i] =
            static_cast<DType>(invstd_eps * (static_cast<AType>(col_vals[i]) - mean)) + beta[i];
      }
    } else if (gamma != nullptr && beta == nullptr) {
      for (int i = tid; i < nchannel; i += nthread) {
        out_col_val[i] =
            gamma[i] * static_cast<DType>(invstd_eps * (static_cast<AType>(col_vals[i]) - mean));
      }
    } else {
      for (int i = tid; i < nchannel; i += nthread) {
        out_col_val[i] = static_cast<DType>(invstd_eps * (static_cast<AType>(col_vals[i]) - mean));
      }
    }
    // Write the out_data and var_data
    if (threadIdx.x == 0 && threadIdx.y == 0) {
      mean_data[bid] = static_cast<DType>(mean);
      std_data[bid]  = static_cast<DType>(std_eps);
    }
  }
}

template <bool safe_acc = false>
void LayerNormGPUContig(const LayerNormParam param,
                        const OpContext& ctx,
                        const std::vector<TBlob>& inputs,
                        const std::vector<OpReqType>& req,
                        const std::vector<TBlob>& outputs) {
  using namespace mshadow;
  CHECK_EQ(inputs.size(), 3U);
  mxnet::TShape data_shape(2, 0);
  mxnet::TShape mean_shape(1, 0);
  size_t in_ndim = inputs[layernorm::kData].ndim();
  data_shape[0] = mean_shape[0] = inputs[layernorm::kData].shape_.ProdShape(0, in_ndim - 1);
  data_shape[1]                 = inputs[layernorm::kData].shape_[in_ndim - 1];
  const TBlob in_data           = inputs[layernorm::kData].reshape(data_shape);
  const TBlob gamma             = inputs[layernorm::kGamma];
  const TBlob beta              = inputs[layernorm::kBeta];
  const TBlob out_data          = outputs[layernorm::kOut].reshape(data_shape);
  const TBlob mean_data         = outputs[layernorm::kMean].reshape(mean_shape);
  const TBlob std_data          = outputs[layernorm::kStd].reshape(mean_shape);
  // Make sure the inputs are contiguous
  CHECK_EQ(in_data.CheckContiguous(), true);
  CHECK_EQ(gamma.CheckContiguous(), true);
  CHECK_EQ(beta.CheckContiguous(), true);
  CHECK_EQ(out_data.CheckContiguous(), true);
  CHECK_EQ(mean_data.CheckContiguous(), true);
  CHECK_EQ(std_data.CheckContiguous(), true);

  // Lauch the kernel. The dynamic shared memory size is
  // sizeof(DType) * blockDim.y * blockDim.x + sizeof(DType) * blockDim.y / 2 * blockDim.x
  int nbatch   = data_shape[0];
  int nchannel = data_shape[1];
  float eps    = param.eps;
  int ngrid_x  = (nbatch > kMaxGridDim) ? (nbatch + kBaseGridNum - 1) / kBaseGridNum : nbatch;
  int ngrid_y  = (nbatch > kMaxGridDim) ? kBaseGridNum : 1;
  int nthread_y;
  const dim3 dimGrid(ngrid_x, ngrid_y);
  if (nchannel <= 128) {
    nthread_y = 1;
  } else if (nchannel <= 512) {
    nthread_y = 2;
  } else {
    nthread_y = 4;
  }
  hipStream_t stream = Stream<gpu>::GetStream(ctx.get_stream<gpu>());
  const dim3 dimBlock(32, nthread_y);
  MXNET_REAL_ACC_TYPE_SWITCH(in_data.type_flag_, DType, AccType, {
    typedef typename std::conditional<safe_acc, AccType, DType>::type AType;
    int nshared =
        nthread_y > 1 ? nthread_y * 32 * sizeof(AType) + (nthread_y / 2) * 32 * sizeof(int) : 0;
    CheckLaunchParam(dimGrid, dimBlock);
    LayerNormFusedForwardKernelContig<AType, DType, int>
        <<<dimGrid, dimBlock, nshared, stream>>>(nbatch,
                                                 nchannel,
                                                 static_cast<AType>(eps),
                                                 in_data.dptr<DType>(),
                                                 gamma.dptr<DType>(),
                                                 beta.dptr<DType>(),
                                                 out_data.dptr<DType>(),
                                                 mean_data.dptr<DType>(),
                                                 std_data.dptr<DType>());
  });
  MSHADOW_CUDA_POST_KERNEL_CHECK(LayerNormFusedForwardKernelContig);
}

template <>
void LayerNormCompute<gpu>(const nnvm::NodeAttrs& attrs,
                           const OpContext& ctx,
                           const std::vector<TBlob>& inputs,
                           const std::vector<OpReqType>& req,
                           const std::vector<TBlob>& outputs) {
  const LayerNormParam& param = nnvm::get<LayerNormParam>(attrs.parsed);
  if (req[0] == kNullOp)
    return;
  CHECK_NE(req[0], kAddTo);
  int axis = param.axis;
  if (axis < 0) {
    axis += static_cast<int>(inputs[0].ndim());
  }
  CHECK(axis >= 0 && axis < inputs[0].ndim()) << "Channel axis out of range: " << param.axis;
  if (axis == inputs[0].ndim() - 1) {
    // Try to use the accelerated CUDA kernels
    bool safe_acc = dmlc::GetEnv("MXNET_SAFE_ACCUMULATION", true);
    if (!safe_acc && inputs[0].type_flag_ == mshadow::kFloat16) {
      common::LogOnce(
          "MXNET_SAFE_ACCUMULATION=1 is recommended for LayerNorm with float16 inputs. "
          "See https://mxnet.apache.org/api/faq/env_var "
          "for more details.");
    }
    if (safe_acc) {
      return LayerNormGPUContig<true>(param, ctx, inputs, req, outputs);
    } else {
      return LayerNormGPUContig<false>(param, ctx, inputs, req, outputs);
    }
  }
  return LayerNormComputeGeneral<gpu>(attrs, ctx, inputs, req, outputs);
}

/* Fused CUDA kernel for calculating the gradient w.r.t gamma/beta in LayerNorm when axis=-1
 * (Contiguous case).
 * The gradient of gamma and beta are:
 *   d_gamma = sum(out_grad * (x - mean) / std, axis=0)
 *   d_beta = sum(out_grad, axis=0)
 *
 * We compute the gradient (mainly reduction over a non-contiguous axis) using two steps to
 * improve the parallelism.
 *
 * In the first step, we divide the rows uniformly into K parts. K independent threadblocks are used
 * to calculate the partial reduction result of each part. Illustrated below:
 *
 *      1st Block          2nd Block          3rd Block              k-th Block
 * | --------------- | ---------------- | --------------- | ... | ---------------- |
 * | --------------- | ---------------- | --------------- | ... | ---------------- |
 * | --------------- | ---------------- | --------------- | ... | ---------------- |
 * | --------------- | ---------------- | --------------- | ... | ---------------- |
 *     part_gamma[0]     part_gamma[1]      part_gamma[2]           part_gamma[k-1]
 *     part_beta[0]      part_beta[1]       part_beta[2]            part_beta[k-1]
 *
 *
 * In the second step, we sum up the row-values in part_gamma and part_beta.
 *
 * This `LayerNormFusedBackwardKernel_PartGammaBeta` function implements the first step and
 * `LayerNormFusedBackwardKernel_GammaBeta` implements the second step.
 */
template <typename AType, typename DType>
__global__ void LayerNormFusedBackwardKernel_PartGammaBeta(const int nbatch,
                                                           const int nchannel,
                                                           const DType* __restrict__ in_data,
                                                           const DType* __restrict__ out_grad,
                                                           const DType* __restrict__ mean_data,
                                                           const DType* __restrict__ std_data,
                                                           AType* __restrict__ part_gamma_grad,
                                                           AType* __restrict__ part_beta_grad) {
  extern __shared__ char buf[];
  AType* d_buf            = reinterpret_cast<AType*>(buf);
  const int npart         = gridDim.y;
  const int block_row_num = (nbatch + npart - 1) / npart;
  // The rows are divided into `npart` parts. Each threadblock calculates the reduction result
  // within the corresponding row ranges.
  int row_stride         = blockDim.x + 1;
  const int c            = blockIdx.x * blockDim.x + threadIdx.x;
  int r_begin            = blockIdx.y * block_row_num;
  int r_end              = min((blockIdx.y + 1) * block_row_num, nbatch);
  AType* buf_gamma_grad  = d_buf;
  AType* buf_beta_grad   = d_buf + blockDim.y * row_stride;
  AType local_gamma_grad = 0;
  AType local_beta_grad  = 0;

  if (c < nchannel) {
    for (int r_b = r_begin; r_b < r_end; r_b += blockDim.y) {
      int r = r_b + threadIdx.y;
      if (r < r_end) {
        AType local_mean     = static_cast<AType>(mean_data[r]);
        AType local_std      = static_cast<AType>(std_data[r]);
        int read_idx         = r * nchannel + c;
        AType local_in_data  = static_cast<AType>(in_data[read_idx]);
        AType local_out_grad = static_cast<AType>(out_grad[read_idx]);
        local_gamma_grad += (local_in_data - local_mean) / local_std * local_out_grad;
        local_beta_grad += local_out_grad;
      }
    }
  }
  buf_gamma_grad[threadIdx.y * row_stride + threadIdx.x] = local_gamma_grad;
  buf_beta_grad[threadIdx.y * row_stride + threadIdx.x]  = local_beta_grad;
  __syncthreads();
  for (int offset = blockDim.y / 2; offset > 1; offset >>= 1) {
    if (threadIdx.y < offset) {
      int idx1 = threadIdx.y * row_stride + threadIdx.x;
      int idx2 = (threadIdx.y + offset) * row_stride + threadIdx.x;
      buf_gamma_grad[idx1] += buf_gamma_grad[idx2];
      buf_beta_grad[idx1] += buf_beta_grad[idx2];
    }
    __syncthreads();
  }
  if (threadIdx.y == 0 && c < nchannel) {
    part_gamma_grad[blockIdx.y * nchannel + c] =
        buf_gamma_grad[threadIdx.x] + buf_gamma_grad[threadIdx.x + row_stride];
    part_beta_grad[blockIdx.y * nchannel + c] =
        buf_beta_grad[threadIdx.x] + buf_beta_grad[threadIdx.x + row_stride];
  }
}

template <bool gamma_addto, bool beta_addto, typename AType, typename DType>
__global__ void LayerNormFusedBackwardKernel_GammaBeta(const int nbatch,
                                                       const int nchannel,
                                                       const int npart,
                                                       const AType* __restrict__ part_gamma_grad,
                                                       const AType* __restrict__ part_beta_grad,
                                                       DType* gamma_grad,
                                                       DType* beta_grad) {
  const int c   = blockIdx.x * blockDim.x + threadIdx.x;
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;
  if (c < nchannel) {
    extern __shared__ char buf[];
    AType* buf_gamma_grad = reinterpret_cast<AType*>(buf);
    AType* buf_beta_grad  = reinterpret_cast<AType*>(buf) + blockDim.x * blockDim.y;
    buf_gamma_grad[tid]   = 0;
    buf_beta_grad[tid]    = 0;
    for (int r = threadIdx.y; r < npart; r += blockDim.y) {
      buf_gamma_grad[tid] += part_gamma_grad[r * nchannel + c];
      buf_beta_grad[tid] += part_beta_grad[r * nchannel + c];
    }
    __syncthreads();
    // Begin for inter-warp reduce
    if (npart > 1) {
      for (int offset = blockDim.y / 2; offset > 0; offset >>= 1) {
        if (threadIdx.y < offset) {
          int idx1 = tid;
          int idx2 = tid + offset * blockDim.x;
          buf_gamma_grad[idx1] += buf_gamma_grad[idx2];
          buf_beta_grad[idx1] += buf_beta_grad[idx2];
        }
        __syncthreads();
      }
    }
    if (threadIdx.y == 0) {
      if (gamma_grad) {
        if (gamma_addto) {
          gamma_grad[c] += static_cast<DType>(buf_gamma_grad[threadIdx.x]);
        } else {
          gamma_grad[c] = static_cast<DType>(buf_gamma_grad[threadIdx.x]);
        }
      }
      if (beta_grad) {
        if (beta_addto) {
          beta_grad[c] += static_cast<DType>(buf_beta_grad[threadIdx.x]);
        } else {
          beta_grad[c] = static_cast<DType>(buf_beta_grad[threadIdx.x]);
        }
      }
    }
  }
}

/*
 *
 *
 */
template <int LOAD_UNROLL, bool data_addto, typename AType, typename DType>
__global__ void LayerNormFusedBackwardKernel_Data(const int nbatch,
                                                  const int nchannel,
                                                  const DType* __restrict__ in_data,
                                                  const DType* __restrict__ out_grad,
                                                  const DType* __restrict__ mean_data,
                                                  const DType* __restrict__ std_data,
                                                  const DType* __restrict__ gamma,
                                                  DType* data_grad) {
  int bid           = blockIdx.x + blockIdx.y * gridDim.x;
  const int nthread = blockDim.x * blockDim.y;
  if (bid < nbatch) {
    // Shared memory with size blockDim.y * blockDim.x * sizeof(DType)
    extern __shared__ char buf[];
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    // 1. Calculate: mean(out_grad * gamma / std, axis=-1)
    //               mean(out_grad * gamma / std * (x - mean) / std, axis=-1)
    AType sum_val0   = 0;  // Stores mean(out_grad * gamma / std, axis=-1)
    AType sum_val1   = 0;  // Stores mean(out_grad * gamma / std * (x - mean) / std, axis=-1)
    AType mean       = static_cast<AType>(mean_data[bid]);
    AType invstd_eps = AType(1) / static_cast<AType>(std_data[bid]);
    int l            = LOAD_UNROLL * tid;
    for (; l + LOAD_UNROLL - 1 < nchannel; l += nthread * LOAD_UNROLL) {
#pragma unroll
      for (int i = 0; i < LOAD_UNROLL; ++i) {
        AType ele_og    = static_cast<AType>(out_grad[bid * nchannel + l + i]);
        AType ele_x     = static_cast<AType>(in_data[bid * nchannel + l + i]);
        AType ele_gamma = static_cast<AType>(gamma[l + i]);
        sum_val0 += ele_og * ele_gamma * invstd_eps;
        sum_val1 += ele_og * ele_gamma * (ele_x - mean) * invstd_eps * invstd_eps;
      }
    }
    for (; l < nchannel; ++l) {
      AType ele_og    = static_cast<AType>(out_grad[bid * nchannel + l]);
      AType ele_x     = static_cast<AType>(in_data[bid * nchannel + l]);
      AType ele_gamma = static_cast<AType>(gamma[l]);
      sum_val0 += ele_og * ele_gamma * invstd_eps;
      sum_val1 += ele_og * ele_gamma * (ele_x - mean) * invstd_eps * invstd_eps;
    }
    // Intra-warp reduction (all-reduce)
    for (int mask = blockDim.x / 2; mask > 0; mask >>= 1) {
      sum_val0 += warp_shfl_xor(sum_val0, mask);
      sum_val1 += warp_shfl_xor(sum_val1, mask);
    }
    // Inter-warp reduction (all-reduce)
    if (blockDim.y > 1) {
      AType* sum_val0_buf = reinterpret_cast<AType*>(buf);
      AType* sum_val1_buf =
          reinterpret_cast<AType*>(buf + blockDim.y / 2 * blockDim.x * sizeof(AType));
      for (int offset = blockDim.y / 2; offset > 0; offset >>= 1) {
        if (threadIdx.y >= offset && threadIdx.y < 2 * offset) {
          const int idx     = (threadIdx.y - offset) * blockDim.x + threadIdx.x;
          sum_val0_buf[idx] = sum_val0;
          sum_val1_buf[idx] = sum_val1;
        }
        __syncthreads();
        if (threadIdx.y < offset) {
          const int idx = threadIdx.y * blockDim.x + threadIdx.x;
          sum_val0 += sum_val0_buf[idx];
          sum_val1 += sum_val1_buf[idx];
        }
        __syncthreads();
      }
      if (threadIdx.y == 0) {
        sum_val0_buf[threadIdx.x] = sum_val0;
        sum_val1_buf[threadIdx.x] = sum_val1;
      }
      __syncthreads();
      sum_val0 = sum_val0_buf[threadIdx.x];
      sum_val1 = sum_val1_buf[threadIdx.x];
    }
    sum_val0 /= nchannel;
    sum_val1 /= nchannel;
    // 2. Calculate the gradient as
    //      out_grad * gamma / std - sum_val0 - (x - mean) / std * sum_val1
    for (int l = tid; l < nchannel; l += nthread) {
      AType ele_out_grad = static_cast<AType>(out_grad[bid * nchannel + l]);
      AType ele_x        = static_cast<AType>(in_data[bid * nchannel + l]);
      AType ele_gamma    = static_cast<AType>(gamma[l]);
      if (data_addto) {
        data_grad[bid * nchannel + l] +=
            static_cast<DType>(ele_out_grad * ele_gamma * invstd_eps - sum_val0 -
                               (ele_x - mean) * invstd_eps * sum_val1);
      } else {
        data_grad[bid * nchannel + l] =
            static_cast<DType>(ele_out_grad * ele_gamma * invstd_eps - sum_val0 -
                               (ele_x - mean) * invstd_eps * sum_val1);
      }
    }
  }
}

void GetGammaBetaGradKernelParams(const int nbatch,
                                  const int nchannel,
                                  dim3* part_grad_block_dim,
                                  dim3* part_grad_grid_dim,
                                  dim3* gb_block_dim,
                                  dim3* gb_grid_dim,
                                  int* npart) {
  *npart               = 16;
  *part_grad_block_dim = dim3(32, 16);
  *part_grad_grid_dim  = dim3((nchannel + 32 - 1) / 32, *npart);
  *gb_block_dim        = dim3(32, *npart);
  *gb_grid_dim         = dim3((nchannel + 32 - 1) / 32);
  CheckLaunchParam(*part_grad_grid_dim, *part_grad_block_dim);
  CheckLaunchParam(*gb_grid_dim, *gb_block_dim);
}

template <bool safe_acc = false>
void LayerNormGradGPUContig(const LayerNormParam param,
                            const OpContext& ctx,
                            const std::vector<TBlob>& inputs,
                            const std::vector<OpReqType>& req,
                            const std::vector<TBlob>& outputs) {
  using namespace mshadow;
#if MXNET_USE_ONEDNN == 1
  CHECK_EQ(inputs.size(), 6U);  // additional beta tensor
#else
  CHECK_EQ(inputs.size(), 5U);
#endif
  const TBlob out_grad   = inputs[0];
  const TBlob in_data    = inputs[1];
  const TBlob gamma      = inputs[2];
  const TBlob mean_data  = inputs[3];
  const TBlob std_data   = inputs[4];
  const TBlob data_grad  = outputs[0];
  const TBlob gamma_grad = outputs[1];
  const TBlob beta_grad  = outputs[2];

  // Make sure the inputs are contiguous
  CHECK_EQ(out_grad.CheckContiguous(), true);
  CHECK_EQ(in_data.CheckContiguous(), true);
  CHECK_EQ(gamma.CheckContiguous(), true);
  CHECK_EQ(mean_data.CheckContiguous(), true);
  CHECK_EQ(std_data.CheckContiguous(), true);
  int nbatch         = in_data.shape_.ProdShape(0, in_data.ndim() - 1);
  int nchannel       = in_data.shape_[in_data.ndim() - 1];
  int data_grad_req  = req[0];
  int gamma_grad_req = req[1];
  int beta_grad_req  = req[2];
  CHECK_NE(data_grad_req, kWriteInplace);
  CHECK_NE(gamma_grad_req, kWriteInplace);
  CHECK_NE(beta_grad_req, kWriteInplace);
  Stream<gpu>* s      = ctx.get_stream<gpu>();
  hipStream_t stream = Stream<gpu>::GetStream(s);

  // Calculate the gradient for gamma/beta
  CHECK_EQ(gamma_grad.CheckContiguous(), true);
  CHECK_EQ(beta_grad.CheckContiguous(), true);
  dim3 part_grad_block_dim, part_grad_grid_dim, gb_block_dim, gb_grid_dim;
  int npart;
  GetGammaBetaGradKernelParams(nbatch,
                               nchannel,
                               &part_grad_block_dim,
                               &part_grad_grid_dim,
                               &gb_block_dim,
                               &gb_grid_dim,
                               &npart);
  if (gamma_grad_req != kNullOp || beta_grad_req != kNullOp) {
    MXNET_REAL_ACC_TYPE_SWITCH(in_data.type_flag_, DType, AccType, {
      typedef typename std::conditional<safe_acc, AccType, DType>::type AType;
      Tensor<gpu, 1, AType> workspace =
          ctx.requested[0].get_space_typed<gpu, 1, AType>(Shape1(2 * npart * nchannel), s);
      AType* part_gamma_grad_ptr = workspace.dptr_;
      AType* part_beta_grad_ptr  = workspace.dptr_ + npart * nchannel;
      const int nshared_K1 =
          2 * (part_grad_block_dim.x + 1) * part_grad_block_dim.y * sizeof(AType);
      const int nshared_K2  = 2 * gb_block_dim.x * gb_block_dim.y * sizeof(AType);
      DType* gamma_grad_ptr = (gamma_grad_req != kNullOp) ? gamma_grad.dptr<DType>() : nullptr;
      DType* beta_grad_ptr  = (beta_grad_req != kNullOp) ? beta_grad.dptr<DType>() : nullptr;
      LayerNormFusedBackwardKernel_PartGammaBeta<<<part_grad_grid_dim,
                                                   part_grad_block_dim,
                                                   nshared_K1,
                                                   stream>>>(nbatch,
                                                             nchannel,
                                                             in_data.dptr<DType>(),
                                                             out_grad.dptr<DType>(),
                                                             mean_data.dptr<DType>(),
                                                             std_data.dptr<DType>(),
                                                             part_gamma_grad_ptr,
                                                             part_beta_grad_ptr);
      MSHADOW_CUDA_POST_KERNEL_CHECK(LayerNormFusedBackwardKernel_PartGammaBeta);
      if (gamma_grad_req == kAddTo && beta_grad_req != kAddTo) {
        LayerNormFusedBackwardKernel_GammaBeta<true, false>
            <<<gb_grid_dim, gb_block_dim, nshared_K2, stream>>>(nbatch,
                                                                nchannel,
                                                                npart,
                                                                part_gamma_grad_ptr,
                                                                part_beta_grad_ptr,
                                                                gamma_grad_ptr,
                                                                beta_grad_ptr);
      } else if (gamma_grad_req != kAddTo && beta_grad_req == kAddTo) {
        LayerNormFusedBackwardKernel_GammaBeta<false, true>
            <<<gb_grid_dim, gb_block_dim, nshared_K2, stream>>>(nbatch,
                                                                nchannel,
                                                                npart,
                                                                part_gamma_grad_ptr,
                                                                part_beta_grad_ptr,
                                                                gamma_grad_ptr,
                                                                beta_grad_ptr);
      } else if (gamma_grad_req == kAddTo && beta_grad_req == kAddTo) {
        LayerNormFusedBackwardKernel_GammaBeta<true, true>
            <<<gb_grid_dim, gb_block_dim, nshared_K2, stream>>>(nbatch,
                                                                nchannel,
                                                                npart,
                                                                part_gamma_grad_ptr,
                                                                part_beta_grad_ptr,
                                                                gamma_grad_ptr,
                                                                beta_grad_ptr);
      } else {
        LayerNormFusedBackwardKernel_GammaBeta<false, false>
            <<<gb_grid_dim, gb_block_dim, nshared_K2, stream>>>(nbatch,
                                                                nchannel,
                                                                npart,
                                                                part_gamma_grad_ptr,
                                                                part_beta_grad_ptr,
                                                                gamma_grad_ptr,
                                                                beta_grad_ptr);
      }
    });
    MSHADOW_CUDA_POST_KERNEL_CHECK(LayerNormFusedBackwardKernel_GammaBeta);
  }

  // Calculate the gradient for data
  CHECK_EQ(data_grad.CheckContiguous(), true);
  int ngrid_x = (nbatch > kMaxGridDim) ? (nbatch + kBaseGridNum - 1) / kBaseGridNum : nbatch;
  int ngrid_y = (nbatch > kMaxGridDim) ? kBaseGridNum : 1;
  const dim3 data_grid_dim(ngrid_x, ngrid_y);
  int nthread_y;
  if (nchannel <= 32) {
    nthread_y = 1;
  } else if (nchannel <= 128) {
    nthread_y = 2;
  } else if (nchannel <= 512) {
    nthread_y = 4;
  } else {
    nthread_y = 8;
  }
  const dim3 data_block_dim(32, nthread_y);
  const int LOAD_UNROLL = 4;
  if (data_grad_req != kNullOp) {
    MXNET_REAL_ACC_TYPE_SWITCH(in_data.type_flag_, DType, AccType, {
      typedef typename std::conditional<safe_acc, AccType, DType>::type AType;
      int nshared = data_block_dim.y > 1 ? data_block_dim.y * data_block_dim.x * sizeof(AType) : 0;
      CheckLaunchParam(data_grid_dim, data_block_dim);
      if (data_grad_req == kAddTo) {
        LayerNormFusedBackwardKernel_Data<LOAD_UNROLL, true, AType>
            <<<data_grid_dim, data_block_dim, nshared, stream>>>(nbatch,
                                                                 nchannel,
                                                                 in_data.dptr<DType>(),
                                                                 out_grad.dptr<DType>(),
                                                                 mean_data.dptr<DType>(),
                                                                 std_data.dptr<DType>(),
                                                                 gamma.dptr<DType>(),
                                                                 data_grad.dptr<DType>());
      } else {
        LayerNormFusedBackwardKernel_Data<LOAD_UNROLL, false, AType>
            <<<data_grid_dim, data_block_dim, nshared, stream>>>(nbatch,
                                                                 nchannel,
                                                                 in_data.dptr<DType>(),
                                                                 out_grad.dptr<DType>(),
                                                                 mean_data.dptr<DType>(),
                                                                 std_data.dptr<DType>(),
                                                                 gamma.dptr<DType>(),
                                                                 data_grad.dptr<DType>());
      }
    });
    MSHADOW_CUDA_POST_KERNEL_CHECK(LayerNormFusedBackwardKernel_Data);
  }
}

template <>
void LayerNormGradCompute<gpu>(const nnvm::NodeAttrs& attrs,
                               const OpContext& ctx,
                               const std::vector<TBlob>& inputs,
                               const std::vector<OpReqType>& req,
                               const std::vector<TBlob>& outputs) {
  const LayerNormParam& param = nnvm::get<LayerNormParam>(attrs.parsed);
  int axis                    = param.axis;
  if (axis < 0) {
    axis += static_cast<int>(inputs[0].ndim());
  }
  CHECK(axis >= 0 && axis < inputs[0].ndim()) << "Channel axis out of range: " << param.axis;
  if (axis == inputs[0].ndim() - 1) {
    // Use the accelerated CUDA kernels
    bool safe_acc = dmlc::GetEnv("MXNET_SAFE_ACCUMULATION", true);
    if (safe_acc) {
      return LayerNormGradGPUContig<true>(param, ctx, inputs, req, outputs);
    } else {
      return LayerNormGradGPUContig<false>(param, ctx, inputs, req, outputs);
    }
  }
  return LayerNormGradComputeGeneral<gpu>(attrs, ctx, inputs, req, outputs);
}

NNVM_REGISTER_OP(LayerNorm).set_attr<FCompute>("FCompute<gpu>", LayerNormCompute<gpu>);

NNVM_REGISTER_OP(_backward_LayerNorm)
    .set_attr<FCompute>("FCompute<gpu>", LayerNormGradCompute<gpu>);

}  // namespace op
}  // namespace mxnet
