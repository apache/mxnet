/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file softmax.cu
 * \brief GPU Implementation of softmax
 */
#include "./softmax-inl.h"
#include "../tensor/elemwise_unary_op.h"

namespace mxnet {
namespace op {

NNVM_REGISTER_OP(softmax)
.set_attr<FCompute>("FCompute<gpu>", SoftmaxCompute<gpu, mxnet_op::softmax_fwd>);

NNVM_REGISTER_OP(_backward_softmax)
.set_attr<FCompute>("FCompute<gpu>", SoftmaxGradCompute<gpu, op::mshadow_op::mul,
                                                        mxnet_op::softmax_bwd>);

}  // namespace op
}  // namespace mxnet
