#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright [2016] <Contributors>
 * \file Correation.cu
 * \brief  Correlation operator
 * \author Xu Dong
*/
#include "./correlation-inl.h"
#include <mshadow/tensor.h>
#include <mshadow/cuda/reduce.cuh>
#include <algorithm>
#include <vector>
#include "./mxnet_op.h"

#define ROUND_OFF 50000
#define WARPS_PER_BLOCK 1
#define THREADS_PER_WARP 32
#define CORRELATION_CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    CHECK_EQ(error, hipSuccess) << " " << hipGetErrorString(error); \
  } while (0)

namespace mshadow {
namespace cuda {
// == Correlation Kernel
template <typename Dtype>
__global__ void CorrelateData(const int nthreads, int num, int topwidth,
  int topheight, int topchannels, int topcount,
  int max_displacement, int neighborhood_grid_radius,
  int neighborhood_grid_width, int kernel_radius, int kernel_size, int stride1, int stride2,
  int bottomwidth, int bottomheight, int bottomchannels,
  const Dtype *bottom0, const Dtype *bottom1, Dtype *top) {
  extern __shared__ char patch_data_char[];
  Dtype *patch_data = reinterpret_cast<Dtype *>(patch_data_char);
  //  First (upper left) position of kernel upper-left corner
  //  in current center position of neighborhood in image 1
  int x1 = blockIdx.x * stride1 + max_displacement;
  int y1 = blockIdx.y * stride1 + max_displacement;
  int item = blockIdx.z;
  int ch_off = threadIdx.x;
  //  Load 3D patch into shared shared memory
  for (int j = 0; j < kernel_size; j++) {  //  HEIGHT
    for (int i = 0; i < kernel_size; i++) {  //  WIDTH
      int ji_off = ((j * kernel_size) + i) * bottomchannels;
      for (int ch = ch_off; ch < bottomchannels; ch += (THREADS_PER_WARP * WARPS_PER_BLOCK))  {
          //  CHANNELS
          int idx1 = ((item * bottomheight + y1+j) * bottomwidth + x1+i) * bottomchannels + ch;
          int idxPatchData = ji_off + ch;
          patch_data[idxPatchData] = bottom0[idx1];
      }
    }
  }
  __syncthreads();
  __shared__ Dtype sum[THREADS_PER_WARP * WARPS_PER_BLOCK];
  //  Compute correlation
  for (int top_channel = 0; top_channel < topchannels; top_channel++) {
    sum[ch_off] = 0;
    int s2o = (top_channel % neighborhood_grid_width - neighborhood_grid_radius) * stride2;
    int s2p = (top_channel / neighborhood_grid_width - neighborhood_grid_radius) * stride2;
    for (int j = 0; j < kernel_size; j++) {  //  HEIGHT
      for (int i = 0; i < kernel_size; i++) {  //  WIDTH
        int ji_off = ((j * kernel_size) + i) * bottomchannels;
        for (int ch = ch_off; ch < bottomchannels; ch += (THREADS_PER_WARP * WARPS_PER_BLOCK)) {
          //  CHANNELS
          int x2 = x1 + s2o;
          int y2 = y1 + s2p;
          int idxPatchData = ji_off + ch;
          int idx2 = ((item * bottomheight + y2 + j) * bottomwidth + x2 + i) * bottomchannels + ch;
          sum[ch_off] += patch_data[idxPatchData] * bottom1[idx2];
        }
      }
    }
    __syncthreads();
    if (ch_off == 0) {
        Dtype total_sum = 0;
        for (int idx = 0; idx < THREADS_PER_WARP * WARPS_PER_BLOCK; idx++) {
            total_sum += sum[idx];
        }
        const int sumelems = kernel_size * kernel_size * bottomchannels;
        const int index = ((top_channel * topheight + blockIdx.y) * topwidth) + blockIdx.x;
        top[index + item*topcount] = total_sum / static_cast<float>(sumelems);
    }  //  Aggregate result of  different threads
  }
}
//  == Correlation Backward Pass Kernel (For data1)
template <typename Dtype>
__global__ void CorrelateDataBackward0(const int nthreads, int num, int item,
  int topwidth, int topheight, int topchannels,
  int max_displacement, int neighborhood_grid_radius,
  int neighborhood_grid_width, int kernel_radius, int stride1, int stride2,
  int bottomwidth, int bottomheight, int pbottomwidth, int pbottomheight,
  int bottomchannels, int bottomcount, int pad_size,
  Dtype *bottom0diff, const Dtype *bottom1, const Dtype *topdiff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index % bottomchannels;  //  channels
    int l = (index / bottomchannels) % bottomwidth + pad_size;  //  w-pos
    int m = (index / bottomchannels / bottomwidth) % bottomheight + pad_size;  //  h-pos
    //  Get X,Y ranges and clamp
    //  round_off is a trick to enable integer division with ceil, even for negative numbers
    //  We use a large offset, for the inner part not to become negative.
    const int round_off = ROUND_OFF;
    const int round_off_s1 = stride1 * round_off;
    //  We add round_off before_s1 the int division and subtract round_off after it,
    //  to ensure the formula matches ceil behavior:
    int xmin = (l - 2*kernel_radius - max_displacement + round_off_s1 - 1)\
     / stride1 + 1 - round_off;  //  ceil (l - 2*kernel_radius - max_displacement) / stride1
    int ymin = (m - 2*kernel_radius - max_displacement + round_off_s1 - 1)\
     / stride1 + 1 - round_off;  //  ceil (l - 2*kernel_radius - max_displacement) / stride1
    //  Same here:
    int xmax = (l - max_displacement + round_off_s1) / stride1 - round_off;
    //  floor (l - max_displacement) / stride1
    int ymax = (m - max_displacement + round_off_s1) / stride1 - round_off;
    //  floor (m - max_displacement) / stride1
    Dtype sum = 0;
    if (xmax >= 0 && ymax >= 0 && (xmin <= topwidth-1) && (ymin <= topheight-1)) {
        xmin = max(0, xmin);
        xmax = min(topwidth-1, xmax);
        ymin = max(0, ymin);
        ymax = min(topheight-1, ymax);
        for (int p = -neighborhood_grid_radius; p <= neighborhood_grid_radius; p++) {
          for (int o = -neighborhood_grid_radius; o <= neighborhood_grid_radius; o++) {
            //  Get bottom1 data:
            int s2o = stride2 * o;
            int s2p = stride2 * p;
            int idxbot1 = ((item * pbottomheight + (m + s2p)) * pbottomwidth + (l + s2o))\
             * bottomchannels + n;
            Dtype bot1tmp = bottom1[idxbot1];  // bottom1[l+s2o,m+s2p,n]
            //  Index offset for topdiff in following loops:
            int op = (p+neighborhood_grid_radius) * neighborhood_grid_width\
             + (o + neighborhood_grid_radius);  //  index [o,p]
            int idxopoffset = (item * topchannels + op);
            for (int y = ymin; y <= ymax; y++) {
              for (int x = xmin; x <= xmax; x++) {
                int idxtopdiff = (idxopoffset * topheight + y) * topwidth + x;  //  topdiff[x,y,o,p]
                sum += topdiff[idxtopdiff] * bot1tmp;
              }
            }
          }
        }
    }
    const int sumelems = (kernel_radius * 2 + 1) * (kernel_radius * 2+1) * bottomchannels;
    const int bot0index = ((n * bottomheight) + (m-pad_size)) * bottomwidth + (l-pad_size);
    bottom0diff[bot0index + item * bottomcount] = sum / static_cast<float>(sumelems);
  }
}
// == Correlation Backward Pass Kernel (For Blob 1)
template <typename Dtype>
__global__ void CorrelateDataBackward1(const int nthreads,
  int num, int item, int topwidth, int topheight, int topchannels,
  int max_displacement, int neighborhood_grid_radius,
  int neighborhood_grid_width, int kernel_radius, int stride1, int stride2,
  int bottomwidth, int bottomheight, int pbottomwidth, int pbottomheight,
  int bottomchannels, int bottomcount, int pad_size,
  const Dtype *bottom0, Dtype *bottom1diff, const Dtype *topdiff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    //  int l = index % bottomwidth + pad_size; //w-pos
    //  int m = (index / bottomwidth) % bottomheight + pad_size; //  h-pos
    //  int n = (index / bottomwidth / bottomheight) % bottomchannels; //  channels
    int n = index % bottomchannels;  //  channels
    int l = (index / bottomchannels) % bottomwidth + pad_size;  //  w-pos
    int m = (index / bottomchannels / bottomwidth) % bottomheight + pad_size;  //  h-pos
    //  round_off is a trick to enable integer division with ceil, even for negative numbers
    //  We use a large offset, for the inner part not to become negative.
    const int round_off = ROUND_OFF;
    const int round_off_s1 = stride1 * round_off;
    Dtype sum = 0;
    for (int p = -neighborhood_grid_radius; p <= neighborhood_grid_radius; p++) {
      for (int o = -neighborhood_grid_radius; o <= neighborhood_grid_radius; o++) {
        int s2o = stride2 * o;
        int s2p = stride2 * p;
        //  Get X,Y ranges and clamp
        //  We add round_off before_s1 the int division and subtract round_off after it,
        //  to ensure the formula matches ceil behavior:
        int xmin = (l - 2*kernel_radius - max_displacement - s2o + round_off_s1 - 1)\
         / stride1 + 1 - round_off;
         // ceil (l - 2*kernel_radius - max_displacement - s2o) / stride1
        int ymin = (m - 2*kernel_radius - max_displacement - s2p + round_off_s1 - 1)\
         / stride1 + 1 - round_off;
        // ceil (l - 2*kernel_radius - max_displacement - s2o) / stride1
        //  Same here:
        int xmax = (l - max_displacement - s2o + round_off_s1) / stride1 - round_off;
        //  floor (l - max_displacement - s2o) / stride1
        int ymax = (m - max_displacement - s2p + round_off_s1) / stride1 - round_off;
        //  floor (m - max_displacement - s2p) / stride1
        if (xmax >= 0 && ymax >= 0 && (xmin <= topwidth - 1) && (ymin <= topheight - 1)) {
            xmin = max(0, xmin);
            xmax = min(topwidth-1, xmax);
            ymin = max(0, ymin);
            ymax = min(topheight-1, ymax);
            //  Get bottom0 data:
            int idxbot0 = ((item * pbottomheight + (m - s2p)) \
            * pbottomwidth + (l - s2o)) * bottomchannels + n;
            Dtype bot0tmp = bottom0[idxbot0];  //  bottom1[l+s2o,m+s2p,n]
            //  Index offset for topdiff in following loops:
            int op = (p+neighborhood_grid_radius) * \
            neighborhood_grid_width + (o+neighborhood_grid_radius);  //  index [o,p]
            int idxOpOffset = (item * topchannels + op);
            for (int y = ymin; y <= ymax; y++) {
              for (int x = xmin; x <= xmax; x++) {
                int idxtopdiff = (idxOpOffset * topheight + y)\
                 * topwidth + x;  //  topdiff[x,y,o,p]
                sum += topdiff[idxtopdiff] * bot0tmp;
              }
            }
        }
      }
    }
    const int sumelems = (kernel_radius*2+1)*(kernel_radius*2+1)*bottomchannels;
    const int bot1index = ((n * bottomheight) + (m - pad_size)) * bottomwidth + (l - pad_size);
    bottom1diff[bot1index + item * bottomcount] = sum / static_cast<float>(sumelems);
  }
}
// == Correlation Kernel Subtraction
template <typename Dtype>
__global__ void CorrelateDataSubtract(const int nthreads, int num, int item,
  int topwidth, int topheight, int topchannels, int topcount,
  int max_displacement, int neighborhood_grid_radius,
  int neighborhood_grid_width, int kernel_radius, int stride1, int stride2,
  int bottomwidth, int bottomheight, int bottomchannels,
  const Dtype *bottom0, const Dtype *bottom1, Dtype *top) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int x = index % topwidth;  //  w-pos
    int y = (index / topwidth) % topheight;  //  h-pos
    int c = (index / topwidth / topheight) % topchannels;  //  channels
    //  Offset of patch in image 2
    int s2o = (c % neighborhood_grid_width - neighborhood_grid_radius) * stride2;
    int s2p = (c / neighborhood_grid_width - neighborhood_grid_radius) * stride2;
    //  First (upper left) position of kernel center in current neighborhood in image 1
    int x1 = x*stride1 + kernel_radius + max_displacement;
    int y1 = y*stride1 + kernel_radius + max_displacement;
    //  Iterate through 3D patch
    Dtype sum = 0;
    for (int j = -kernel_radius; j <= kernel_radius; j++) {  //  HEIGHT
      for (int i = -kernel_radius; i <= kernel_radius; i++) {  //  WIDTH
        for (int l = 0; l < bottomchannels; l++) {  //  CHANNELS
          //  Calculate position in image 2
          int x2 = x1 + s2o;
          int y2 = y1 + s2p;
          //  Indices in bottom data: (CH=l,W=x2,H=y2,N)
          int idx1 = ((item * bottomheight + y1 + j) * bottomwidth + x1 + i) \
          * bottomchannels + l;
          int idx2 = ((item * bottomheight + y2 + j) * bottomwidth + x2 + i) \
          * bottomchannels + l;
          //  Do the correlation:
          sum += fabsf(bottom0[idx1] - bottom1[idx2]);
        }
      }
    }
    const int sumelems = (kernel_radius * 2 + 1) * (kernel_radius * 2 + 1) * bottomchannels;
    top[index + item * topcount] = sum / static_cast<float>(sumelems);
  }
}
//  == Correlation Backward Pass Kernel (For Blob 0)
template <typename Dtype>
__global__ void CorrelateDataBackward0Subtract(const int nthreads, int num,
  int item, int topwidth, int topheight, int topchannels,
  int max_displacement, int neighborhood_grid_radius,
  int neighborhood_grid_width, int kernel_radius,
  int stride1, int stride2, int bottomwidth, int bottomheight,
  int pbottomwidth, int pbottomheight,
  int bottomchannels, int bottomcount, int pad_size,
  Dtype *bottom0diff, const Dtype *bottom0, const Dtype *bottom1, const Dtype *topdiff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index % bottomchannels;  //  channels
    int l = (index / bottomchannels) % bottomwidth + pad_size;  //  w-pos
    int m = (index / bottomchannels / bottomwidth) % bottomheight + pad_size;  //  h-pos
    //  Get X,Y ranges and clamp
    //  round_off is a trick to enable integer division with ceil, even for negative numbers
    //  We use a large offset, for the inner part not to become negative.
    const int round_off = ROUND_OFF;
    const int round_off_s1 = stride1 * round_off;
    int idxbot0 = ((item * pbottomheight + m) * pbottomwidth + l)\
             * bottomchannels + n;
    //  We add round_off before_s1 the int division and subtract round_off after it,
    //  to ensure the formula matches ceil behavior:
    int xmin = (l - 2*kernel_radius - max_displacement + round_off_s1 - 1)\
     / stride1 + 1 - round_off;  //  ceil (l - 2*kernel_radius - max_displacement) / stride1
    int ymin = (m - 2*kernel_radius - max_displacement + round_off_s1 - 1)\
     / stride1 + 1 - round_off;  //  ceil (l - 2*kernel_radius - max_displacement) / stride1
    //  Same here:
    int xmax = (l - max_displacement + round_off_s1) / stride1 - round_off;
    //  floor (l - max_displacement) / stride1
    int ymax = (m - max_displacement + round_off_s1) / stride1 - round_off;
    //  floor (m - max_displacement) / stride1
    Dtype sum = 0;
    if (xmax >= 0 && ymax >= 0 && (xmin <= topwidth-1) && (ymin <= topheight-1)) {
        xmin = max(0, xmin);
        xmax = min(topwidth-1, xmax);
        ymin = max(0, ymin);
        ymax = min(topheight-1, ymax);
        for (int p = -neighborhood_grid_radius; p <= neighborhood_grid_radius; p++) {
          for (int o = -neighborhood_grid_radius; o <= neighborhood_grid_radius; o++) {
            //  Get bottom1 data:
            int s2o = stride2 * o;
            int s2p = stride2 * p;
            int idxbot1 = ((item * pbottomheight + (m+s2p)) * pbottomwidth\
             + (l+s2o)) * bottomchannels + n;
            Dtype bot0tmp = bottom0[idxbot0];
            Dtype bot1tmp = bottom1[idxbot1];
            Dtype sign = (bot0tmp >= bot1tmp) ? Dtype(1.0) : Dtype(-1.0);
            //  Index offset for topdiff in following loops:
            int op = (p+neighborhood_grid_radius) * neighborhood_grid_width\
             + (o + neighborhood_grid_radius);  //  index [o,p]
            int idxopoffset = (item * topchannels + op);
            for (int y = ymin; y <= ymax; y++) {
              for (int x = xmin; x <= xmax; x++) {
                int idxtopdiff = (idxopoffset * topheight + y) * topwidth + x;  //  topdiff[x,y,o,p]
                sum += topdiff[idxtopdiff] * sign;
              }
            }
          }
        }
    }
    const int sumelems = (kernel_radius * 2 + 1) * (kernel_radius * 2+1) * bottomchannels;
    const int bot0index = ((n * bottomheight) + (m-pad_size)) * bottomwidth + (l-pad_size);
    bottom0diff[bot0index + item * bottomcount] = sum / static_cast<float>(sumelems);
  }
}
//  == Correlation Backward Pass Kernel (For Blob 1)
template <typename Dtype>
__global__ void CorrelateDataBackward1Subtract(const int nthreads, int num,
  int item, int topwidth, int topheight, int topchannels,
  int max_displacement, int neighborhood_grid_radius,
  int neighborhood_grid_width, int kernel_radius,
  int stride1, int stride2, int bottomwidth, int bottomheight,
  int pbottomwidth, int pbottomheight, int bottomchannels,
  int bottomcount, int pad_size, const Dtype *bottom0,
  const Dtype *bottom1, Dtype *bottom1diff, const Dtype *topdiff) {
    CUDA_KERNEL_LOOP(index, nthreads) {
    //  int l = index % bottomwidth + pad_size; //w-pos
    //  int m = (index / bottomwidth) % bottomheight + pad_size; //  h-pos
    //  int n = (index / bottomwidth / bottomheight) % bottomchannels; //  channels
    int n = index % bottomchannels;  //  channels
    int l = (index / bottomchannels) % bottomwidth + pad_size;  //  w-pos
    int m = (index / bottomchannels / bottomwidth) % bottomheight + pad_size;  //  h-pos
    //  round_off is a trick to enable integer division with ceil, even for negative numbers
    //  We use a large offset, for the inner part not to become negative.
    const int round_off = ROUND_OFF;
    const int round_off_s1 = stride1 * round_off;
    Dtype sum = 0;
    int idxbot1 = ((item * pbottomheight + m) * pbottomwidth + l)\
             * bottomchannels + n;
    for (int p = -neighborhood_grid_radius; p <= neighborhood_grid_radius; p++) {
      for (int o = -neighborhood_grid_radius; o <= neighborhood_grid_radius; o++) {
        int s2o = stride2 * o;
        int s2p = stride2 * p;
        //  Get X,Y ranges and clamp
        //  We add round_off before_s1 the int division and subtract round_off after it,
        //  to ensure the formula matches ceil behavior:
        int xmin = (l - 2*kernel_radius - max_displacement - s2o + round_off_s1 - 1)\
         / stride1 + 1 - round_off;
         // ceil (l - 2*kernel_radius - max_displacement - s2o) / stride1
        int ymin = (m - 2*kernel_radius - max_displacement - s2p + round_off_s1 - 1)\
         / stride1 + 1 - round_off;
        // ceil (l - 2*kernel_radius - max_displacement - s2o) / stride1
        //  Same here:
        int xmax = (l - max_displacement - s2o + round_off_s1) / stride1 - round_off;
        //  floor (l - max_displacement - s2o) / stride1
        int ymax = (m - max_displacement - s2p + round_off_s1) / stride1 - round_off;
        //  floor (m - max_displacement - s2p) / stride1
        if (xmax >= 0 && ymax >= 0 && (xmin <= topwidth - 1) && (ymin <= topheight - 1)) {
            xmin = max(0, xmin);
            xmax = min(topwidth-1, xmax);
            ymin = max(0, ymin);
            ymax = min(topheight-1, ymax);
            //  Get bottom0 data:
            int idxbot0 = ((item * pbottomheight + (m - s2p)) * pbottomwidth + (l - s2o))\
             * bottomchannels + n;
            //  bottom0[l+s2o,m+s2p,n]
            Dtype bot0tmp = bottom0[idxbot0];
            Dtype bot1tmp = bottom1[idxbot1];
            Dtype sign = (bot0tmp >= bot1tmp) ? Dtype(-1.0) : Dtype(1.0);
            //  Index offset for topdiff in following loops:
            int op = (p+neighborhood_grid_radius) * \
            neighborhood_grid_width + (o+neighborhood_grid_radius);  //  index [o,p]
            int idxOpOffset = (item * topchannels + op);
            for (int y = ymin; y <= ymax; y++) {
              for (int x = xmin; x <= xmax; x++) {
                int idxtopdiff = (idxOpOffset * topheight + y)\
                 * topwidth + x;  //  topdiff[x,y,o,p]
                sum += topdiff[idxtopdiff] * sign;
              }
            }
        }
      }
    }
    const int sumelems = (kernel_radius*2+1)*(kernel_radius*2+1)*bottomchannels;
    const int bot1index = ((n * bottomheight) + (m - pad_size)) * bottomwidth + (l - pad_size);
    bottom1diff[bot1index + item * bottomcount] = sum / static_cast<float>(sumelems);
  }
}
//  == Forward
//  == Dimension rearrangement Kernel
template <typename Dtype>
__global__ void blob_rearrange_kernel2(const Dtype* in, Dtype* out, int num,
int channels, int width, int height, int widthheight, int padding, int pwidthheight) {
    //  change shape from [batchsize,channel,y,x] to [batchsize,y,x,channel]
    int xy = blockIdx.x * blockDim.x + threadIdx.x;
    if (xy >= widthheight )
        return;
    int ch = blockIdx.y;
    int n  = blockIdx.z;
    Dtype value = in[(n * channels + ch) * widthheight + xy];
    __syncthreads();
    int xpad  = (xy % width + padding);
    int ypad  = (xy / width + padding);
    int xypad = ypad * (width + 2 * padding) + xpad;
    out[(n * pwidthheight + xypad) * channels + ch] = value;
}
template <typename Dtype>
void Forward_gpu(
      const Tensor<gpu, 4, Dtype> &out,
      const Tensor<gpu, 4, Dtype> &data1,
      const Tensor<gpu, 4, Dtype> &data2,
      const Tensor<gpu, 4, Dtype> &tmp1,
      const Tensor<gpu, 4, Dtype> &tmp2,
      int top_channels_, int top_height_, int top_width_, int pad_size_,
      bool is_multiply, int max_displacement_, int kernel_size_,
      int neighborhood_grid_radius_, int neighborhood_grid_width_,
      int  kernel_radius_, int stride1_, int stride2_, hipStream_t stream,
      hipStream_t stream_tmp1, hipStream_t stream_tmp2) {
    const Dtype *bottom_data1 = data1.dptr_;
    const Dtype *bottom_data2 = data2.dptr_;
    Dtype *rbot1 = tmp1.dptr_;
    Dtype *rbot2 = tmp2.dptr_;
    Dtype *top = out.dptr_;
    const int bnum = data1.size(0);
    const int bchannels = data1.size(1);
    const int bheight = data1.size(2);
    const int bwidth = data1.size(3);
    const int bwidthheight = bwidth * bheight;
    const int topcount = top_width_ * top_height_ * top_channels_;
    dim3 threadsPerBlock(THREADS_PER_WARP * WARPS_PER_BLOCK);
    int threads_per_block = 16;
    dim3 totalBlocksRearr((bwidthheight - 1) / threads_per_block + 1, bchannels, bnum);
    const int pwidthheight = (bwidth + 2 * pad_size_) * (bheight + 2 * pad_size_);
    blob_rearrange_kernel2<Dtype><<<totalBlocksRearr, threads_per_block, 0, stream_tmp1>>>
    (bottom_data1, rbot1, bnum, bchannels, bwidth, bheight, bwidthheight, pad_size_, pwidthheight);
    blob_rearrange_kernel2<Dtype><<<totalBlocksRearr, threads_per_block, 0, stream_tmp2>>>
    (bottom_data2, rbot2, bnum, bchannels, bwidth, bheight, bwidthheight, pad_size_, pwidthheight);
    const int num = bnum;
    const int channels = bchannels;
    const int height = bheight + 2 * pad_size_;
    const int width = bwidth + 2 * pad_size_;
    const int shared_memory_per_block = (kernel_size_ * kernel_size_) * bchannels;
    if (is_multiply == true) {
        //  CorrelationLayer
        int topThreadCount = topcount;
        dim3 totalBlocksCorr(top_width_, top_height_, num);
        CorrelateData<Dtype><<<totalBlocksCorr, threadsPerBlock,
        shared_memory_per_block * sizeof(Dtype), stream>>>(
            topThreadCount,
            num, top_width_, top_height_, top_channels_, topcount,
            max_displacement_, neighborhood_grid_radius_,
            neighborhood_grid_width_, kernel_radius_, kernel_size_,
            stride1_, stride2_,
            width, height, channels,
            rbot1, rbot2, top);
        CORRELATION_CUDA_CHECK(hipGetLastError());
    } else {
        //  CorrelationLayer
        for (int n = 0; n < num; n++) {
            int topThreadCount = topcount;
            const int gridSize = (topThreadCount + kMaxThreadsPerBlock - 1)\
             / kMaxThreadsPerBlock;
            CorrelateDataSubtract<Dtype><<<gridSize, kMaxThreadsPerBlock, 0, stream>>>(
                topThreadCount,
                num, n, top_width_, top_height_, top_channels_, topcount,
                max_displacement_, neighborhood_grid_radius_,
                neighborhood_grid_width_, kernel_radius_,
                stride1_, stride2_, width, height, channels, rbot1, rbot2, top);
         CORRELATION_CUDA_CHECK(hipGetLastError());
        }
    }
}
template <typename Dtype>
void Backward_gpu(
       const Tensor<gpu, 4, Dtype> &out_grad,
      const Tensor<gpu, 4, Dtype> &in_grad1,
      const Tensor<gpu, 4, Dtype> &in_grad2,
      const Tensor<gpu, 4, Dtype> &tmp1,
      const Tensor<gpu, 4, Dtype> &tmp2,
      int top_channels_, int top_height_,
      int top_width_, int pad_size_, bool is_multiply,
      int max_displacement_, int kernel_size_,
      int neighborhood_grid_radius_, int neighborhood_grid_width_,
      int  kernel_radius_, int stride1_, int stride2_,
      hipStream_t stream0, hipStream_t stream1,
      int num, int channels, int height, int width) {
    //  Get top diff, compute bottom diff
    const Dtype* top_diff = out_grad.dptr_;
    Dtype* bottom0_diff = in_grad1.dptr_;
    Dtype* bottom1_diff = in_grad2.dptr_;
    const Dtype* rbot1 = tmp1.dptr_;
    const Dtype* rbot2 = tmp2.dptr_;
    const int paddedheight = height + 2 * pad_size_;
    const int paddedwidth = width + 2 * pad_size_;
    const int bottomcount = channels * height * width;
    int botThreadCount = bottomcount;
    const int gridSize = (botThreadCount + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock;
    //  CorrelationLayerBackward
    if (is_multiply == true) {
        //  == Run kernel Backward 0
        dim3 totalBlocksBackward0(width, height, channels * num);  //  First dim is fastest
        const int buffer_size_backw0 = \
        (static_cast<int>(ceil(static_cast<float>(2 * kernel_radius_)\
         / static_cast<float>(stride1_))) + 1) * top_channels_;
        //  == Run kernel Backward 0
        for (int n = 0; n < num; n++) {
        CorrelateDataBackward0<Dtype><<<gridSize, kMaxThreadsPerBlock, 0, stream0>>>(
            botThreadCount,
            num, n, top_width_, top_height_, top_channels_,
            max_displacement_, neighborhood_grid_radius_, neighborhood_grid_width_, kernel_radius_,
            stride1_, stride2_,
            width, height, paddedwidth, paddedheight, channels, bottomcount, pad_size_,
            bottom0_diff, rbot2, top_diff);
        CORRELATION_CUDA_CHECK(hipGetLastError());
        }
        //  == Run kernel Backward 1
        for (int n = 0; n < num; n++) {
        CorrelateDataBackward1<Dtype><<<gridSize, kMaxThreadsPerBlock, 0, stream1>>>(
            botThreadCount,
            num, n, top_width_, top_height_, top_channels_,
            max_displacement_, neighborhood_grid_radius_, neighborhood_grid_width_, kernel_radius_,
            stride1_, stride2_,
            width, height, paddedwidth, paddedheight, channels, bottomcount, pad_size_,
            rbot1, bottom1_diff, top_diff);
       CORRELATION_CUDA_CHECK(hipGetLastError());
        }
    } else  {
        for (int n = 0; n < num; n++) {
        //  Bottom0:
        CorrelateDataBackward0Subtract<Dtype><<<gridSize, kMaxThreadsPerBlock, 0, stream0>>>(
            botThreadCount,
            num, n, top_width_, top_height_, top_channels_,
            max_displacement_, neighborhood_grid_radius_, neighborhood_grid_width_, kernel_radius_,
            stride1_, stride2_,
            width, height, paddedwidth, paddedheight, channels, bottomcount, pad_size_,
            bottom0_diff, rbot1, rbot2, top_diff);
        CORRELATION_CUDA_CHECK(hipGetLastError());
        }
        for (int n = 0; n < num; n++) {
        //  Bottom1:
        CorrelateDataBackward1Subtract<Dtype><<<gridSize, kMaxThreadsPerBlock, 0, stream1>>>(
            botThreadCount,
            num, n, top_width_, top_height_, top_channels_,
            max_displacement_, neighborhood_grid_radius_, neighborhood_grid_width_, kernel_radius_,
            stride1_, stride2_,
            width, height, paddedwidth, paddedheight, channels, bottomcount, pad_size_,
            rbot1, rbot2, bottom1_diff, top_diff);
        CORRELATION_CUDA_CHECK(hipGetLastError());
        }
    }
}
}  // namespace cuda
template<typename Dtype>
inline void CorrelationForward(const Tensor<gpu, 4, Dtype> &out,
                               const Tensor<gpu, 4, Dtype> &data1,
                               const Tensor<gpu, 4, Dtype> &data2,
                               const Tensor<gpu, 4, Dtype> &tmp1,
                               const Tensor<gpu, 4, Dtype> &tmp2,
                               int top_channels_, int top_height_,
                               int top_width_, int pad_size_, bool is_multiply,
                               int max_displacement_, int kernel_size_,
                               int neighborhood_grid_radius_, int neighborhood_grid_width_,
                               int kernel_radius_, int stride1_, int stride2_
                           ) {
  hipStream_t stream = Stream<gpu>::GetStream(out.stream_);
  hipStream_t stream_tmp1 = Stream<gpu>::GetStream(tmp1.stream_);
  hipStream_t stream_tmp2 = Stream<gpu>::GetStream(tmp2.stream_);
  cuda::Forward_gpu(out, data1, data2, tmp1, tmp2, top_channels_, top_height_,
                    top_width_, pad_size_, is_multiply, max_displacement_, kernel_size_,
                    neighborhood_grid_radius_, neighborhood_grid_width_, kernel_radius_,
                    stride1_, stride2_, stream, stream_tmp1, stream_tmp2);
}

template<typename Dtype>
inline void CorrelationBackward(const Tensor<gpu, 4, Dtype> &out_grad,
                            const Tensor<gpu, 4, Dtype> &in_grad1,
                            const Tensor<gpu, 4, Dtype> &in_grad2,
                            const Tensor<gpu, 4, Dtype> &tmp1,
                            const Tensor<gpu, 4, Dtype> &tmp2,
                            int top_channels_, int top_height_,
                            int top_width_, int pad_size_, bool is_multiply,
                            int max_displacement_, int kernel_size_,
                            int neighborhood_grid_radius_, int neighborhood_grid_width_,
                            int  kernel_radius_, int stride1_,
                            int stride2_, int num, int channels, int height, int width
                            ) {
  hipStream_t stream0 = Stream<gpu>::GetStream(in_grad1.stream_);
  hipStream_t stream1 = Stream<gpu>::GetStream(in_grad2.stream_);
  cuda::Backward_gpu(out_grad, in_grad1, in_grad2, tmp1, tmp2, top_channels_,
                      top_height_, top_width_, pad_size_, is_multiply,
                      max_displacement_, kernel_size_, neighborhood_grid_radius_,
                      neighborhood_grid_width_, kernel_radius_, stride1_, stride2_,
                      stream0, stream1, num, channels, height, width);
}
}  // namespace mshadow
namespace mxnet {
namespace op {
template<>
Operator* CreateOp<gpu>(CorrelationParam param, int dtype) {
  Operator* op = nullptr;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new CorrelationOp<gpu, DType>(param);
  });
  return op;
}
}  // namespace op
}  // namespace mxnet
