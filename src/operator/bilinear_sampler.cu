#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2017 by Contributors
 * \file bilinear_sampler.cu
 * \brief
 * \author Xu Dong
*/

#include "./bilinear_sampler-inl.h"
#include <algorithm>
#include "../common/cuda/utils.h"
#if MXNET_USE_CUDNN == 1
#include "./cudnn_bilinear_sampler-inl.h"
#endif  // MXNET_USE_CUDNN

namespace mshadow {
namespace cuda {
template<typename DType>
__device__ bool between(DType value, int lowerBound, int upperBound) {
  return (value >= lowerBound && value <= upperBound);
}
template<typename DType>
__global__ void BilinearSamplerForwardKernel(const int i_c, const int i_h,
                                              const int i_w, const DType* data,
                                              const DType* grid, const int o_n,
                                              const int o_c, const int o_h,
                                              const int o_w, DType* out) {
  for (int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
       index < o_n * o_c * o_h * o_w;
       index += blockDim.x * gridDim.x * gridDim.y) {
    // (n, c, h, w) is the element in out
    int w = index % o_w;
    int h = (index / o_w) % o_h;
    int c = (index / o_w / o_h) % o_c;
    int n = index / o_w / o_h / o_c;
    int out_index = n * o_c * o_h * o_w + c * o_h * o_w + h * o_w + w;
    int grid_index = n * o_h * o_w * 2 + h * o_w + w;
    DType y_real = (*(grid + grid_index + o_h * o_w) + 1) * (i_h - 1) / 2;
    DType x_real = (*(grid + grid_index) + 1) * (i_w - 1) / 2;
    int top_left_y = static_cast<int>(floor(y_real));
    int top_left_x = static_cast<int>(floor(x_real));
    DType top_left_y_w = 1.0 - (y_real - top_left_y);
    DType top_left_x_w = 1.0 - (x_real - top_left_x);
    int data_index = n * i_c * i_h * i_w + c * i_h * i_w + top_left_y * i_w + top_left_x;
    DType top_left_v = 0;
    DType top_right_v = 0;
    DType bottom_left_v = 0;
    DType bottom_right_v = 0;
    if (between(top_left_x, 0, i_w-1) && between(top_left_y, 0, i_h-1))
      top_left_v = *(data + data_index);
    if (between(top_left_x + 1, 0, i_w-1) && between(top_left_y, 0, i_h-1))
      top_right_v = *(data + data_index + 1);
    if (between(top_left_x, 0, i_w-1) && between(top_left_y + 1, 0, i_h-1))
      bottom_left_v = *(data + data_index + i_w);
    if (between(top_left_x+1, 0, i_w-1) && between(top_left_y + 1, 0, i_h-1))
      bottom_right_v = *(data + data_index + i_w + 1);
    *(out+out_index) = top_left_v * top_left_y_w * top_left_x_w +
                        top_right_v * top_left_y_w * (1.0 - top_left_x_w) +
                        bottom_left_v * (1.0 - top_left_y_w) * top_left_x_w +
                        bottom_right_v * (1.0 - top_left_y_w) * (1.0 - top_left_x_w);
  }
}

template<typename DType, int Req1, int Req2>
__global__ void BilinearSamplerBackwardKernel(const int i_c, const int i_h,
                                              const int i_w, const DType* grad,
                                              const DType* data, const int o_n,
                                              const int o_c, const int o_h,
                                              const int o_w, DType* g_input,
                                              const DType* grid_src,
                                              DType* grad_grid) {
  for (int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
       index < o_n * o_h * o_w;
       index += blockDim.x * gridDim.x * gridDim.y) {
    // (n, c, h, w) is the element in grad
    int w = index % o_w;
    int h = (index / o_w) % o_h;
    int n = index / o_w / o_h;
    DType top_left_y_gw = 0.0;
    DType top_left_x_gw = 0.0;
    int grid_src_index = n * o_h * o_w * 2 + h * o_w + w;
    DType y_real = (*(grid_src + grid_src_index + o_h * o_w) + 1) * (i_h - 1) / 2;
    DType x_real = (*(grid_src + grid_src_index) + 1) * (i_w - 1) / 2;

    int top_left_y = static_cast<int>(floor(y_real));
    int top_left_x = static_cast<int>(floor(x_real));
    DType top_left_y_w = 1.0 - (y_real - top_left_y);
    DType top_left_x_w = 1.0 - (x_real - top_left_x);
    for (int c = 0; c < o_c; ++c) {
      int grad_index = n * o_c * o_h * o_w + c * o_h * o_w + h * o_w + w;
      int data_index = n * i_c * i_h * i_w + c * i_h * i_w + top_left_y * i_w + top_left_x;
      // calc 4 vertex value in input data
      DType top_left_v = 0;
      DType top_right_v = 0;
      DType bottom_left_v = 0;
      DType bottom_right_v = 0;
      // calc input grad
      if (between(top_left_x, 0, i_w-1) && between(top_left_y, 0, i_h-1)) {
        if (Req1 != mxnet::kNullOp) {
          atomicAdd(&g_input[data_index], *(grad + grad_index) * top_left_y_w * top_left_x_w);
        }
        top_left_v = *(data + data_index);
      }
      if (between(top_left_x+1, 0, i_w-1) && between(top_left_y, 0, i_h-1)) {
        if (Req1 != mxnet::kNullOp) {
          atomicAdd(&g_input[data_index + 1],
                    *(grad + grad_index) * top_left_y_w * (1.0 - top_left_x_w));
        }
        top_right_v = *(data + data_index + 1);
      }
      if (between(top_left_x, 0, i_w-1) && between(top_left_y+1, 0, i_h-1)) {
        if (Req1 != mxnet::kNullOp) {
          atomicAdd(&g_input[data_index+ i_w],
                    *(grad + grad_index) * (1.0 - top_left_y_w) * top_left_x_w);
        }
        bottom_left_v = *(data + data_index + i_w);
      }
      if (between(top_left_x+1, 0, i_w-1) && between(top_left_y+1, 0, i_h-1)) {
        if (Req1 != mxnet::kNullOp) {
          atomicAdd(&g_input[data_index+ i_w + 1],
                    *(grad + grad_index) * (1.0 - top_left_y_w) * (1.0 - top_left_x_w));
        }
        bottom_right_v = *(data + data_index + i_w + 1);
      }
      // calc weight grad of top_left_w, then multiple -1 is the grad of grid_src
      top_left_y_gw -= *(grad + grad_index) * (top_right_v - bottom_right_v +
                        (top_left_v - top_right_v - bottom_left_v + bottom_right_v)
                        * top_left_x_w);
      top_left_x_gw -= *(grad + grad_index) * (bottom_left_v - bottom_right_v +
                        (top_left_v - top_right_v - bottom_left_v + bottom_right_v)
                        * top_left_y_w);
    }
    if (Req2 != mxnet::kNullOp) {
      // calc grad of grid
      *(grad_grid + grid_src_index + o_h * o_w) += top_left_y_gw * (i_h - 1) / 2;
      *(grad_grid + grid_src_index) += top_left_x_gw * (i_w - 1) / 2;
    }
  }
}
}  // namespace cuda

template<typename DType>
inline void BilinearSamplerForward(const Tensor<gpu, 4, DType> &output,
                                    const Tensor<gpu, 4, DType> &input,
                                    const Tensor<gpu, 4, DType> &grid_src) {
    DType *out = output.dptr_;
    const DType *data = input.dptr_;
    const DType *grid = grid_src.dptr_;
    int o_n = output.size(0), o_c = output.size(1), o_h = output.size(2), o_w = output.size(3);
    int i_c = input.size(1), i_h = input.size(2), i_w = input.size(3);
    using namespace cuda;
    const int max_block = (output.shape_.Size() + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock;
    const int grid_dim_x = (max_block > kMaxGridDim) ? kMaxGridDim : max_block;
    const int grid_dim_y =
      (max_block > kMaxGridDim) ? (max_block + kMaxGridDim - 1) / kMaxGridDim : 1;
    dim3 num_blocks(grid_dim_x, grid_dim_y);
    dim3 threads_per_block(kMaxThreadsPerBlock);
    CheckLaunchParam(num_blocks, threads_per_block, "bilinear sampler forward");
    hipStream_t stream = Stream<gpu>::GetStream(output.stream_);
    cuda::BilinearSamplerForwardKernel<DType> << <num_blocks, threads_per_block, 0, stream >> >(
      i_c, i_h, i_w, data, grid, o_n, o_c, o_h, o_w, out);
    // post kernel check
    hipError_t err = hipGetLastError();
    CHECK_EQ(err, hipSuccess) << hipGetErrorString(err);
}

template<typename DType>
inline void BilinearSamplerBackward(const Tensor<gpu, 4, DType> &input_grad,
                                    const Tensor<gpu, 4, DType> &ggrid,
                                    const Tensor<gpu, 4, DType> &output_grad,
                                    const Tensor<gpu, 4, DType> &input_data,
                                    const Tensor<gpu, 4, DType> &grid,
                                    const mxnet::OpReqType data_req,
                                    const mxnet::OpReqType grid_req) {
  using namespace mxnet;
  DType *g_input = input_grad.dptr_;
  DType *grad_grid = ggrid.dptr_;
  const DType *grid_src = grid.dptr_;
  const DType *grad = output_grad.dptr_;
  const DType *data = input_data.dptr_;
  int o_n = output_grad.size(0), o_c = output_grad.size(1),
      o_h = output_grad.size(2), o_w = output_grad.size(3);
  int i_c = input_data.size(1), i_h = input_data.size(2), i_w = input_data.size(3);
  using namespace cuda;
  const int max_block = (output_grad.shape_.Size() / o_c + kMaxThreadsPerBlock - 1)
                        / kMaxThreadsPerBlock;
  const int grid_dim_x = (max_block > kMaxGridDim) ? kMaxGridDim : max_block;
  const int grid_dim_y =
    (max_block > kMaxGridDim) ? (max_block + kMaxGridDim - 1) / kMaxGridDim : 1;
  dim3 num_blocks(grid_dim_x, grid_dim_y);
  dim3 threads_per_block(kMaxThreadsPerBlock);
  CheckLaunchParam(num_blocks, threads_per_block, "bilinear sampler backward");
  hipStream_t stream = Stream<gpu>::GetStream(input_grad.stream_);
  MXNET_REQ_TYPE_SWITCH(data_req, Req1, {
    MXNET_REQ_TYPE_SWITCH(grid_req, Req2, {
      cuda::BilinearSamplerBackwardKernel<DType, Req1, Req2>
      <<<num_blocks, threads_per_block, 0, stream >>>(
        i_c, i_h, i_w, grad, data, o_n, o_c, o_h, o_w, g_input, grid_src, grad_grid);
    });
  });
  // post kernel check
  hipError_t err = hipGetLastError();
  CHECK_EQ(err, hipSuccess) << hipGetErrorString(err);
}

}  // namespace mshadow

namespace mxnet {
namespace op {
template<>
Operator* CreateOp<gpu>(BilinearSamplerParam param, int dtype) {
  Operator *op = nullptr;
#if MXNET_USE_CUDNN == 1
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    if (param.cudnn_off.has_value() && param.cudnn_off.value()) {
      op = new BilinearSamplerOp<gpu, DType>(param);
    } else {
      op = new CuDNNBilinearSamplerOp<DType>(param);
    }
  })
#else
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new BilinearSamplerOp<gpu, DType>(param);
  })
#endif  // MXNET_USE_CUDNN
  return op;
}

}  // namespace op
}  // namespace mxnet
