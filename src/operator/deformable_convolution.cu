/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2017 Microsoft
 * Licensed under The Apache-2.0 License [see LICENSE for details]
 * \file deformable_convolution.cu
 * \brief
 * \author Yuwen Xiong, Haozhi Qi, Jifeng Dai
*/

#include "./deformable_convolution-inl.h"
#include <vector>

namespace mxnet {
namespace op {

  template<>
  Operator* CreateOp<gpu>(DeformableConvolutionParam param, int dtype,
    mxnet::ShapeVector *in_shape,
    mxnet::ShapeVector *out_shape,
    Context ctx) {
    Operator *op = nullptr;
    MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
      op = new DeformableConvolutionOp<gpu, DType>(param);
    })
      return op;
  }

}  // namespace op
}  // namespace mxnet

