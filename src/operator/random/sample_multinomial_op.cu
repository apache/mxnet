#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2017 by Contributors
 * \file sample_multinomial_op.h
 * \brief Operator for sampling from multinomial distributions
 */
#include "./sample_multinomial_op.h"

namespace mxnet {
namespace op {

NNVM_REGISTER_OP(_sample_multinomial)
.set_attr<FCompute>("FCompute<gpu>", SampleMultinomialForward<gpu>);


struct SampleMultinomialBackwardGPUKernel {
  template<typename DType, typename IType>
  MSHADOW_XINLINE static void Map(int i, index_t K, index_t M,
                                  DType* ograd, DType* dist, IType* out,
                                  DType* igrad) {
    for (index_t j = 0; j < M; ++j) {
      atomicAdd(&igrad[i*K + static_cast<size_t>(out[i*M + j])],
        ograd[i*M + j] / dist[i*K + static_cast<size_t>(out[i*M + j])]);
    }
  }
};


NNVM_REGISTER_OP(_backward_sample_multinomial)
.set_attr<FCompute>("FCompute<gpu>",
  SampleMultinomialBackward<SampleMultinomialBackwardGPUKernel, gpu>);


}  // namespace op
}  // namespace mxnet
