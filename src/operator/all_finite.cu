#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 *  Copyright (c) 2019 by Contributors
 * \file all_finite.cu
 * \brief operator for checking if a group of array is all finite
 * \author Clement Fuji Tsang
 */

#include "./all_finite-inl.h"

namespace mxnet {
namespace op {

template <typename DType>
__global__ void AllFiniteGPUKernel(const int size, const DType* in, float* out) {
  bool is_finite = true;
  CUDA_KERNEL_LOOP(i, size) {
    is_finite = isfinite(static_cast<float>(in[i])) ? is_finite : false;
  }
  __syncthreads();
  if (!is_finite) {
    out[0] = 0.;
  }
}

inline void AllFiniteGPU(const nnvm::NodeAttrs& attrs,
                         const OpContext &ctx,
                         const std::vector<TBlob> &inputs,
                         const std::vector<OpReqType> &req,
                         const std::vector<TBlob> &outputs) {
  using namespace mxnet_op;
  Stream<gpu>* s = ctx.get_stream<gpu>();
  const AllFiniteParam& op_param = nnvm::get<AllFiniteParam>(attrs.parsed);
  Tensor<gpu, 2, float> out = outputs[0].FlatTo2D<gpu, float>(s);
  if (op_param.init_output)
    out = 1.;
  MSHADOW_REAL_TYPE_SWITCH(inputs[0].type_flag_, DType, {
    Tensor<gpu, 2, DType> in = inputs[0].FlatTo2D<gpu, DType>(s);
    const int n = in.shape_.Size();
    AllFiniteGPUKernel<DType><<<cuda_get_num_blocks(n),
                                mshadow::cuda::kBaseThreadNum, 0,
                                mshadow::Stream<gpu>::GetStream(s)>>>(n, in.dptr_, out.dptr_);
    MSHADOW_CUDA_POST_KERNEL_CHECK(AllFiniteGPUKernel<DType>);
  });
}

template <typename DType>
__global__ void MultiAllFiniteGPUKernel(const MultiAllFiniteKernelParam<DType> param, float* out) {
  bool is_finite = true;
  for (int index = 0; index < param.count; ++index) {
    CUDA_KERNEL_LOOP(i, param.sizes[index]) {
      is_finite = isfinite(static_cast<float>(param.arrays[index][i])) ? is_finite : false;
    }
  }
  __syncthreads();
  if (!is_finite) {
    out[0] = 0.;
  }
}

inline void MultiAllFiniteGPU(const nnvm::NodeAttrs& attrs,
                              const OpContext &ctx,
                              const std::vector<TBlob> &inputs,
                              const std::vector<OpReqType> &req,
                              const std::vector<TBlob> &outputs) {
  using namespace mxnet_op;
  Stream<gpu>* s = ctx.get_stream<gpu>();
  const MultiAllFiniteParam& op_param = nnvm::get<MultiAllFiniteParam>(attrs.parsed);
  Tensor<gpu, 2, float> out = outputs[0].FlatTo2D<gpu, float>(s);
  if (op_param.init_output)
    out = 1.;
  MSHADOW_REAL_TYPE_SWITCH(inputs[0].type_flag_, DType, {
    MultiAllFiniteKernelParam<DType> param =
      FillMultiAllFiniteParam<gpu, DType>(op_param, ctx, inputs);
    MultiAllFiniteGPUKernel<DType><<<cuda_get_num_blocks(param.max_size),
                                     mshadow::cuda::kBaseThreadNum, 1,
                                     mshadow::Stream<gpu>::GetStream(s)>>>(param, out.dptr_);
    MSHADOW_CUDA_POST_KERNEL_CHECK(MultiAllFiniteGPUKernel<DType>);
  });
}

NNVM_REGISTER_OP(all_finite)
.set_attr<FCompute>("FCompute<gpu>", AllFiniteGPU);

NNVM_REGISTER_OP(multi_all_finite)
.set_attr<FCompute>("FCompute<gpu>", MultiAllFiniteGPU);

}  // namespace op
}  // namespace mxnet
