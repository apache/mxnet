#include "hip/hip_runtime.h"
/*
* Licensed to the Apache Software Foundation (ASF) under one
* or more contributor license agreements.  See the NOTICE file
* distributed with this work for additional information
* regarding copyright ownership.  The ASF licenses this file
* to you under the Apache License, Version 2.0 (the
* "License"); you may not use this file except in compliance
* with the License.  You may obtain a copy of the License at
*
*   http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing,
* software distributed under the License is distributed on an
* "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
* KIND, either express or implied.  See the License for the
* specific language governing permissions and limitations
* under the License.
*/

/*!
* \file image_random.cu
* \brief GPU Implementation of image transformation operators
*/
#include <hip/hip_runtime_api.h>
#include "./image_random-inl.h"
#include "../elemwise_op_common.h"

namespace mxnet {
namespace op {
namespace image {

using namespace mshadow;

// ToTensor Kernel for 3D input
/*
 * In order to not generate the code that uses too many
 * registers (resulting in too many resources requested
 * error) we need to tell the compiler that we will be
 * launching this kernel with cuda::kMaxThreadsPerBlock
 * threads per block. Setting __launch_bounds__ ensures
 * that such configuration can always be launched.
 */
template<typename xpu, typename Dtype>
__global__ void
__launch_bounds__(cuda::kMaxThreadsPerBlock, 1)
ToTensorCudaKernel(const Tensor<xpu, 3, Dtype> input,
                   const Tensor<xpu, 3, float> output,
                   const int req,
                   const int N,
                   const int H,
                   const int W,
                   const int C,
                   const float normalize_factor) {
    // We process one image per thread block.
    // In 3D case, we have only 1 block i.e., blockIdx.x
    // We do not use it.
    for (int c = 0; c < C; ++c) {
        for (int h = threadIdx.y; h < H; h += blockDim.y) {
            for (int w = threadIdx.x; w < W; w += blockDim.x) {
                KERNEL_ASSIGN(output[c][h][w], req,
                              input[h][w][c] / normalize_factor);
            }
        }
    }
}

// ToTensor Kernel for 4D input
template<typename xpu, typename Dtype>
__global__ void
__launch_bounds__(cuda::kMaxThreadsPerBlock, 1)
ToTensorCudaKernel(const Tensor<xpu, 4, Dtype> input,
                   const Tensor<xpu, 4, float> output,
                   const int req,
                   const int N,
                   const int H,
                   const int W,
                   const int C,
                   const float normalize_factor) {
    // We process one image per thread block.
    const int n = blockIdx.x;

    for (int c = 0; c < C; ++c) {
        for (int h = threadIdx.y; h < H; h += blockDim.y) {
            for (int w = threadIdx.x; w < W; w += blockDim.x) {
                KERNEL_ASSIGN(output[n][c][h][w], req,
                              input[n][h][w][c] / normalize_factor);
            }
        }
    }
}

template<typename DType, typename T1, typename T2>
void ToTensorImplCUDA(mshadow::Stream<gpu> *s,
                      const T1 input,
                      const T2 output,
                      const int req,
                      const float normalize_factor) {
    int blocks, H, W, C, N;
    hipStream_t stream = mshadow::Stream<gpu>::GetStream(s);
    if (std::is_same<T1, Tensor<gpu, 3, DType>>::value) {
        // 3D Input - (H, W, C)
        N = 0;
        H = input.size(0);
        W = input.size(1);
        C = input.size(2);
        blocks = 1;
    } else {
        // 4D Input - (N, H, W, C)
        N = input.size(0);
        H = input.size(1);
        W = input.size(2);
        C = input.size(3);
        blocks = N > 0 ? N : 1;
    }

    ToTensorCudaKernel<gpu, DType>
            <<<blocks, dim3(32, 32), 0, stream>>>(input, output,
                req, N, H, W, C, normalize_factor);
        MSHADOW_CUDA_POST_KERNEL_CHECK(ToTensorCudaKernel);
}

// Normalize Forward CUDA Kernel
template<typename xpu, typename DType>
__global__ void
__launch_bounds__(cuda::kMaxThreadsPerBlock, 1)
NormalizeCudaKernel(const DType* input,
                    DType* output,
                    const int req,
                    const int N,
                    const int C,
                    const int H,
                    const int W,
                    const float mean_d0,
                    const float mean_d1,
                    const float mean_d2,
                    const float std_d0,
                    const float std_d1,
                    const float std_d2) {
    // We process one image per thread block.
    const int n = blockIdx.x;
    const int length = H * W;
    const int step = C * length * n;

    float mean = mean_d0;
    float std = std_d0;
    for (int c = 0; c < C; ++c) {
        switch (c) {
            case 0 : break;
            case 1 : mean = mean_d1;
                     std = std_d1;
                     break;
            case 2 : mean = mean_d2;
                     std = std_d2;
                     break;
        }
        for (int i = threadIdx.x; i < length; i += blockDim.x) {
            KERNEL_ASSIGN(*(output + step + i + (c * length)), req,
                      (*(input + step + i + (c * length)) - mean) / std);
        }
    }
}

template<typename DType>
void NormalizeImplCUDA(mshadow::Stream<gpu> *s,
                       const DType* input,
                       DType* output,
                       const int req,
                       const int N,
                       const int C,
                       const int H,
                       const int W,
                       const float mean_d0,
                       const float mean_d1,
                       const float mean_d2,
                       const float std_d0,
                       const float std_d1,
                       const float std_d2) {
    hipStream_t stream = mshadow::Stream<gpu>::GetStream(s);
    NormalizeCudaKernel<gpu, DType>
    // 1 image per block. N is batch size.
    <<<N, dim3(cuda::kMaxThreadsPerBlock, 1), 0, stream>>>(input, output,
        req, N, C, H, W, mean_d0, mean_d1, mean_d2,
        std_d0, std_d1, std_d2);
    MSHADOW_CUDA_POST_KERNEL_CHECK(NormalizeCudaKernel);
}

// Normalize Backward Kernel
template<typename xpu, typename DType>
__global__ void
__launch_bounds__(cuda::kMaxThreadsPerBlock, 1)
NormalizeBackwardCudaKernel(const DType *out_grad,
                            DType *in_grad,
                            const int req,
                            const int N,
                            const int C,
                            const int H,
                            const int W,
                            const float std_d0,
                            const float std_d1,
                            const float std_d2) {
    // We process one image per thread block.
    const int n = blockIdx.x;
    const int length = H * W;
    const int step = C * length * n;

    float std = std_d0;
    for (int c = 0; c < C; ++c) {
        switch (c) {
            case 0 : break;
            case 1 : std = std_d1;
                     break;
            case 2 : std = std_d2;
                     break;
        }
        for (int i = threadIdx.x; i < length; i += blockDim.x) {
            KERNEL_ASSIGN(*(in_grad + step + i + (c * length)), req,
                          *(out_grad + step + i + (c * length)) * (1.0 / std));
        }
    }
}

template<typename DType>
void NormalizeBackwardImplCUDA(mshadow::Stream<gpu> *s,
                               const DType *out_grad,
                               DType *in_grad,
                               const int req,
                               const int N,
                               const int C,
                               const int H,
                               const int W,
                               const float std_d0,
                               const float std_d1,
                               const float std_d2) {
    hipStream_t stream = mshadow::Stream<gpu>::GetStream(s);
    NormalizeBackwardCudaKernel<gpu, DType>
    // 1 image per block. N is batch size.
    <<<N, dim3(cuda::kMaxThreadsPerBlock, 1), 0, stream>>>(out_grad, in_grad,
        req, N, C, H, W, std_d0, std_d1, std_d2);
    MSHADOW_CUDA_POST_KERNEL_CHECK(NormalizeBackwardCudaKernel);
}

NNVM_REGISTER_OP(_image_to_tensor)
.set_attr<FCompute>("FCompute<gpu>", ToTensorOpForward<gpu>);

NNVM_REGISTER_OP(_image_normalize)
.set_attr<FCompute>("FCompute<gpu>", NormalizeOpForward<gpu>);

NNVM_REGISTER_OP(_backward_image_normalize)
.set_attr<FCompute>("FCompute<gpu>", NormalizeOpBackward<gpu>);

}  // namespace image
}  // namespace op
}  // namespace mxnet
