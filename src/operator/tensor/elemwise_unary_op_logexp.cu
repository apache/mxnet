#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file elemwise_unary_op_logexp.cu
 * \brief GPU Implementation of unary log and exp functions.
 */
#include "./elemwise_binary_op.h"

namespace mxnet {
namespace op {

// exp
NNVM_REGISTER_OP(exp)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"exp"});

// log
NNVM_REGISTER_OP(log)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"log"});

// log10
NNVM_REGISTER_OP(log10)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"log10"});

// log2
NNVM_REGISTER_OP(log2)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"log2"});

NNVM_REGISTER_OP(_backward_log)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_log"});

NNVM_REGISTER_OP(_backward_log10)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_log10"});

NNVM_REGISTER_OP(_backward_log2)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_log2"});

// log1p
NNVM_REGISTER_OP(log1p)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"log1p"})
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryRTCCompute{"log1p"});

NNVM_REGISTER_OP(_backward_log1p)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_log1p"});

// expm1
NNVM_REGISTER_OP(expm1)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"expm1"})
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryRTCCompute{"expm1"});

NNVM_REGISTER_OP(_backward_expm1)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_expm1"});

}  // namespace op
}  // namespace mxnet
