#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2015 by Contributors
 * \file elemwise_sum.cu
 * \brief GPU implementation of elementwise sum operator
*/
#include "./elemwise_sum.h"
#include "../../ndarray/ndarray_function.h"
#include "../../common/cuda/rtc.h"
#include "../../common/cuda/rtc/vectorization-inl.h"

namespace mxnet {
namespace op {

namespace {

constexpr size_t num_inputs_per_kernel = 4;

struct elementwise_sum_params {
  int num_inputs;
  const void* inputs[num_inputs_per_kernel];
  void* outputs[1];
};

const char elementwise_sum_kernel[] = R"code(
constexpr size_t num_inputs_per_kernel = 4;

struct elementwise_sum_params {
  int num_inputs;
  const void* inputs[num_inputs_per_kernel];
  void* outputs[1];
};

__launch_bounds__(kRTCMaxThreadsPerBlock)
__global__ void elementwise_sum_kernel(
    const elementwise_sum_params params,
    const index_t lead_dim,
    const index_t other_dim,
    const index_t N,
    const index_t num_aligned_elements) {
  using namespace vector;
  VectorizedStorer<OutputType0, nvec, aligned> storer(
    reinterpret_cast<OutputType0*>(params.outputs[0]), N);

  using IType = AccType<InputType0>;
  using OType = AccType<OutputType0>;

  const index_t M = num_aligned_elements;

  for (index_t tid = blockIdx.x * blockDim.x + threadIdx.x;
      tid < M;
      tid += gridDim.x * blockDim.x) {
    typename OType::type temp[nvec];
    if (req == OpReqType::kAddTo) {
      storer.load(tid, N);
#pragma unroll
      for (int i = 0; i < nvec; ++i) {
        temp[i] = OType::from(storer.separate()[i]);
      }
    } else {
#pragma unroll
      for (int i = 0; i < nvec; ++i) {
        temp[i] = 0;
      }
    }
#pragma unroll
    for (int i = 0; i < num_inputs_per_kernel; ++i) {
      if (i < params.num_inputs) {
        VectorizedLoader<InputType0, nvec, aligned> loader(
          reinterpret_cast<const InputType0*>(params.inputs[i]), N);
        loader.load(tid, N);
#pragma unroll
        for (int i = 0; i < nvec; ++i) {
          temp[i] += IType::from(loader.separate()[i]);
        }
      }
    }
#pragma unroll
    for (int i = 0; i < nvec; ++i) {
      storer.separate()[i] = OType::to(temp[i]);
    }

    storer.store(tid, N);
  }
}
)code";

void VectorizedElementwiseSum(const nnvm::NodeAttrs &attrs,
                              const OpContext &ctx,
                              const std::vector<TBlob> &inputs,
                              const std::vector<OpReqType> &req,
                              const std::vector<TBlob> &outputs) {
  using namespace mxnet::common::cuda::rtc;
  mshadow::Stream<gpu> *s = ctx.get_stream<gpu>();
  if (req[0] == kNullOp) return;
  CHECK_EQ(outputs.size(), 1U);
  size_t output_type_size = common::mshadow_type_info(outputs[0].type_flag_).size;
  const int nvec = output_type_size <= sizeof(uint2)
                     ? (sizeof(uint2) / output_type_size)
                     : 1;
  const index_t size = inputs[0].Size();
  for (size_t i = 0; i < inputs.size(); i += num_inputs_per_kernel) {
    const std::string code = std::string("const OpReqType req = ") +
                             util::to_string(i == 0 ? req[0] : kAddTo) +
                             ";\n";
    elementwise_sum_params params{};
    params.num_inputs = std::min(num_inputs_per_kernel, inputs.size() - i);
    for (int j = 0; j < params.num_inputs; ++j) {
      params.inputs[j] = inputs[i + j].dptr_;
    }
    params.outputs[0] = outputs[0].dptr_;
    const std::vector<TBlob> new_inputs(inputs.begin() + i,
                                        inputs.begin() + i + params.num_inputs);
    VectorizedKernelRTCLauncher(code, "elementwise_sum_kernel",
                                elementwise_sum_kernel, nvec,
                                size, 1, s, params,
                                new_inputs, outputs,
                                ctx.run_ctx.get_ctx().dev_id);
  }
}

void ElementWiseSumComputeExGPU(const nnvm::NodeAttrs& attrs,
                                const OpContext& ctx,
                                const std::vector<NDArray>& inputs,
                                const std::vector<OpReqType>& req,
                                const std::vector<NDArray>& outputs) {
  CHECK(!inputs.empty());
  CHECK_EQ(outputs.size(), 1U);
  CHECK_EQ(req.size(), 1U);
  if (req[0] == kNullOp) return;
  CHECK_EQ(req[0], kWriteTo) << "ElementWiseSumComputeExGPU only supports req = kWriteTo";
  if (common::ContainsOnlyStorage(inputs, kRowSparseStorage) ||
      (inputs.size() == 3U && inputs[0].storage_type() == kDefaultStorage &&
       inputs[1].storage_type() == kCSRStorage && inputs[2].storage_type() == kDefaultStorage) ||
      (inputs.size() > 4U && common::ContainsStorageType(inputs, kDefaultStorage) &&
       outputs[0].storage_type() == kDefaultStorage)) {
    mshadow::Stream<gpu>* s = ctx.get_stream<gpu>();
    NDArray out_nd = outputs[0];
    mxnet::ndarray::ElementwiseSum<gpu>(s, ctx.requested[0], inputs, &out_nd);
  } else {
    LogUnimplementedOp(attrs, ctx, inputs, req, outputs);
  }
}

}  // namespace

NNVM_REGISTER_OP(add_n)
.set_attr<FCompute>("FCompute<gpu>", VectorizedElementwiseSum)
.set_attr<FComputeEx>("FComputeEx<gpu>", ElementWiseSumComputeExGPU);

}  // namespace op
}  // namespace mxnet
