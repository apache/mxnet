#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 *  Copyright (c) 2015 by Contributors
 * \file matrix_op.cu
 * \brief GPU Implementation of matrix operations
 */
#include <hipcub/hipcub.hpp>
#include "./matrix_op-inl.h"
#include "./elemwise_unary_op.h"


namespace mxnet {
namespace op {

/*!
 * \brief Compute the number of elements of every row.
 */
struct SliceMarkCsrIndPtr {
  /*!
   * \brief
   * \param i           the i-th row of the output csr ndarray
   * \param prefix_sum  indptr array of the output csr ndarray
   * \param in_idx      indices array of the input csr ndarray
   * \param in_indptr   indptr array of the input csr ndarray
   * \param begin_col   starting indice
   * \param end_col     ending indice
   */
  template<typename IType, typename RType>
  MSHADOW_XINLINE static void Map(int i,
                                  RType* prefix_sum,
                                  const IType* in_idx,
                                  const RType* in_indptr,
                                  const int begin_col, const int end_col) {
    if (i == 0) {
      prefix_sum[0] = 0;
    }
    RType size = 0;
    for (RType j = in_indptr[i]; j < in_indptr[i+1]; j++) {
      // indices of CSRNDArray are in ascending order per row
      if (in_idx[j] >= end_col) {
        break;
      } else if (in_idx[j] >= begin_col) {
        size++;
      }
    }
    prefix_sum[i+1] = size;
  }
};


template<>
void SliceDimTwoCsrImpl<gpu>(const mxnet::TShape &begin, const mxnet::TShape &end,
                             const OpContext& ctx, const NDArray &in, const NDArray &out) {
  using namespace mshadow;
  using namespace mxnet_op;
  using namespace csr;

  Stream<gpu> *s = ctx.get_stream<gpu>();

  nnvm::dim_t begin_row = begin[0], end_row = end[0];
  nnvm::dim_t begin_col = begin[1], end_col = end[1];
  nnvm::dim_t indptr_len = end_row - begin_row + 1;
  out.CheckAndAllocAuxData(kIndPtr, Shape1(indptr_len));
  // assume idx indptr share the same type
  MSHADOW_IDX_TYPE_SWITCH(in.aux_type(kIndPtr), RType, {
    MSHADOW_IDX_TYPE_SWITCH(in.aux_type(kIdx), IType, {
      MSHADOW_TYPE_SWITCH(in.dtype(), DType, {
        RType *in_indptr = in.aux_data(kIndPtr).dptr<RType>();
        IType *in_idx = in.aux_data(kIdx).dptr<IType>();
        DType *in_data = in.data().dptr<DType>();

        RType *out_indptr = out.aux_data(kIndPtr).dptr<RType>();

        Kernel<SliceMarkCsrIndPtr, gpu>::Launch(s, indptr_len - 1,
                                                out_indptr,
                                                in_idx,
                                                in_indptr + begin_row,
                                                begin_col, end_col);
        void* d_temp_storage = nullptr;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceScan::InclusiveSum(d_temp_storage,
                                      temp_storage_bytes,
                                      out_indptr,
                                      out_indptr,
                                      indptr_len,
                                      Stream<gpu>::GetStream(s));
        Tensor<gpu, 1, char> workspace = ctx.requested[0]
            .get_space_typed<gpu, 1, char>(Shape1(temp_storage_bytes), s);
        d_temp_storage = workspace.dptr_;

        hipcub::DeviceScan::InclusiveSum(d_temp_storage,
                                      temp_storage_bytes,
                                      out_indptr,
                                      out_indptr,
                                      indptr_len,
                                      Stream<gpu>::GetStream(s));
        // retrieve nnr
        RType nnr = 0;
        CUDA_CALL(hipMemcpyAsync(&nnr, &out_indptr[indptr_len-1], sizeof(RType),
                                  hipMemcpyDeviceToHost, mshadow::Stream<gpu>::GetStream(s)));
        CUDA_CALL(hipStreamSynchronize(mshadow::Stream<gpu>::GetStream(s)));

        // returns zeros in csr format if nnr = 0
        if (nnr == 0) {
          out.set_aux_shape(kIdx, Shape1(0));
          return;
        }
        out.CheckAndAllocAuxData(kIdx, Shape1(nnr));
        out.CheckAndAllocData(Shape1(nnr));
        IType *out_idx = out.aux_data(kIdx).dptr<IType>();
        DType *out_data = out.data().dptr<DType>();

        Kernel<SliceDimTwoCsrAssign, gpu>::Launch(s, indptr_len - 1, out_idx, out_data,
                                                  out_indptr, in_idx, in_data,
                                                  in_indptr + begin_row,
                                                  begin_col, end_col);
      });
    });
  });
}

template <typename DType>
struct split_tensor_data {
  static const int MaxSections = 128;
  size_t num_sections;
  DType* outputs[MaxSections];
  size_t indices[MaxSections+1];
  DType* inputs[1];
};

template <bool split_last_axis, typename LType, typename DType>
__global__ void split_tensor_kernel(size_t input_size,
                                    const split_tensor_data<DType> params,
                                    size_t split_axis_size,
                                    size_t tail_size,
                                    size_t last_axis_size,
                                    size_t blocks_last_axis) {
  const int entries_per_load = sizeof(LType)/sizeof(DType);
  const LType* in_aligned = reinterpret_cast<const LType*>(params.inputs[0]);
  const size_t last_axis_size_aligned = entries_per_load > 0 ?
                                        last_axis_size / entries_per_load : last_axis_size;
  if (split_last_axis) {
    size_t input_offset_leading = (blockIdx.x / blocks_last_axis) * last_axis_size_aligned;
    size_t position_last_axis = (blockIdx.x % blocks_last_axis) * blockDim.x * entries_per_load +
                                 params.indices[0] + threadIdx.x * entries_per_load;
    if (position_last_axis < params.indices[params.num_sections]) {
      size_t position_last_axis_aligned = entries_per_load > 0 ?
                                          position_last_axis / entries_per_load :
                                          position_last_axis;
      LType input_data = in_aligned[input_offset_leading + position_last_axis_aligned];
      // Binary search to find section of each thread
      size_t lower = 0;
      size_t upper = params.num_sections - 1;
      while (lower < upper) {
        size_t mid = (lower + upper + 1) / 2;
        if (position_last_axis >=  params.indices[mid])
          lower = mid;
        else
          upper = mid - 1;
      }
      size_t section = upper;
      size_t section_size = params.indices[section + 1] - params.indices[section];
      LType* out_aligned = reinterpret_cast<LType*>(params.outputs[section]);
      size_t section_size_aligned = entries_per_load > 0 ? section_size / entries_per_load :
                                                           section_size;
      size_t index_aligned = entries_per_load > 0 ? params.indices[section] / entries_per_load :
                                                    params.indices[section];
      size_t output_offset_leading = (blockIdx.x / blocks_last_axis) * section_size_aligned;
      size_t output_position = output_offset_leading + position_last_axis_aligned - index_aligned;
      out_aligned[output_position] = input_data;
    }
  } else {
    size_t split_axis_size_iter = params.indices[params.num_sections] - params.indices[0];
    size_t blocks_per_leading_dim = (split_axis_size_iter * tail_size * blocks_last_axis);
    // input offsets: leading (axes pre-split-axis), at split-axis, tail, and blocks_last_axis
    size_t input_offset_leading = (blockIdx.x / blocks_per_leading_dim) *
                                   split_axis_size * tail_size * last_axis_size_aligned;
    size_t pos_in_split_axis = (blockIdx.x / (tail_size * blocks_last_axis)) %
                               split_axis_size_iter + params.indices[0];
    size_t input_offset_split_axis = pos_in_split_axis * tail_size * last_axis_size_aligned;
    size_t offset_tail = ((blockIdx.x / blocks_last_axis) % tail_size) *
                         last_axis_size_aligned;
    size_t input_offset = input_offset_leading + input_offset_split_axis + offset_tail +
                          (blockIdx.x % blocks_last_axis) * blockDim.x;
    // Binary search to find section for this block
    size_t lower = 0;
    size_t upper = params.num_sections - 1;
    while (lower < upper) {
      size_t mid = (lower + upper + 1) / 2;
      if (pos_in_split_axis >=  params.indices[mid])
        lower = mid;
      else
        upper = mid - 1;
    }
    size_t section = upper;
    size_t section_size = params.indices[section + 1] - params.indices[section];
    LType* out_aligned = reinterpret_cast<LType*>(params.outputs[section]);
    // output offsets: leading (axes pre-split-axis), at split-axis,and blocks_last_axis
    size_t output_offset_leading = (blockIdx.x / blocks_per_leading_dim) *
                                   section_size * tail_size * last_axis_size_aligned;
    size_t output_offset_split_axis = ((blockIdx.x % blocks_per_leading_dim) / blocks_last_axis -
                                      ((params.indices[section] - params.indices[0]) * tail_size)) *
                                      last_axis_size_aligned;
    size_t output_offset = output_offset_leading + output_offset_split_axis +
                           (blockIdx.x % blocks_last_axis) * blockDim.x;
    if (threadIdx.x < last_axis_size_aligned) {
      LType input_data = in_aligned[input_offset + threadIdx.x];
      out_aligned[output_offset + threadIdx.x] = input_data;
    }
  }
}

template <typename DType>
int get_load_type_split(size_t last_axis_size,
                        bool splitting_last_axis,
                        size_t n_sections,
                        size_t* indices) {
  using namespace mshadow;
  int sections_largest_multiple = 8;
  if (splitting_last_axis) {
    for (size_t i = 0; i < n_sections; ++i) {
      size_t size_section = indices[i+1] - indices[i];
      if (size_section * sizeof(DType) % 8)
        sections_largest_multiple = std::min(sections_largest_multiple, 4);
      if (size_section * sizeof(DType) % 4)
        sections_largest_multiple = std::min(sections_largest_multiple, 2);
      if (size_section * sizeof(DType) % 2)
        sections_largest_multiple = std::min(sections_largest_multiple, 1);
    }
  }
  if (last_axis_size * sizeof(DType) % 8 == 0 && sections_largest_multiple == 8) {
    return kFloat64;
  } else if (last_axis_size * sizeof(DType) % 4 == 0 && sections_largest_multiple >= 4) {
    return kFloat32;
  } else if (last_axis_size * sizeof(DType) % 2 == 0 && sections_largest_multiple >= 2) {
    return kFloat16;
  } else {
    return kUint8;
  }
}

inline void SplitOpForwardGPU(const nnvm::NodeAttrs& attrs,
                           const OpContext& ctx,
                           const std::vector<TBlob>& inputs,
                           const std::vector<OpReqType>& req,
                           const std::vector<TBlob>& outputs) {
  using namespace mshadow;
  using namespace mshadow::expr;
  using namespace mxnet_op;
  const SplitParam& param = nnvm::get<SplitParam>(attrs.parsed);
  CHECK_EQ(inputs.size(), 1U);
  CHECK_EQ(outputs.size(), (param.sections > 0) ? param.sections : param.indices.ndim());
  const TBlob& input_data = inputs[split_enum::kData];
  int real_axis = param.axis;
  if (real_axis < 0) {
    real_axis += input_data.ndim();
  }
  size_t last_axis_size = input_data.shape_[inputs[0].ndim()-1];
  size_t split_axis_size = input_data.shape_[real_axis];
  size_t tail_size = 1;  // does not include last dim
  for (int i = real_axis + 1; i < input_data.ndim()-1; ++i) {
    tail_size *= input_data.shape_[i];
  }
  if (last_axis_size < 128) {
    // custom kernel will not be efficient with less than 128 elemnts in last axis
    SplitOpForwardImpl<gpu>(attrs, ctx, inputs, req, outputs, real_axis);
  } else {
    Stream<gpu> *s = ctx.get_stream<gpu>();
    CHECK_LT(real_axis, input_data.ndim());
    const mxnet::TShape& ishape = input_data.shape_;
    const mxnet::TShape split_pts =
      (param.sections > 0) ? GetSplitIndices(ishape, real_axis, param.sections) : param.indices;
    std::vector<size_t> indices;
    for (const auto& split_pos : split_pts) {
      indices.push_back(split_pos);
    }
    if (param.sections == 0) {
      indices.push_back(ishape[real_axis]);
    }
    size_t n_sections = indices.size() - 1;
    bool splitting_last_axis = (real_axis == inputs[0].ndim() - 1);

    for (size_t sections_processed = 0; sections_processed < n_sections;) {
      size_t remaining_sections = n_sections - sections_processed;
      MSHADOW_TYPE_SWITCH(input_data.type_flag_, DType, {
        // set parameters
        split_tensor_data<DType> params{};
        params.num_sections = std::min<size_t>(remaining_sections, params.MaxSections);
        params.inputs[0] = input_data.dptr<DType>();
        for (size_t i = 0; i < params.num_sections; ++i) {
          params.outputs[i] = outputs[sections_processed + i].dptr<DType>();
          params.indices[i] = indices[sections_processed + i];
        }
        params.indices[params.num_sections] = indices[sections_processed + params.num_sections];
        // load type: we need to check that last axis size is multiple of ltype
        // and if splitting_last_axis, all section sizes as well
        int ltype = get_load_type_split<DType>(last_axis_size, splitting_last_axis,
                                               params.num_sections, params.indices);
        MXNET_LOAD_TYPE_SWITCH(ltype, LType, {
          CHECK_LE(sizeof(DType), sizeof(LType));
          const size_t entries_per_load = sizeof(LType) / sizeof(DType);
          size_t block_size = 32;
          size_t max_threads_block = 256;
          size_t last_axis_elements = entries_per_load > 0 ? (last_axis_size / entries_per_load): 0;
          if (splitting_last_axis) {
            // may not be possible to include whole axis if too many sections
            last_axis_elements = entries_per_load > 0 ?
                ((params.indices[params.num_sections] - params.indices[0]) / entries_per_load): 0;
          }
          while (block_size < last_axis_elements && (block_size < max_threads_block)) {
            block_size += 32;
          }
          size_t blocks_last_axis = (last_axis_elements + block_size - 1) / block_size;
          size_t n_blocks = blocks_last_axis;
          for (int i = 0 ; i < input_data.ndim() - 1; ++i) {
            if (i == real_axis) {
              // may not be possible to include all sections if too many
              n_blocks *= (params.indices[params.num_sections] - params.indices[0]);
            } else {
              n_blocks *= input_data.shape_[i];
            }
          }
          if (splitting_last_axis) {
            split_tensor_kernel<true, LType><<<n_blocks, block_size, 0, s->stream_>>>
              (input_data.Size(), params, split_axis_size, tail_size,
               last_axis_size, blocks_last_axis);
          } else {
            split_tensor_kernel<false, LType><<<n_blocks, block_size, 0, s->stream_>>>
              (input_data.Size(), params, split_axis_size, tail_size,
               last_axis_size, blocks_last_axis);
          }
        });
        sections_processed += params.num_sections;
      });
    }
  }
}

NNVM_REGISTER_OP(Reshape)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::IdentityCompute<gpu>);

NNVM_REGISTER_OP(Flatten)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::IdentityCompute<gpu>);

NNVM_REGISTER_OP(transpose)
.set_attr<FCompute>("FCompute<gpu>", Transpose<gpu>);

NNVM_REGISTER_OP(expand_dims)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::IdentityCompute<gpu>);

NNVM_REGISTER_OP(slice)
.set_attr<FCompute>("FCompute<gpu>", SliceOpForward<gpu>)
.set_attr<FComputeEx>("FComputeEx<gpu>", SliceEx<gpu>);

NNVM_REGISTER_OP(_backward_slice)
.set_attr<FCompute>("FCompute<gpu>", SliceOpBackward<gpu>);

NNVM_REGISTER_OP(_slice_assign)
.set_attr<FCompute>("FCompute<gpu>", SliceAssignOpForward<gpu>);

NNVM_REGISTER_OP(_slice_assign_scalar)
.set_attr<FCompute>("FCompute<gpu>", SliceAssignScalarOpForward<gpu>);

NNVM_REGISTER_OP(slice_axis)
.set_attr<FCompute>("FCompute<gpu>", SliceAxis<gpu>);

NNVM_REGISTER_OP(_backward_slice_axis)
.set_attr<FCompute>("FCompute<gpu>", SliceAxisGrad_<gpu>);

NNVM_REGISTER_OP(slice_like)
.set_attr<FCompute>("FCompute<gpu>", SliceLikeForward<gpu>);

NNVM_REGISTER_OP(_backward_slice_like)
.set_attr<FCompute>("FCompute<gpu>", SliceLikeBackward<gpu>);

NNVM_REGISTER_OP(clip)
.set_attr<FCompute>("FCompute<gpu>", Clip<gpu>)
.set_attr<FComputeEx>("FComputeEx<gpu>", ClipEx<gpu>);

NNVM_REGISTER_OP(_backward_clip)
.set_attr<FCompute>("FCompute<gpu>", ClipGrad_<gpu>);

NNVM_REGISTER_OP(repeat)
.set_attr<FCompute>("FCompute<gpu>", RepeatOpForward<gpu>);

NNVM_REGISTER_OP(_backward_repeat)
.set_attr<FCompute>("FCompute<gpu>", RepeatOpBackward<gpu>);

NNVM_REGISTER_OP(tile)
.set_attr<FCompute>("FCompute<gpu>", TileOpForward<gpu>);

NNVM_REGISTER_OP(_backward_tile)
.set_attr<FCompute>("FCompute<gpu>", TileOpBackward<gpu>);

NNVM_REGISTER_OP(reverse)
.set_attr<FCompute>("FCompute<gpu>", ReverseOpForward<gpu>);

NNVM_REGISTER_OP(_backward_reverse)
.set_attr<FCompute>("FCompute<gpu>", ReverseOpForward<gpu>);

NNVM_REGISTER_OP(stack)
.set_attr<FCompute>("FCompute<gpu>", StackOpForward<gpu>);

NNVM_REGISTER_OP(_backward_stack)
.set_attr<FCompute>("FCompute<gpu>", StackOpBackward<gpu>);

NNVM_REGISTER_OP(squeeze)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::IdentityCompute<gpu>);

NNVM_REGISTER_OP(_backward_squeeze)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::IdentityCompute<gpu>);

NNVM_REGISTER_OP(depth_to_space)
.set_attr<FCompute>("FCompute<gpu>", DepthToSpaceOpForward<gpu>);

NNVM_REGISTER_OP(space_to_depth)
.set_attr<FCompute>("FCompute<gpu>", SpaceToDepthOpForward<gpu>);

NNVM_REGISTER_OP(_split_v2)
.set_attr<FCompute>("FCompute<gpu>", SplitOpForwardGPU);

NNVM_REGISTER_OP(_split_v2_backward)
.set_attr<FCompute>("FCompute<gpu>", SplitOpBackward<gpu>);

}  // namespace op
}  // namespace mxnet
