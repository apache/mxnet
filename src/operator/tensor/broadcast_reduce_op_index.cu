/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 *  Copyright (c) 2016 by Contributors
 * \file broadcast_reduce_op_index.cu
 * \brief GPU Implementation of broadcast and reduce functions based on index.
 */
#include "./broadcast_reduce_op.h"

namespace mxnet {
namespace op {
NNVM_REGISTER_OP(argmax)
.set_attr<FCompute>("FCompute<gpu>", SearchAxisCompute<gpu, mshadow::red::maximum>);

NNVM_REGISTER_OP(argmin)
.set_attr<FCompute>("FCompute<gpu>", SearchAxisCompute<gpu, mshadow::red::minimum>);

// Legacy support
NNVM_REGISTER_OP(argmax_channel)
.set_attr<FCompute>("FCompute<gpu>", SearchAxisCompute<gpu, mshadow::red::maximum>);

NNVM_REGISTER_OP(pick)
.set_attr<FCompute>("FCompute<gpu>", PickOpForward<gpu>);


NNVM_REGISTER_OP(_backward_pick)
.set_attr<FCompute>("FCompute<gpu>", PickOpBackward<gpu>);

}  // namespace op
}  // namespace mxnet
