#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file elemwise_binary_op_extended.cu
 * \brief GPU Implementation of binary function.
 */
#include "./elemwise_unary_op.h"
#include "./elemwise_binary_op.h"

namespace mxnet {
namespace op {
NNVM_REGISTER_OP(_power)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<gpu, mshadow_op::power>);

NNVM_REGISTER_OP(_backward_power)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::BackwardUseIn<gpu,
  mshadow_op::power_grad, mshadow_op::power_rgrad>);

NNVM_REGISTER_OP(_maximum)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<gpu, mshadow_op::maximum>);

NNVM_REGISTER_OP(_backward_maximum)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::BackwardUseIn<gpu, mshadow_op::ge,
  mshadow_op::lt>);

NNVM_REGISTER_OP(_minimum)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<gpu, mshadow_op::minimum>);

NNVM_REGISTER_OP(_backward_minimum)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::BackwardUseIn<gpu, mshadow_op::le,
  mshadow_op::gt>);

NNVM_REGISTER_OP(_hypot)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<gpu, mshadow_op::hypot>);

NNVM_REGISTER_OP(_backward_hypot)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::BackwardUseIn<gpu,
  mshadow_op::hypot_grad_left, mshadow_op::hypot_grad_right>);

}  // namespace op
}  // namespace mxnet
