#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file elemwise_binary_scalar_op_extended.cu
 * \brief GPU Implementation of extended binary scalar functions.
 */
#include "./elemwise_unary_op.h"
#include "./elemwise_binary_op.h"
#include "./elemwise_binary_scalar_op.h"

namespace mxnet {
namespace op {
NNVM_REGISTER_OP(_maximum_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::maximum>);

NNVM_REGISTER_OP(_backward_maximum_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Backward<gpu, mshadow_op::ge>);

NNVM_REGISTER_OP(_minimum_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::minimum>);

NNVM_REGISTER_OP(_backward_minimum_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Backward<gpu, mshadow_op::le>);

NNVM_REGISTER_OP(_power_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::power>);

NNVM_REGISTER_OP(_backward_power_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Backward<
  gpu, mshadow_op::power_grad>);

NNVM_REGISTER_OP(_rpower_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::rpower>);

NNVM_REGISTER_OP(_backward_rpower_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Backward<
  gpu, mshadow_op::rpower_grad>);

NNVM_REGISTER_OP(_hypot_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::hypot>);

NNVM_REGISTER_OP(_backward_hypot_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Backward<
  gpu, mshadow_op::hypot_grad_left>);

NNVM_REGISTER_OP(smooth_l1)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<
  gpu, mshadow_op::smooth_l1_loss>);

NNVM_REGISTER_OP(_backward_smooth_l1)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Backward<
  gpu, mshadow_op::smooth_l1_gradient>);

}  // namespace op
}  // namespace mxnet
