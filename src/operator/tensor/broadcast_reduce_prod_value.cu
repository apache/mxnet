/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file broadcast_reduce_prod_value.cu
 * \brief GPU Implementation of broadcast and reduce prod functions based on value.
 */
#include "./broadcast_reduce_op.h"

namespace mxnet {
namespace op {

NNVM_REGISTER_OP(prod).set_attr<FCompute>("FCompute<gpu>",
                                          ReduceAxesRTCCompute<ReduceAxesParam, 0>{"identity",
                                                                                   "red::product{}",
                                                                                   false});

NNVM_REGISTER_OP(_backward_prod)
    .set_attr<FCompute>("FCompute<gpu>", ReduceAxesBackwardUseInOut<gpu, mshadow_op::rdiv>);

NNVM_REGISTER_OP(nanprod).set_attr<FCompute>(
    "FCompute<gpu>",
    ReduceAxesRTCCompute<ReduceAxesParam, 0>{"identity", "red::nanprod{}", false});

NNVM_REGISTER_OP(_backward_nanprod)
    .set_attr<FCompute>("FCompute<gpu>", ReduceAxesBackwardUseInOut<gpu, mshadow_op::nanprod_grad>);

}  // namespace op
}  // namespace mxnet
