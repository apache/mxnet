#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file broadcast_reduce_minmax_value.cu
 * \brief GPU Implementation of broadcast and reduce min and max functions based on value.
 */
#include "./broadcast_reduce_op.h"

namespace mxnet {
namespace op {

NNVM_REGISTER_OP(max).set_attr<FCompute>("FCompute<gpu>",
                                         ReduceAxesRTCCompute<ReduceAxesParam, 0>{"identity",
                                                                                  "red::maximum{}",
                                                                                  false});

NNVM_REGISTER_OP(_backward_max)
    .set_attr<FCompute>("FCompute<gpu>", ReduceAxesBackwardUseInOut<gpu, mshadow_op::eq>);

NNVM_REGISTER_OP(min).set_attr<FCompute>("FCompute<gpu>",
                                         ReduceAxesRTCCompute<ReduceAxesParam, 0>{"identity",
                                                                                  "red::minimum{}",
                                                                                  false});

NNVM_REGISTER_OP(_backward_min)
    .set_attr<FCompute>("FCompute<gpu>", ReduceAxesBackwardUseInOut<gpu, mshadow_op::eq>);

}  // namespace op
}  // namespace mxnet
