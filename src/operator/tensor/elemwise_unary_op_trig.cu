#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file elemwise_unary_op_trig.cu
 * \brief GPU Implementation of unary trigonometric function.
 */
#include "./elemwise_binary_op.h"
#include "./elemwise_unary_op.h"

namespace mxnet {
namespace op {

// sin
NNVM_REGISTER_OP(sin)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"sin"})
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryRTCCompute{"sin"});

NNVM_REGISTER_OP(_backward_sin)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_sin"});

// cos
NNVM_REGISTER_OP(cos)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"cos"});

NNVM_REGISTER_OP(_backward_cos)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_cos"});

// tan
NNVM_REGISTER_OP(tan)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"tan"})
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryRTCCompute{"tan"});

NNVM_REGISTER_OP(_backward_tan)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_tan"});

// arcsin
NNVM_REGISTER_OP(arcsin)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"arcsin"})
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryRTCCompute{"arcsin"});

NNVM_REGISTER_OP(_backward_arcsin)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_arcsin"});

// arccos
NNVM_REGISTER_OP(arccos)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"arccos"});

NNVM_REGISTER_OP(_backward_arccos)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_arccos"});

// arctan
NNVM_REGISTER_OP(arctan)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"arctan"})
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryRTCCompute{"arctan"});

NNVM_REGISTER_OP(_backward_arctan)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_arctan"});

// degrees
NNVM_REGISTER_OP(degrees)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"degrees"})
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryRTCCompute{"degrees"});

NNVM_REGISTER_OP(_backward_degrees)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_degrees"});

// radians
NNVM_REGISTER_OP(radians)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"radians"})
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryRTCCompute{"radians"});

NNVM_REGISTER_OP(_backward_radians)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_radians"});

// cosh
NNVM_REGISTER_OP(cosh)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"cosh"});

NNVM_REGISTER_OP(_backward_cosh)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_cosh"});

// sinh
NNVM_REGISTER_OP(sinh)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"sinh"})
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryRTCCompute{"sinh"});

NNVM_REGISTER_OP(_backward_sinh)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_sinh"});

// tanh
NNVM_REGISTER_OP(tanh)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"tanh"})
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryRTCCompute{"tanh"});

NNVM_REGISTER_OP(_backward_tanh)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_tanh"});

// arcsinh
NNVM_REGISTER_OP(arcsinh)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"arcsinh"})
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryRTCCompute{"arcsinh"});

NNVM_REGISTER_OP(_backward_arcsinh)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_arcsinh"});

// arccosh
NNVM_REGISTER_OP(arccosh)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"arccosh"});

NNVM_REGISTER_OP(_backward_arccosh)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_arccosh"});

// arctanh
NNVM_REGISTER_OP(arctanh)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"arctanh"})
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryRTCCompute{"arctanh"});

NNVM_REGISTER_OP(_backward_arctanh)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_arctanh"});

}  // namespace op
}  // namespace mxnet
