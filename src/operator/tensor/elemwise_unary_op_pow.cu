#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file elemwise_unary_op_pow.cu
 * \brief GPU Implementation of power (x^k for fixed k) functions.
 */
#include "./elemwise_binary_op.h"
#include "./elemwise_unary_op.h"

namespace mxnet {
namespace op {

// reciprocal
NNVM_REGISTER_OP(reciprocal)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"reciprocal"});

NNVM_REGISTER_OP(_backward_reciprocal)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_reciprocal"});

// square
NNVM_REGISTER_OP(square)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"square"})
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryRTCCompute{"square"});

NNVM_REGISTER_OP(_backward_square)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_square"});

// sqrt
NNVM_REGISTER_OP(sqrt)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"sqrt"})
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryRTCCompute{"sqrt"});

NNVM_REGISTER_OP(_backward_sqrt)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_sqrt"});

// rsqrt
NNVM_REGISTER_OP(rsqrt)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"rsqrt"});

NNVM_REGISTER_OP(_backward_rsqrt)
.set_attr<FCompute>("FCompute<gpu>",
  ElemwiseBinaryRTCCompute{"backward_rsqrt"});

// cbrt
NNVM_REGISTER_OP(cbrt)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"cbrt"})
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryRTCCompute{"cbrt"});


NNVM_REGISTER_OP(_backward_cbrt)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_cbrt"});

// rcbrt
NNVM_REGISTER_OP(rcbrt)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"rcbrt"});

NNVM_REGISTER_OP(_backward_rcbrt)
.set_attr<FCompute>("FCompute<gpu>",
  ElemwiseBinaryRTCCompute{"backward_rcbrt"});

}  // namespace op
}  // namespace mxnet
