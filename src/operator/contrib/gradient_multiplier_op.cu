/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2018 by Contributors
 * \file gradient_multiplier_op.cu
 * \brief
 * \author Istvan Fehervari
*/
#include "../tensor/elemwise_unary_op.h"
#include "../tensor/elemwise_binary_scalar_op.h"

namespace mxnet {
namespace op {

NNVM_REGISTER_OP(_contrib_gradientmultiplier)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::IdentityComputeEx<gpu>)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::IdentityCompute<gpu>);

NNVM_REGISTER_OP(_contrib_backward_gradientmultiplier)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarRTCCompute{"mul"})
.set_attr<FComputeEx>("(FComputeEx<gpu>", BinaryScalarRTCCompute{"mul"});

}  // namespace op
}  // namespace mxnet
