/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */
#include <mshadow/tensor.h>
#include "./index_array-inl.h"

namespace mxnet {
namespace op {

using namespace mshadow::cuda;

void IndexArrayForwardGPU(const nnvm::NodeAttrs &attrs,
                          const OpContext &ctx,
                          const std::vector<TBlob> &inputs,
                          const std::vector<OpReqType> &req,
                          const std::vector<TBlob> &outputs) {
  using namespace mshadow;
  CHECK_EQ(inputs.size(), 1U);
  CHECK_EQ(outputs.size(), 1U);
  CHECK_EQ(req.size(), 1U);
  const TBlob& in_data = inputs[0];
  const TBlob& out_data = outputs[0];

  const IndexArrayParam& param = nnvm::get<IndexArrayParam>(attrs.parsed);

  const TShape inshape = in_data.shape_;
  const int ndim = inshape.ndim();

  Stream<gpu> *s = ctx.get_stream<gpu>();
  hipStream_t stream = Stream<gpu>::GetStream(s);

  using namespace mxnet_op;

  if (param.axes.has_value()) {
    const mxnet::Tuple<int>& axes = param.axes.value();
    const int naxes = axes.ndim();

    std::vector<int64_t> index_products = IndexArrayComputeIndexProducts(inshape);

    std::vector<int64_t> cpu_workspace(2 * naxes);
    IndexArrayBuildSelectedAxesWorkspace(axes, index_products, cpu_workspace.data(), ndim);

    Tensor<gpu, 1, int64_t> workspace =
        ctx.requested[0].get_space_typed<gpu, 1, int64_t>(Shape1(2 * naxes), s);

    CUDA_CALL(hipMemcpyAsync(workspace.dptr_, cpu_workspace.data(), sizeof(int64_t) * (2 * naxes),
                              hipMemcpyHostToDevice, stream));

    MXNET_ASSIGN_REQ_SWITCH(req[0], req_type, {
      Kernel<IndexArrayKernel<req_type>, gpu>::Launch(s, in_data.Size(),
          out_data.dptr<int64_t>(), naxes, workspace.dptr_);
    });
  } else {
    Tensor<gpu, 1, dim_t> workspace =
        ctx.requested[0].get_space_typed<gpu, 1, dim_t>(Shape1(ndim), s);

    CUDA_CALL(hipMemcpyAsync(workspace.dptr_, inshape.data(), sizeof(dim_t) * ndim,
                              hipMemcpyHostToDevice, stream));

    MXNET_ASSIGN_REQ_SWITCH(req[0], req_type, {
      Kernel<IndexArrayDefaultKernel<req_type>, gpu>::Launch(s, in_data.Size(),
          out_data.dptr<int64_t>(), ndim, workspace.dptr_);
    });
  }
}

NNVM_REGISTER_OP(_contrib_index_array)
.set_attr<FCompute>("FCompute<gpu>", IndexArrayForwardGPU);

}  // namespace op
}  // namespace mxnet
