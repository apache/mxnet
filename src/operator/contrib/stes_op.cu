#include "hip/hip_runtime.h"
/*
  * Licensed to the Apache Software Foundation (ASF) under one
  * or more contributor license agreements.  See the NOTICE file
  * distributed with this work for additional information
  * regarding copyright ownership.  The ASF licenses this file
  * to you under the Apache License, Version 2.0 (the
  * "License"); you may not use this file except in compliance
  * with the License.  You may obtain a copy of the License at
  *
  *   http://www.apache.org/licenses/LICENSE-2.0
  *
  * Unless required by applicable law or agreed to in writing,
  * software distributed under the License is distributed on an
  * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
  * KIND, either express or implied.  See the License for the
  * specific language governing permissions and limitations
  * under the License.
  */

  /*!
  *  Copyright (c) 2019 by Contributors
  * \file stes_op.cu
  * \Straight-through-estimators round and sign operators.
  * \author Itay Golan
  */

#include "stes_op.h"

namespace mxnet {
namespace op {

// Round STE
NNVM_REGISTER_OP(_contrib_round_ste)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"round"})
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryRTCCompute{"round"});

// Sign STE
NNVM_REGISTER_OP(_contrib_sign_ste)
.set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{"sign"})
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryRTCCompute{"sign"});

}  // namespace op
}  // namespace mxnet
