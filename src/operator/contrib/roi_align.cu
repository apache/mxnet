#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */
/*!
 * Copyright (c) 2018 by Contributors
 * \file roi_align.cu
 * \brief roi align operator
 * \author Hang Zhang, Shesung
 * Adapted from Caffe2
*/
#include "./roi_align-inl.h"
#include "../mxnet_op.h"


namespace mxnet {
namespace op {

using namespace mshadow::cuda;

// The maximum number of blocks to use in the default kernel call.
constexpr int ROI_MAXIMUM_NUM_BLOCKS = 4096;

/**
 * @brief Compute the number of blocks needed to run N threads.
 */
inline int ROI_GET_BLOCKS(const int N) {
  return std::max(
      std::min(
          (N + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock,
          ROI_MAXIMUM_NUM_BLOCKS),
      // Use at least 1 block, since CUDA does not allow empty block
      1);
}


template <typename T>
__device__ T bilinear_interpolate(
    const T* bottom_data,
    const int height,
    const int width,
    T y,
    T x,
    const int index /* index for debug only*/) {
  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    // empty
    return 0;
  }

  if (y <= 0) {
    y = 0;
  }
  if (x <= 0) {
    x = 0;
  }

  int y_low = static_cast<int>(y);
  int x_low = static_cast<int>(x);
  int y_high;
  int x_high;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T)y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T)x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;
  // do bilinear interpolation
  T v1 = bottom_data[y_low * width + x_low];
  T v2 = bottom_data[y_low * width + x_high];
  T v3 = bottom_data[y_high * width + x_low];
  T v4 = bottom_data[y_high * width + x_high];
  T w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  return val;
}

template <typename T>
__global__ void RoIAlignForwardKernel(
    const int nthreads,
    const T* bottom_data,
    const T spatial_scale,
    const bool position_sensitive,
    const bool continuous_coordinate,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const int sampling_ratio,
    const T* bottom_rois,
    T* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];

    if (roi_batch_ind < 0) {
      top_data[index] = 0.;
      continue;
    }

    // Do not using rounding; this implementation detail is critical
    T roi_offset = continuous_coordinate ? static_cast<T>(0.5) : static_cast<T>(0);
    T roi_start_w = offset_bottom_rois[1] * spatial_scale - roi_offset;
    T roi_start_h = offset_bottom_rois[2] * spatial_scale - roi_offset;
    T roi_end_w = offset_bottom_rois[3] * spatial_scale - roi_offset;
    T roi_end_h = offset_bottom_rois[4] * spatial_scale - roi_offset;

    T roi_width = roi_end_w - roi_start_w;
    T roi_height = roi_end_h - roi_start_h;
    if (!continuous_coordinate) {  // backward compatiblity
      // Force malformed ROIs to be 1x1
      roi_width = max(roi_width, (T)1.);
      roi_height = max(roi_height, (T)1.);
    }
    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    int c_unpooled = c;
    int channels_unpooled = channels;
    if (position_sensitive) {
      c_unpooled = c * pooled_height * pooled_width + ph * pooled_width + pw;
      channels_unpooled = channels * pooled_height * pooled_width;
    }
    const T* offset_bottom_data =
        bottom_data + (roi_batch_ind * channels_unpooled + c_unpooled)
        * height * width;

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h = (sampling_ratio > 0)
        ? sampling_ratio
        : ceil(roi_height / pooled_height);  // e.g., = 2
    int roi_bin_grid_w =
        (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

    // We do average (integral) pooling inside a bin
    const T count = roi_bin_grid_h * roi_bin_grid_w;  // e.g. = 4

    T output_val = 0.;
    for (int iy = 0; iy < roi_bin_grid_h; iy++) {  // e.g., iy = 0, 1
      const T y = roi_start_h + ph * bin_size_h +
          static_cast<T>(iy + .5f) * bin_size_h /
              static_cast<T>(roi_bin_grid_h);  // e.g., 0.5, 1.5
      for (int ix = 0; ix < roi_bin_grid_w; ix++) {
        const T x = roi_start_w + pw * bin_size_w +
            static_cast<T>(ix + .5f) * bin_size_w /
                static_cast<T>(roi_bin_grid_w);

        T val = bilinear_interpolate(
            offset_bottom_data, height, width, y, x, index);
        output_val += val;
      }
    }
    output_val /= count;

    top_data[index] = output_val;
  }
}


template <typename T>
__device__ void bilinear_interpolate_gradient(
    const int height,
    const int width,
    T y,
    T x,
    T* w1,
    T* w2,
    T* w3,
    T* w4,
    int* x_low,
    int* x_high,
    int* y_low,
    int* y_high,
    const int /*index*/ /* index for debug only*/) {
  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    // empty
    *w1 = *w2 = *w3 = *w4 = 0.;
    *x_low = *x_high = *y_low = *y_high = -1;
    return;
  }

  if (y <= 0) {
    y = 0;
  }
  if (x <= 0) {
    x = 0;
  }

  *y_low = static_cast<int>(y);
  *x_low = static_cast<int>(x);

  if (*y_low >= height - 1) {
    *y_high = *y_low = height - 1;
    y = (T)*y_low;
  } else {
    *y_high = *y_low + 1;
  }

  if (*x_low >= width - 1) {
    *x_high = *x_low = width - 1;
    x = (T)*x_low;
  } else {
    *x_high = *x_low + 1;
  }

  T ly = y - *y_low;
  T lx = x - *x_low;
  T hy = 1. - ly, hx = 1. - lx;

  *w1 = hy * hx, *w2 = hy * lx, *w3 = ly * hx, *w4 = ly * lx;

  return;
}

template <typename T>
__global__ void RoIAlignBackwardKernel(
    const int nthreads,
    const T* top_diff,
    const int num_rois,
    const T spatial_scale,
    const bool position_sensitive,
    const bool continuous_coordinate,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const int sampling_ratio,
    T* bottom_diff,
    const T* bottom_rois) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];
    if (roi_batch_ind < 0) continue;

    // Do not using rounding; this implementation detail is critical
    T roi_offset = continuous_coordinate ? static_cast<T>(0.5) : static_cast<T>(0);
    T roi_start_w = offset_bottom_rois[1] * spatial_scale - roi_offset;
    T roi_start_h = offset_bottom_rois[2] * spatial_scale - roi_offset;
    T roi_end_w = offset_bottom_rois[3] * spatial_scale - roi_offset;
    T roi_end_h = offset_bottom_rois[4] * spatial_scale - roi_offset;

    T roi_width = roi_end_w - roi_start_w;
    T roi_height = roi_end_h - roi_start_h;
    if (!continuous_coordinate) {  // backward compatiblity
      // Force malformed ROIs to be 1x1
      roi_width = max(roi_width, (T)1.);
      roi_height = max(roi_height, (T)1.);
    }
    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    int c_unpooled = c;
    int channels_unpooled = channels;
    if (position_sensitive) {
      c_unpooled = c * pooled_height * pooled_width + ph * pooled_width + pw;
      channels_unpooled = channels * pooled_height * pooled_width;
    }
    T* offset_bottom_diff =
        bottom_diff + (roi_batch_ind * channels_unpooled + c_unpooled)
        * height * width;

    int top_offset = (n * channels + c) * pooled_height * pooled_width;
    const T* offset_top_diff = top_diff + top_offset;
    const T top_diff_this_bin = offset_top_diff[ph * pooled_width + pw];

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h = (sampling_ratio > 0)
        ? sampling_ratio
        : ceil(roi_height / pooled_height);  // e.g., = 2
    int roi_bin_grid_w =
        (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

    // We do average (integral) pooling inside a bin
    const T count = roi_bin_grid_h * roi_bin_grid_w;  // e.g. = 4

    for (int iy = 0; iy < roi_bin_grid_h; iy++) {  // e.g., iy = 0, 1
      const T y = roi_start_h + ph * bin_size_h +
          static_cast<T>(iy + .5f) * bin_size_h /
              static_cast<T>(roi_bin_grid_h);  // e.g., 0.5, 1.5
      for (int ix = 0; ix < roi_bin_grid_w; ix++) {
        const T x = roi_start_w + pw * bin_size_w +
            static_cast<T>(ix + .5f) * bin_size_w /
                static_cast<T>(roi_bin_grid_w);

        T w1, w2, w3, w4;
        int x_low, x_high, y_low, y_high;

        bilinear_interpolate_gradient(
            height,
            width,
            y,
            x,
            &w1,
            &w2,
            &w3,
            &w4,
            &x_low,
            &x_high,
            &y_low,
            &y_high,
            index);

        T g1 = top_diff_this_bin * w1 / count;
        T g2 = top_diff_this_bin * w2 / count;
        T g3 = top_diff_this_bin * w3 / count;
        T g4 = top_diff_this_bin * w4 / count;

        if (x_low >= 0 && x_high >= 0 && y_low >= 0 && y_high >= 0) {
          atomicAdd(
              offset_bottom_diff + y_low * width + x_low, static_cast<T>(g1));
          atomicAdd(
              offset_bottom_diff + y_low * width + x_high, static_cast<T>(g2));
          atomicAdd(
              offset_bottom_diff + y_high * width + x_low, static_cast<T>(g3));
          atomicAdd(
              offset_bottom_diff + y_high * width + x_high, static_cast<T>(g4));
        }  // if
      }  // ix
    }  // iy
  }  // CUDA_KERNEL_LOOP
}  // RoIAlignBackward

template<typename xpu>
void ROIAlignForwardCompute(const nnvm::NodeAttrs& attrs,
                            const OpContext& ctx,
                            const std::vector<TBlob>& in_data,
                            const std::vector<OpReqType>& req,
                            const std::vector<TBlob>& out_data) {
  using namespace mshadow;
  size_t expected_in = 2;
  size_t expected_out = 1;
  CHECK_EQ(in_data.size(), expected_in);
  CHECK_EQ(out_data.size(), expected_out);
  CHECK_EQ(out_data[roialign::kOut].shape_[0], in_data[roialign::kBox].shape_[0]);

  const ROIAlignParam param = nnvm::get<ROIAlignParam>(attrs.parsed);

  const int count = out_data[roialign::kOut].Size();
  const int num_rois = in_data[roialign::kBox].size(0);
  const int channels = out_data[roialign::kOut].size(1);  // channels of pooled output
  const int height = in_data[roialign::kData].size(2);
  const int width = in_data[roialign::kData].size(3);
  const int pooled_height = out_data[roialign::kOut].size(2);
  const int pooled_width = out_data[roialign::kOut].size(3);

  Stream<gpu> *s = ctx.get_stream<gpu>();
  hipStream_t stream = mshadow::Stream<gpu>::GetStream(s);
  MSHADOW_REAL_TYPE_SWITCH(in_data[0].type_flag_, DType, {
    const DType *bottom_data = in_data[roialign::kData].dptr<DType>();
    const DType *bottom_rois = in_data[roialign::kBox].dptr<DType>();
    DType *top_data = out_data[roialign::kOut].dptr<DType>();
    RoIAlignForwardKernel<DType>
      <<<ROI_GET_BLOCKS(count),
         kMaxThreadsPerBlock,
         0,
         stream>>>(
          count,
          bottom_data,
          param.spatial_scale,
          param.position_sensitive,
          param.aligned,
          channels,
          height,
          width,
          pooled_height,
          pooled_width,
          param.sample_ratio,
          bottom_rois,
          top_data);
  })
}


template<typename xpu>
void ROIAlignBackwardCompute(const nnvm::NodeAttrs& attrs,
                             const OpContext& ctx,
                             const std::vector<TBlob>& inputs,
                             const std::vector<OpReqType>& req,
                             const std::vector<TBlob>& outputs) {
  using namespace mshadow;

  CHECK_EQ(inputs.size(), 2);
  CHECK_EQ(outputs.size(), 2);
  // the order here relates to the order in ROIAlignGrad
  std::vector<TBlob> out_grad(1, inputs[0]);
  std::vector<TBlob> in_data(1, inputs[1]);
  // std::vector<TBlob> out_data(1, inputs[2]);

  CHECK_EQ(out_grad[0].shape_[0], in_data[0].shape_[0]);
  CHECK_NE(req[0], kWriteInplace) <<
    "ROIAlign: Backward doesn't support kWriteInplace.";
  CHECK_NE(req[1], kWriteInplace) <<
    "ROIAlign: Backward doesn't support kWriteInplace.";

  const ROIAlignParam param = nnvm::get<ROIAlignParam>(attrs.parsed);

  const int count = out_grad[0].Size();
  const int num_rois = in_data[0].size(0);
  const int channels = out_grad[0].size(1);  // channels of pooled output
  const int height = outputs[0].size(2);
  const int width = outputs[0].size(3);
  const int pooled_height = out_grad[0].size(2);
  const int pooled_width = out_grad[0].size(3);

  Stream<gpu> *s = ctx.get_stream<gpu>();
  hipStream_t stream = mshadow::Stream<gpu>::GetStream(s);

  // assume all the data and gradient have the same type
  MSHADOW_REAL_TYPE_SWITCH(out_grad[0].type_flag_, DType, {
    const DType *top_diff = out_grad[0].dptr<DType>();
    const DType *bottom_rois = in_data[0].dptr<DType>();
    DType *grad_in = outputs[0].dptr<DType>();

    if (kWriteTo == req[roialign::kBox]) {
      Fill<false>(s, outputs[1], kWriteTo, static_cast<DType>(0));
    }
    if (kNullOp == req[roialign::kData]) return;
    if (kWriteTo == req[roialign::kData]) {
      Fill<false>(s, outputs[0], kWriteTo, static_cast<DType>(0));
    }
    RoIAlignBackwardKernel<DType>
    <<<ROI_GET_BLOCKS(count),
       kMaxThreadsPerBlock,
       0,
       stream>>>(
        count,
        top_diff,
        num_rois,
        param.spatial_scale,
        param.position_sensitive,
        param.aligned,
        channels,
        height,
        width,
        pooled_height,
        pooled_width,
        param.sample_ratio,
        grad_in,
        bottom_rois);
  })
}


NNVM_REGISTER_OP(_contrib_ROIAlign)
.set_attr<FCompute>("FCompute<gpu>", ROIAlignForwardCompute<gpu>);

NNVM_REGISTER_OP(_backward_ROIAlign)
.set_attr<FCompute>("FCompute<gpu>", ROIAlignBackwardCompute<gpu>);

}  // namespace op
}  // namespace mxnet
