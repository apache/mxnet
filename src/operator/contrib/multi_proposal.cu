#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2015 by Contributors
 * Copyright (c) 2017 Microsoft
 * Licensed under The Apache-2.0 License [see LICENSE for details]
 * \file multi_proposal.cu
 * \brief MultiProposal Operator
 * \author Shaoqing Ren, Xizhou Zhu, Jian Guo
*/
#include <dmlc/logging.h>
#include <dmlc/parameter.h>
#include <mxnet/operator.h>
#include <mshadow/tensor.h>
#include <mshadow/cuda/reduce.cuh>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>

#include <map>
#include <vector>
#include <string>
#include <utility>
#include <ctime>
#include <iostream>

#include "../operator_common.h"
#include "../mshadow_op.h"
#include "./multi_proposal-inl.h"

#define DIVUP(m, n) ((m) / (n) + ((m) % (n) > 0))

#define FRCNN_CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    CHECK_EQ(error, hipSuccess) << " " << hipGetErrorString(error); \
} while (0)

namespace mshadow {
namespace cuda {
namespace multi_proposal {

// scores are (b, 2 * anchor, h, w)
// workspace_proposals are (b, h * w * anchor, 5)
// w defines "x" and h defines "y"
// count should be total anchors numbers, h * w * anchors
template<typename Dtype>
__global__ void ProposalGridKernel(const int count,
                                   const int num_anchors,
                                   const int height,
                                   const int width,
                                   const int feature_stride,
                                   const Dtype* scores,
                                   Dtype* workspace_proposals) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < count;
       index += blockDim.x * gridDim.x) {
    int a = index % num_anchors;
    int w = (index / num_anchors) % width;
    int h = (index / num_anchors / width) % height;
    int b = index / num_anchors / width / height;

    workspace_proposals[index * 5 + 0] = workspace_proposals[a * 5 + 0] + w * feature_stride;
    workspace_proposals[index * 5 + 1] = workspace_proposals[a * 5 + 1] + h * feature_stride;
    workspace_proposals[index * 5 + 2] = workspace_proposals[a * 5 + 2] + w * feature_stride;
    workspace_proposals[index * 5 + 3] = workspace_proposals[a * 5 + 3] + h * feature_stride;
    workspace_proposals[index * 5 + 4] =
        scores[((b * (2 * num_anchors) + a + num_anchors) * height + h) * width + w];
  }
}

// boxes are (b, h * w * anchor, 5)
// deltas are (b, 4 * anchor, h, w)
// out_pred_boxes are (b, h * w * anchor, 5)
// count should be total anchors numbers, b * h * w * anchors
// in-place write: boxes and out_pred_boxes are the same location
template<typename Dtype>
__global__ void BBoxPredKernel(const int count,
                               const int num_anchors,
                               const int feat_height,
                               const int feat_width,
                               const int feature_stride,
                               const Dtype* im_infos,
                               const Dtype* boxes,
                               const Dtype* deltas,
                               Dtype* out_pred_boxes) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < count;
       index += blockDim.x * gridDim.x) {
    int a = index % num_anchors;
    int w = (index / num_anchors) % feat_width;
    int h = (index / num_anchors / feat_width) % feat_height;
    int b = index / num_anchors / feat_width / feat_height;

    float im_height = im_infos[b * 3];
    float im_width = im_infos[b * 3 + 1];
    int real_height = static_cast<int>(im_height / feature_stride);
    int real_width = static_cast<int>(im_width / feature_stride);

    float width = boxes[index * 5 + 2] - boxes[index * 5 + 0] + 1.0f;
    float height = boxes[index * 5 + 3] - boxes[index * 5 + 1] + 1.0f;
    float ctr_x = boxes[index * 5 + 0] + 0.5f * (width - 1.0f);
    float ctr_y = boxes[index * 5 + 1] + 0.5f * (height - 1.0f);

    int ba = (b * num_anchors + a);
    float dx = deltas[((ba * 4) * feat_height + h) * feat_width + w];
    float dy = deltas[((ba * 4 + 1) * feat_height + h) * feat_width + w];
    float dw = deltas[((ba * 4 + 2) * feat_height + h) * feat_width + w];
    float dh = deltas[((ba * 4 + 3) * feat_height + h) * feat_width + w];

    float pred_ctr_x = dx * width + ctr_x;
    float pred_ctr_y = dy * height + ctr_y;
    float pred_w = exp(dw) * width;
    float pred_h = exp(dh) * height;

    float pred_x1 = pred_ctr_x - 0.5f * (pred_w - 1.0f);
    float pred_y1 = pred_ctr_y - 0.5f * (pred_h - 1.0f);
    float pred_x2 = pred_ctr_x + 0.5f * (pred_w - 1.0f);
    float pred_y2 = pred_ctr_y + 0.5f * (pred_h - 1.0f);

    pred_x1 = max(min(pred_x1, im_width - 1.0f), 0.0f);
    pred_y1 = max(min(pred_y1, im_height - 1.0f), 0.0f);
    pred_x2 = max(min(pred_x2, im_width - 1.0f), 0.0f);
    pred_y2 = max(min(pred_y2, im_height - 1.0f), 0.0f);

    out_pred_boxes[index * 5 + 0] = pred_x1;
    out_pred_boxes[index * 5 + 1] = pred_y1;
    out_pred_boxes[index * 5 + 2] = pred_x2;
    out_pred_boxes[index * 5 + 3] = pred_y2;

    if (h >= real_height || w >= real_width) {
      out_pred_boxes[index * 5 + 4] = -1.0f;
    }
  }
}

// boxes are (b, h * w * anchor, 5)
// deltas are (b, 4 * anchor, h, w)
// out_pred_boxes are (b, h * w * anchor, 5)
// count should be total anchors numbers, b * h * w * anchors
// in-place write: boxes and out_pred_boxes are the same location
template<typename Dtype>
__global__ void IoUPredKernel(const int count,
                              const int num_anchors,
                              const int feat_height,
                              const int feat_width,
                              const int feature_stride,
                              const Dtype* im_infos,
                              const Dtype* boxes,
                              const Dtype* deltas,
                              Dtype* out_pred_boxes) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < count;
       index += blockDim.x * gridDim.x) {
    int a = index % num_anchors;
    int w = (index / num_anchors) % feat_width;
    int h = (index / num_anchors / feat_width) % feat_height;
    int b = index / num_anchors / feat_width / feat_height;

    float im_height = im_infos[b * 3];
    float im_width = im_infos[b * 3 + 1];
    int real_height = static_cast<int>(im_height / feature_stride);
    int real_width = static_cast<int>(im_width / feature_stride);

    float x1 = boxes[index * 5 + 0];
    float y1 = boxes[index * 5 + 1];
    float x2 = boxes[index * 5 + 2];
    float y2 = boxes[index * 5 + 3];

    int ba = (b * num_anchors + a);
    float dx1 = deltas[((ba * 4) * feat_height + h) * feat_width + w];
    float dy1 = deltas[((ba * 4 + 1) * feat_height + h) * feat_width + w];
    float dx2 = deltas[((ba * 4 + 2) * feat_height + h) * feat_width + w];
    float dy2 = deltas[((ba * 4 + 3) * feat_height + h) * feat_width + w];

    float pred_x1 = max(min(x1 + dx1, im_width - 1.0f), 0.0f);
    float pred_y1 = max(min(y1 + dy1, im_height - 1.0f), 0.0f);
    float pred_x2 = max(min(x2 + dx2, im_width - 1.0f), 0.0f);
    float pred_y2 = max(min(y2 + dy2, im_height - 1.0f), 0.0f);

    out_pred_boxes[index * 5 + 0] = pred_x1;
    out_pred_boxes[index * 5 + 1] = pred_y1;
    out_pred_boxes[index * 5 + 2] = pred_x2;
    out_pred_boxes[index * 5 + 3] = pred_y2;

    if (h >= real_height || w >= real_width) {
      out_pred_boxes[index * 5 + 4] = -1.0f;
    }
  }
}

// filter box with stride less than rpn_min_size
// filter: set score to zero
// dets (b, n, 5)
template<typename Dtype>
__global__ void FilterBoxKernel(const int count,
                                const int count_anchors,
                                const float original_min_size,
                                const Dtype* im_infos,
                                Dtype* dets) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < count;
       index += blockDim.x * gridDim.x) {
    int b = index / count_anchors;
    float iw = dets[index * 5 + 2] - dets[index * 5 + 0] + 1.0f;
    float ih = dets[index * 5 + 3] - dets[index * 5 + 1] + 1.0f;
    float min_size = original_min_size * im_infos[b * 3 + 2];
    if (iw < min_size || ih < min_size) {
      dets[index * 5 + 0] -= min_size / 2;
      dets[index * 5 + 1] -= min_size / 2;
      dets[index * 5 + 2] += min_size / 2;
      dets[index * 5 + 3] += min_size / 2;
      dets[index * 5 + 4] = -1.0f;
    }
  }
}

// copy score and init order
// dets (n, 5); score (n, ); order (n, )
// count should be n (total anchors or proposals)
template<typename Dtype>
__global__ void CopyScoreKernel(const int count,
                                const Dtype* dets,
                                Dtype* score,
                                int* order) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < count;
       index += blockDim.x * gridDim.x) {
    score[index] = dets[index * 5 + 4];
    order[index] = index;
  }
}

// reorder proposals according to order and keep the top_n proposals
// prev_dets (n, 5); order (n, ); dets (n, 5)
// count should be output anchor numbers (top_n)
template<typename Dtype>
__global__ void ReorderProposalsKernel(const int count,
                                       const Dtype* prev_dets,
                                       const int* order,
                                       Dtype* dets) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < count;
       index += blockDim.x * gridDim.x) {
    const int order_i = order[index];
    for (int j = 0; j < 5; j ++) {
      dets[index * 5 + j] = prev_dets[order_i * 5 + j];
    }
  }
}

__device__ inline float devIoU(float const * const a, float const * const b) {
  float left = max(a[0], b[0]), right = min(a[2], b[2]);
  float top = max(a[1], b[1]), bottom = min(a[3], b[3]);
  float width = max(right - left + 1, 0.f), height = max(bottom - top + 1, 0.f);
  float interS = width * height;
  float Sa = (a[2] - a[0] + 1) * (a[3] - a[1] + 1);
  float Sb = (b[2] - b[0] + 1) * (b[3] - b[1] + 1);
  return interS / (Sa + Sb - interS);
}

__global__ void nms_kernel(const int n_boxes, const float nms_overlap_thresh,
                           const float *dev_boxes, uint64_t *dev_mask) {
  const int threadsPerBlock = sizeof(uint64_t) * 8;
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;

  const int row_size =
        min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ float block_boxes[threadsPerBlock * 5];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 5 + 0] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 0];
    block_boxes[threadIdx.x * 5 + 1] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 1];
    block_boxes[threadIdx.x * 5 + 2] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 2];
    block_boxes[threadIdx.x * 5 + 3] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 3];
    block_boxes[threadIdx.x * 5 + 4] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 4];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const float *cur_box = dev_boxes + cur_box_idx * 5;
    int i = 0;
    uint64_t t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (devIoU(cur_box, block_boxes + i * 5) > nms_overlap_thresh) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = DIVUP(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

void _nms(mshadow::Stream<gpu> *s,
          const mshadow::Tensor<gpu, 2>& boxes,
          const float nms_overlap_thresh,
          const int rpn_post_nms_top_n,
          int *keep,
          int *num_out) {
  const int threadsPerBlock = sizeof(uint64_t) * 8;
  const int boxes_num = boxes.size(0);
  const int boxes_dim = boxes.size(1);

  float* boxes_dev = boxes.dptr_;
  uint64_t* mask_dev = nullptr;

  const int col_blocks = DIVUP(boxes_num, threadsPerBlock);
  FRCNN_CUDA_CHECK(hipMalloc(&mask_dev,
                              boxes_num * col_blocks * sizeof(uint64_t)));

  dim3 blocks(DIVUP(boxes_num, threadsPerBlock),
              DIVUP(boxes_num, threadsPerBlock));
  dim3 threads(threadsPerBlock);
  nms_kernel<<<blocks, threads>>>(boxes_num,
                                  nms_overlap_thresh,
                                  boxes_dev,
                                  mask_dev);
  FRCNN_CUDA_CHECK(hipGetLastError());
  std::vector<uint64_t> mask_host(boxes_num * col_blocks);

  hipStream_t stream = mshadow::Stream<gpu>::GetStream(s);
  FRCNN_CUDA_CHECK(hipMemcpyAsync(&mask_host[0],
                                   mask_dev,
                                   sizeof(uint64_t) * boxes_num * col_blocks,
                                   hipMemcpyDeviceToHost, stream));
  FRCNN_CUDA_CHECK(hipStreamSynchronize(stream));

  std::vector<uint64_t> remv(col_blocks);
  memset(&remv[0], 0, sizeof(uint64_t) * col_blocks);

  int num_to_keep = 0;
  for (int i = 0; i < boxes_num; i++) {
    int nblock = i / threadsPerBlock;
    int inblock = i % threadsPerBlock;

    if (!(remv[nblock] & (1ULL << inblock))) {
      keep[num_to_keep++] = i;
      if (num_to_keep >= rpn_post_nms_top_n) break;
      uint64_t *p = &mask_host[0] + i * col_blocks;
      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];
      }
    }
  }
  *num_out = num_to_keep;

  FRCNN_CUDA_CHECK(hipFree(mask_dev));
}

// copy proposals to output
// dets (top_n, 5); keep (top_n, ); out (top_n, )
// count should be top_n (total anchors or proposals)
template<typename Dtype>
__global__ void PrepareOutput(const int count,
                              const Dtype* dets,
                              const int* keep,
                              const int out_size,
                              const int image_index,
                              Dtype* out,
                              Dtype* score) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < count;
       index += blockDim.x * gridDim.x) {
    out[index * 5] = image_index;
    if (index < out_size) {
      int keep_i = keep[index];
      for (int j = 0; j < 4; ++j) {
        out[index * 5 + j + 1] = dets[keep_i * 5 + j];
      }
      score[index] = dets[keep_i * 5 + 4];
    } else {
      int keep_i = keep[index % out_size];
      for (int j = 0; j < 4; ++j) {
        out[index * 5 + j + 1] = dets[keep_i * 5 + j];
      }
      score[index] = dets[keep_i * 5 + 4];
    }
  }
}
}  // namespace multi_proposal
}  // namespace cuda
}  // namespace mshadow

namespace mxnet {
namespace op {

template<typename xpu>
class MultiProposalGPUOp : public Operator{
 public:
  explicit MultiProposalGPUOp(MultiProposalParam param) {
    this->param_ = param;
  }

  virtual void Forward(const OpContext &ctx,
                       const std::vector<TBlob> &in_data,
                       const std::vector<OpReqType> &req,
                       const std::vector<TBlob> &out_data,
                       const std::vector<TBlob> &aux_states) {
    using namespace mshadow;
    using namespace mshadow::expr;
    using namespace mshadow::cuda;
    using namespace mshadow::cuda::multi_proposal;
    CHECK_EQ(in_data.size(), 3);
    CHECK_EQ(out_data.size(), 2);
    CHECK_GT(req.size(), 1);
    CHECK_EQ(req[proposal::kOut], kWriteTo);
    /*CHECK_EQ(in_data[proposal::kClsProb].shape_[0], 1)
      << "Sorry, multiple images each device is not implemented.";*/

    Stream<xpu> *s = ctx.get_stream<xpu>();

    Tensor<xpu, 4> scores = in_data[proposal::kClsProb].get<xpu, 4, real_t>(s);
    Tensor<xpu, 4> bbox_deltas = in_data[proposal::kBBoxPred].get<xpu, 4, real_t>(s);
    Tensor<xpu, 2> im_info = in_data[proposal::kImInfo].get<xpu, 2, real_t>(s);

    Tensor<xpu, 2> out = out_data[proposal::kOut].get<xpu, 2, real_t>(s);
    Tensor<xpu, 2> out_score = out_data[proposal::kScore].get<xpu, 2, real_t>(s);

    int num_images = scores.size(0);
    int num_anchors = scores.size(1) / 2;
    int height = scores.size(2);
    int width = scores.size(3);
    int count_anchors = num_anchors * height * width;  // count of total anchors
    int count = num_images * count_anchors;
    // set to -1 for max
    int rpn_pre_nms_top_n = (param_.rpn_pre_nms_top_n > 0) ? param_.rpn_pre_nms_top_n
                                                           : count_anchors;
    rpn_pre_nms_top_n = std::min(rpn_pre_nms_top_n, count_anchors);
    int rpn_post_nms_top_n = std::min(param_.rpn_post_nms_top_n, rpn_pre_nms_top_n);

    // Generate first anchors based on base anchor
    std::vector<float> base_anchor(4);
    base_anchor[0] = 0.0;
    base_anchor[1] = 0.0;
    base_anchor[2] = param_.feature_stride - 1.0;
    base_anchor[3] = param_.feature_stride - 1.0;
    CHECK_EQ(num_anchors, param_.ratios.ndim() * param_.scales.ndim());
    std::vector<float> anchors;
    utils::GenerateAnchors(base_anchor,
                           param_.ratios,
                           param_.scales,
                           &anchors);

    // Copy generated anchors to GPU
    float* workspace_proposals_ptr = nullptr;
    FRCNN_CUDA_CHECK(hipMalloc(&workspace_proposals_ptr,
                                sizeof(float) * num_images * count_anchors * 5));
    Tensor<xpu, 3> workspace_proposals(workspace_proposals_ptr,
                                       Shape3(num_images, count_anchors, 5));

    hipStream_t stream = mshadow::Stream<gpu>::GetStream(s);

    FRCNN_CUDA_CHECK(hipMemcpyAsync(workspace_proposals.dptr_, &anchors[0],
                                     sizeof(float) * anchors.size(),
                                     hipMemcpyHostToDevice, stream));

    // Copy proposals to a mesh grid
    dim3 dimGrid((count + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock);
    dim3 dimBlock(kMaxThreadsPerBlock);
    CheckLaunchParam(dimGrid, dimBlock, "ProposalGrid");
    ProposalGridKernel<<<dimGrid, dimBlock>>>(
      count, num_anchors, height, width, param_.feature_stride,
      scores.dptr_, workspace_proposals.dptr_);
    FRCNN_CUDA_CHECK(hipGetLastError());

    // Transform anchors and bbox_deltas into bboxes
    CheckLaunchParam(dimGrid, dimBlock, "BBoxPred");
    if (param_.iou_loss) {
      IoUPredKernel<<<dimGrid, dimBlock>>>(
        count, num_anchors, height, width, param_.feature_stride, im_info.dptr_,
        workspace_proposals.dptr_, bbox_deltas.dptr_, workspace_proposals.dptr_);
    } else {
      BBoxPredKernel<<<dimGrid, dimBlock>>>(
        count, num_anchors, height, width, param_.feature_stride, im_info.dptr_,
        workspace_proposals.dptr_, bbox_deltas.dptr_, workspace_proposals.dptr_);
    }
    FRCNN_CUDA_CHECK(hipGetLastError());

    // filter boxes with less than rpn_min_size
    CheckLaunchParam(dimGrid, dimBlock, "FilterBox");
    FilterBoxKernel<<<dimGrid, dimBlock>>>(
      count, count_anchors, param_.rpn_min_size, im_info.dptr_, workspace_proposals.dptr_);
    FRCNN_CUDA_CHECK(hipGetLastError());



    dimGrid = dim3((count_anchors + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock);
    dimBlock = dim3(kMaxThreadsPerBlock);
    // Copy score to a continuous memory
    float* score_ptr = nullptr;
    FRCNN_CUDA_CHECK(hipMalloc(&score_ptr, sizeof(float) * count_anchors));
    Tensor<xpu, 1> score(score_ptr, Shape1(count_anchors));
    int* order_ptr = nullptr;
    FRCNN_CUDA_CHECK(hipMalloc(&order_ptr, sizeof(int) * count_anchors));
    Tensor<xpu, 1, int> order(order_ptr, Shape1(count_anchors));

    float* workspace_ordered_proposals_ptr = nullptr;
    FRCNN_CUDA_CHECK(hipMalloc(&workspace_ordered_proposals_ptr,
        sizeof(float) * rpn_pre_nms_top_n * 5));
    Tensor<xpu, 2> workspace_ordered_proposals(workspace_ordered_proposals_ptr,
        Shape2(rpn_pre_nms_top_n, 5));

    int* keep;
    FRCNN_CUDA_CHECK(hipMalloc(&keep, sizeof(int) * rpn_pre_nms_top_n));

    for (int b = 0; b < num_images; b++) {
      CheckLaunchParam(dimGrid, dimBlock, "CopyScore");
      CopyScoreKernel << <dimGrid, dimBlock >> >(
          count_anchors, workspace_proposals.dptr_ + b * count_anchors * 5,
          score.dptr_, order.dptr_);
      FRCNN_CUDA_CHECK(hipGetLastError());

      // argsort score, save order
      thrust::stable_sort_by_key(thrust::device,
          score.dptr_,
          score.dptr_ + score.size(0),
          order.dptr_,
          thrust::greater<real_t>());
      FRCNN_CUDA_CHECK(hipGetLastError());

      // Reorder proposals according to order

      dimGrid.x = (rpn_pre_nms_top_n + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock;
      CheckLaunchParam(dimGrid, dimBlock, "ReorderProposals");
      ReorderProposalsKernel << <dimGrid, dimBlock >> >(
          rpn_pre_nms_top_n, workspace_proposals.dptr_ + b * count_anchors * 5,
          order.dptr_, workspace_ordered_proposals.dptr_);
      FRCNN_CUDA_CHECK(hipGetLastError());

      // perform nms
      std::vector<int> _keep(workspace_ordered_proposals.size(0));
      int out_size = 0;
      _nms(s, workspace_ordered_proposals,
           param_.threshold,
           rpn_post_nms_top_n,
           &_keep[0],
           &out_size);

      // copy nms result to gpu
      FRCNN_CUDA_CHECK(hipMemcpyAsync(keep, &_keep[0], sizeof(int) * _keep.size(),
                                       hipMemcpyHostToDevice, stream));

      // copy results after nms
      dimGrid.x = (param_.rpn_post_nms_top_n + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock;
      CheckLaunchParam(dimGrid, dimBlock, "PrepareOutput");
      PrepareOutput << <dimGrid, dimBlock >> >(
          param_.rpn_post_nms_top_n, workspace_ordered_proposals.dptr_, keep, out_size, b,
          out.dptr_ + b * param_.rpn_post_nms_top_n * 5,
          out_score.dptr_ + b * param_.rpn_post_nms_top_n);
      FRCNN_CUDA_CHECK(hipGetLastError());
    }
    // free temporary memory
    FRCNN_CUDA_CHECK(hipFree(keep));
    FRCNN_CUDA_CHECK(hipFree(workspace_ordered_proposals_ptr));
    FRCNN_CUDA_CHECK(hipFree(workspace_proposals_ptr));
    FRCNN_CUDA_CHECK(hipFree(score_ptr));
    FRCNN_CUDA_CHECK(hipFree(order_ptr));
  }

  virtual void Backward(const OpContext &ctx,
                        const std::vector<TBlob> &out_grad,
                        const std::vector<TBlob> &in_data,
                        const std::vector<TBlob> &out_data,
                        const std::vector<OpReqType> &req,
                        const std::vector<TBlob> &in_grad,
                        const std::vector<TBlob> &aux_states) {
    using namespace mshadow;
    using namespace mshadow::expr;
    CHECK_EQ(in_grad.size(), 3);

    Stream<xpu> *s = ctx.get_stream<xpu>();
    Tensor<xpu, 4> gscores = in_grad[proposal::kClsProb].get<xpu, 4, real_t>(s);
    Tensor<xpu, 4> gbbox = in_grad[proposal::kBBoxPred].get<xpu, 4, real_t>(s);
    Tensor<xpu, 2> ginfo = in_grad[proposal::kImInfo].get<xpu, 2, real_t>(s);

    // can not assume the grad would be zero
    Assign(gscores, req[proposal::kClsProb], 0);
    Assign(gbbox, req[proposal::kBBoxPred], 0);
    Assign(ginfo, req[proposal::kImInfo], 0);
  }

 private:
  MultiProposalParam param_;
};  // class MultiProposalGPUOp

template<>
Operator* CreateOp<gpu>(MultiProposalParam param) {
  return new MultiProposalGPUOp<gpu>(param);
}
}  // namespace op
}  // namespace mxnet
