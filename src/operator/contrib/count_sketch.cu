#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2015 by Contributors
 * \file count_sketch.cu
 * \brief count_sketch op
 * \author Chen Zhu, Yang Shi
*/
#include "./count_sketch-inl.h"
#include <mshadow/tensor.h>
#include <stdio.h>
#include <algorithm>



#define WARPS_PER_BLOCK 1
#define THREADS_PER_BLOCK 512

namespace mshadow {
namespace cuda {
// wrappers to deal with atomic add
// supporting only single precision
__device__ void atomic_add(float* dst, float val) {
  atomicAdd(dst, val);
}

// for double precision
__device__ void atomic_add(double* address, double val) {
  // code example in the official document at:
  // http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html
  // #atomic-functions

  // NOLINT_NEXT_LINE(runtime/int)
  unsigned long long int* address_as_ull = (unsigned long long int*) address;  // NOLINT(*)
  unsigned long long int old = *address_as_ull, assumed;                     // NOLINT(*)
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
            __double_as_longlong(val + __longlong_as_double(assumed)));
    // Note: uses integer comparison to avoid hang in case of NaN
    // (since NaN != NaN)
  } while (assumed != old);
}

template <typename DType>
__global__ void sketch_forward_kernel(const int nthreads, DType *out, const DType *h,
                    const DType *s, const DType *in, const int n_smaples,
                    const int in_dim, const int out_dim) {
  // input: n_smaples * in_dim
  // output: n_smaples * out_dim
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= nthreads) {
    return;
  }
  // nthreads is the maximum of thread indices, should be equal to in_dim
  // index is point index
  const int i_indim = index % in_dim;
  const int i_sample = index / in_dim;

  // get the target location in the output
  const int target = i_sample*out_dim + h[i_indim];
  atomic_add(out + target, s[i_indim] * in[index]);
}

template <typename DType>
__global__ void sketch_backward_kernel(const int nthreads, DType *in_grad, const DType *h,
                    const DType *s, const DType *out_grad, const int n_smaples,
                    const int in_dim, const int out_dim) {
  // only calculate gradient regarding x
  // can also calculate gradient regarding s if needed
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int i_indim = index % in_dim;
  const int i_sample = index / in_dim;
  const int i_outdim = i_sample*out_dim + h[i_indim];
  in_grad[index] = out_grad[i_outdim] * s[i_indim];
}

}  // namespace cuda

// CountSketch Forward
template <typename DType>
inline void CountSketchForward(const Tensor<gpu, 2, DType> &out,
                               const Tensor<gpu, 2, DType> &in,
                               const Tensor<gpu, 1, DType> &h,
                               const Tensor<gpu, 1, DType> &s,
                               const int n_samples,
                               const int processing_batch_size,
                               const int in_dim,
                               const int out_dim) {
  DType *out_ptr = out.dptr_;
  const DType *in_ptr = in.dptr_;
  const DType *h_ptr = h.dptr_;
  const DType *s_ptr = s.dptr_;
  int upper_bound = n_samples/processing_batch_size;
  if (n_samples%processing_batch_size == 0) {
    upper_bound = upper_bound-1;
  }
  // guarantee there are at least one iteration
  upper_bound = upper_bound > 0? upper_bound:0;
  int bstart = 0;
  for ( int i = 0; i <= upper_bound; i++ ) {
    const int batchlen = min(processing_batch_size, n_samples - bstart);
    const int nthreads = batchlen * in_dim;
    // to make number of threads the same as input
    const int threads_per_block = min(THREADS_PER_BLOCK, nthreads);
    int nblocks = (nthreads + threads_per_block - 1) / threads_per_block;
    cuda::sketch_forward_kernel<DType><<<nblocks, threads_per_block>>>(
                                    nthreads, out_ptr+bstart*out_dim, h_ptr,
                                    s_ptr, in_ptr+bstart*in_dim, batchlen,
                                    in_dim, out_dim);
    hipError_t err = hipDeviceSynchronize();
    CHECK_EQ(err, hipSuccess) << "Error occured! CUDA: " << hipGetErrorString(err);
    bstart = (i+1)*batchlen;
  }
}

template<typename DType>
inline void CountSketchBackward(const Tensor<gpu, 2, DType> &in_grad,
                                const Tensor<gpu, 2, DType> &out_grad,
                                const Tensor<gpu, 1, DType> &h,
                                const Tensor<gpu, 1, DType> &s,
                                const int n_samples,
                                const int processing_batch_size,
                                const int in_dim,
                                const int out_dim) {
  DType *in_grad_ptr = in_grad.dptr_;
  const DType *out_grad_ptr = out_grad.dptr_;
  const DType *h_ptr = h.dptr_;
  const DType *s_ptr = s.dptr_;
  int upper_bound = n_samples/processing_batch_size;
  if (n_samples%processing_batch_size == 0) {
    upper_bound = upper_bound-1;
  }
  // guarantee there are at least one iteration
  upper_bound = upper_bound > 0 ? upper_bound : 0;
  int bstart = 0;
  for ( int i = 0; i <= upper_bound; i++ ) {
    const int batchlen = min(processing_batch_size, n_samples - bstart);
    const int nthreads = batchlen * in_dim;
    // to make number of threads the same as input
    const int threads_per_block = min(THREADS_PER_BLOCK, nthreads);
    int nblocks = (nthreads + threads_per_block - 1) / threads_per_block;
    cuda::sketch_backward_kernel<DType><<<nblocks, threads_per_block>>>(
                            nthreads, in_grad_ptr+bstart*in_dim, h_ptr,
                            s_ptr, out_grad_ptr+bstart*out_dim, batchlen,
                            in_dim, out_dim);
    hipError_t err = hipDeviceSynchronize();
    CHECK_EQ(err, hipSuccess) << "Error occured! CUDA: " << hipGetErrorString(err);
    bstart = (i+1)*batchlen;
  }
}
}  // namespace mshadow
namespace mxnet {
namespace op {
template<>
Operator* CreateOp<gpu>(CountSketchParam param, int dtype) {
  Operator *op = nullptr;
  switch (dtype) {
      case mshadow::kFloat32:
          op = new CountSketchOp<gpu, float>(param);
          break;
      case mshadow::kFloat64:
          op = new CountSketchOp<gpu, double>(param);
          break;
      case mshadow::kFloat16:
          LOG(FATAL) << "float16 count sketch layer is currently"
                  "not supported.";
          break;
      default:
          LOG(FATAL) << "Unsupported type " << dtype;
  }
  return op;
}
}  // namespace op
}  // namespace mxnet
