#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2017 by Contributors
 * Copyright (c) 2017 Microsoft
 * Licensed under The Apache-2.0 License [see LICENSE for details]
 * \file psroi_pooling.cu
 * \brief psroi pooling operator
 * \author Yi Li, Tairui Chen, Guodong Zhang, Haozhi Qi, Jifeng Dai
*/
#include "./psroi_pooling-inl.h"
#include <mshadow/tensor.h>
#include <mshadow/cuda/reduce.cuh>
#include <algorithm>
#include <vector>
#include "../../common/cuda/utils.h"
#include "../mxnet_op.h"

#define PSROIPOOLING_CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    CHECK_EQ(error, hipSuccess) << " " << hipGetErrorString(error); \
  } while (0)

namespace mshadow {
namespace cuda {

template <typename DType>
__global__ void PSROIPoolForwardKernel(
  const int count,
  const DType* bottom_data,
  const DType spatial_scale,
  const int channels,
  const int height, const int width,
  const int pooled_height, const int pooled_width,
  const DType* bottom_rois,
  const int output_dim,
  const int group_size,
  DType* top_data) {
  CUDA_KERNEL_LOOP(index, count) {
    // The output is in order (n, ctop, ph, pw)
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int ctop = (index / pooled_width / pooled_height) % output_dim;
    int n = index / pooled_width / pooled_height / output_dim;

    // [start, end) interval for spatial sampling
    const DType* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];
    DType roi_start_w = static_cast<DType>(round(offset_bottom_rois[1])) * spatial_scale;
    DType roi_start_h = static_cast<DType>(round(offset_bottom_rois[2])) * spatial_scale;
    DType roi_end_w = static_cast<DType>(round(offset_bottom_rois[3]) + 1.) * spatial_scale;
    DType roi_end_h = static_cast<DType>(round(offset_bottom_rois[4]) + 1.) * spatial_scale;

    // Force too small ROIs to be 1x1
    DType roi_width = max(roi_end_w - roi_start_w, 0.1);  // avoid 0
    DType roi_height = max(roi_end_h - roi_start_h, 0.1);

    // Compute w and h at bottom
    DType bin_size_h = roi_height / static_cast<DType>(pooled_height);
    DType bin_size_w = roi_width / static_cast<DType>(pooled_width);

    int hstart = floor(static_cast<DType>(ph) * bin_size_h
                        + roi_start_h);
    int wstart = floor(static_cast<DType>(pw)* bin_size_w
                        + roi_start_w);
    int hend = ceil(static_cast<DType>(ph + 1) * bin_size_h
                      + roi_start_h);
    int wend = ceil(static_cast<DType>(pw + 1) * bin_size_w
                      + roi_start_w);
    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart, 0), height);
    hend = min(max(hend, 0), height);
    wstart = min(max(wstart, 0), width);
    wend = min(max(wend, 0), width);
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    int gw = floor(static_cast<DType>(pw)* group_size / pooled_width);
    int gh = floor(static_cast<DType>(ph)* group_size / pooled_height);
    gw = min(max(gw, 0), group_size - 1);
    gh = min(max(gh, 0), group_size - 1);
    int c = (ctop*group_size + gh)*group_size + gw;

    const DType* offset_bottom_data = bottom_data + (roi_batch_ind * channels + c) * height * width;
    DType out_sum = 0;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int bottom_index = h*width + w;
        out_sum += offset_bottom_data[bottom_index];
      }
    }

    DType bin_area = (hend - hstart)*(wend - wstart);
    top_data[index] = is_empty? (DType)0. : out_sum/bin_area;
  }
}

template<typename DType>
inline void PSROIPoolForward(const Tensor<gpu, 4, DType> &out,
                           const Tensor<gpu, 4, DType> &data,
                           const Tensor<gpu, 2, DType> &bbox,
                           const float spatial_scale,
                           const int output_dim_,
                           const int group_size_) {
  const DType *bottom_data = data.dptr_;
  const DType *bottom_rois = bbox.dptr_;
  DType *top_data = out.dptr_;
  const int count = out.shape_.Size();
  const int channels = data.size(1);
  const int height = data.size(2);
  const int width = data.size(3);
  const int pooled_height = out.size(2);
  const int pooled_width = out.size(3);
  hipStream_t stream = Stream<gpu>::GetStream(out.stream_);
  PSROIPoolForwardKernel<DType> << <mxnet::op::mxnet_op::cuda_get_num_blocks(count),
    kBaseThreadNum, 0, stream >> >(
      count, bottom_data, spatial_scale, channels, height, width,
      pooled_height, pooled_width, bottom_rois, output_dim_, group_size_, top_data);
  PSROIPOOLING_CUDA_CHECK(hipGetLastError());
}


template <typename DType>
__global__ void PSROIPoolBackwardAccKernel(
  const int count,
  const DType* top_diff,
  const int num_rois,
  const DType spatial_scale,
  const int channels,
  const int height, const int width,
  const int pooled_height, const int pooled_width,
  const int group_size,
  const int output_dim,
  DType* bottom_diff,
  const DType* bottom_rois) {
  CUDA_KERNEL_LOOP(index, count) {
    // The output is in order (n, ctop, ph, pw)
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int ctop = (index / pooled_width / pooled_height) % output_dim;
    int n = index / pooled_width / pooled_height / output_dim;

    // [start, end) interval for spatial sampling
    const DType* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];
    DType roi_start_w = static_cast<DType>(round(offset_bottom_rois[1])) * spatial_scale;
    DType roi_start_h = static_cast<DType>(round(offset_bottom_rois[2])) * spatial_scale;
    DType roi_end_w = static_cast<DType>(round(offset_bottom_rois[3]) + 1.) * spatial_scale;
    DType roi_end_h = static_cast<DType>(round(offset_bottom_rois[4]) + 1.) * spatial_scale;

    // Force too small ROIs to be 1x1
    DType roi_width = max(roi_end_w - roi_start_w, 0.1);  // avoid 0
    DType roi_height = max(roi_end_h - roi_start_h, 0.1);

    // Compute w and h at bottom
    DType bin_size_h = roi_height / static_cast<DType>(pooled_height);
    DType bin_size_w = roi_width / static_cast<DType>(pooled_width);

    int hstart = floor(static_cast<DType>(ph)* bin_size_h
      + roi_start_h);
    int wstart = floor(static_cast<DType>(pw)* bin_size_w
      + roi_start_w);
    int hend = ceil(static_cast<DType>(ph + 1) * bin_size_h
      + roi_start_h);
    int wend = ceil(static_cast<DType>(pw + 1) * bin_size_w
      + roi_start_w);
    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart, 0), height);
    hend = min(max(hend, 0), height);
    wstart = min(max(wstart, 0), width);
    wend = min(max(wend, 0), width);
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    // Compute c at bottom
    int gw = floor(static_cast<DType>(pw)* group_size / pooled_width);
    int gh = floor(static_cast<DType>(ph)* group_size / pooled_height);
    gw = min(max(gw, 0), group_size - 1);
    gh = min(max(gh, 0), group_size - 1);
    int c = (ctop*group_size + gh)*group_size + gw;
    DType* offset_bottom_diff = bottom_diff + (roi_batch_ind * channels + c) * height * width;
    DType bin_area = (hend - hstart)*(wend - wstart);
    DType diff_val = is_empty ? (DType)0. : top_diff[index] / bin_area;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int bottom_index = h*width + w;
        atomicAdd(offset_bottom_diff + bottom_index, diff_val);
      }
    }
  }
}


template<typename DType>
inline void PSROIPoolBackwardAcc(const Tensor<gpu, 4, DType> &in_grad,
                            const Tensor<gpu, 4, DType> &out_grad,
                            const Tensor<gpu, 2, DType> &bbox,
                            const float spatial_scale,
                            const int output_dim_,
                            const int group_size_) {
  // LOG(INFO) << "PSROIPoolBackward";
  const DType *top_diff = out_grad.dptr_;
  const DType *bottom_rois = bbox.dptr_;
  DType *bottom_diff = in_grad.dptr_;
  const int count = out_grad.shape_.Size();
  const int num_rois = bbox.size(0);
  const int channels = in_grad.size(1);
  const int height = in_grad.size(2);
  const int width = in_grad.size(3);
  const int pooled_height = out_grad.size(2);
  const int pooled_width = out_grad.size(3);
  hipStream_t stream = Stream<gpu>::GetStream(in_grad.stream_);
  PSROIPoolBackwardAccKernel<DType> << <mxnet::op::mxnet_op::cuda_get_num_blocks(count),
    kBaseThreadNum, 0, stream >> >(
      count, top_diff, num_rois, spatial_scale, channels, height, width,
      pooled_height, pooled_width, group_size_, output_dim_, bottom_diff, bottom_rois);
  PSROIPOOLING_CUDA_CHECK(hipGetLastError());
}

}  // namespace cuda

template<typename DType>
inline void PSROIPoolForward(const Tensor<gpu, 4, DType> &out,
                           const Tensor<gpu, 4, DType> &data,
                           const Tensor<gpu, 2, DType> &bbox,
                           const float spatial_scale,
                           const int output_dim_,
                           const int group_size_) {
  cuda::PSROIPoolForward(out, data, bbox, spatial_scale, output_dim_, group_size_);
}

template<typename DType>
inline void PSROIPoolBackwardAcc(const Tensor<gpu, 4, DType> &in_grad,
                            const Tensor<gpu, 4, DType> &out_grad,
                            const Tensor<gpu, 2, DType> &bbox,
                            const float spatial_scale,
                            const int output_dim_,
                            const int group_size_) {
  cuda::PSROIPoolBackwardAcc(in_grad, out_grad, bbox, spatial_scale, output_dim_, group_size_);
}

}  // namespace mshadow


namespace mxnet {
namespace op {

template<>
Operator* CreateOp<gpu>(PSROIPoolingParam param, int dtype) {
  Operator* op = nullptr;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new PSROIPoolingOp<gpu, DType>(param);
  });
  return op;
}

}  // namespace op
}  // namespace mxnet
