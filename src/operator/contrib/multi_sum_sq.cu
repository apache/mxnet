#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 *  Copyright (c) 2020 by Contributors
 * \file multi_sum_sq.cu
 * \brief vectorized sums of squares norm over multiple arrays operators
 * \author Clement Fuji Tsang, Andrei Ivanov, Moises Hernandez, Shuai Zheng
 */
#include "./multi_sum_sq-inl.h"
#include <hipcub/hipcub.hpp>

#define ILP 4
#define BLOCK_LIMIT 320
#define ARRAY_LIMIT 110

namespace mxnet {
namespace op {

// Shamelessly gotten from:
// https://github.com/NVIDIA/apex/blob/master/csrc/multi_tensor_apply.cuh
// https://github.com/NVIDIA/apex/blob/master/csrc/multi_tensor_l2norm_kernel.cu
// https://github.com/NVIDIA/apex/blob/master/csrc/type_shim.h

const int chunk_size = 32768;

template <typename DType>
struct MultiSumSqKernelParam {
  DType* addresses[ARRAY_LIMIT];
  int sizes[ARRAY_LIMIT];
  unsigned char block_to_tensor[BLOCK_LIMIT];
  int block_to_chunk[BLOCK_LIMIT];
  int max_chunks_per_tensor = -1;
};

template<typename DType>
__device__ __forceinline__ DType ReduceBlockIntoLanes(DType* x,
                                                      DType val) {
  int tid = threadIdx.x;
  int block_size = blockDim.x;

  if (block_size >= 64) {
    x[tid] = val;
    __syncthreads();
  }

  #pragma unroll
  for (int i = (block_size >> 1); i >= 64; i >>= 1) {
    if (tid < i)
      x[tid] = x[tid] + x[tid+i];
    __syncthreads();
  }

  DType final;
  if (tid < 32) {
    if (block_size >= 64)
      final = x[tid] + x[tid+32];
    else
      final = val;

    #pragma unroll
    for (int i = 16; i >= 1; i >>= 1)
      final = final + __shfl_down_sync(0xffffffff, final, i);
  }
  return final;
}

template<typename DType>
__global__ void MultiSumSqKernel(int chunk_size,
                                 MultiSumSqKernelParam<DType> param,
                                 float* block_reductions,
                                 int start_tensor_id,
                                 float scale) {
  const int tensor_loc = param.block_to_tensor[blockIdx.x];
  const int chunk_len = param.block_to_chunk[blockIdx.x] * chunk_size;
  const int n = param.sizes[tensor_loc] - chunk_len;
  const DType* x = param.addresses[tensor_loc] + chunk_len;
  const auto i_max = n <= chunk_size ? n : chunk_size;
  __shared__ float vals[512];

  // Non-divergent exit condition for __syncthreads, not necessary here
  float val = 0;
  for (int i_start = 0;
       i_start < i_max;
       i_start += blockDim.x * ILP) {
    int i = i_start + threadIdx.x;
#pragma unroll
    for (int ii = 0; ii < ILP && i < i_max; ++ii, i += blockDim.x) {
      auto incoming_val = static_cast<float>(x[i]);
      if (scale != 1.0f) {
         incoming_val *= scale;
      }
      val += incoming_val * incoming_val;
    }
  }
  const float final = ReduceBlockIntoLanes(vals, val);

  if (threadIdx.x == 0) {
    block_reductions[(start_tensor_id + tensor_loc) * param.max_chunks_per_tensor +
                    param.block_to_chunk[blockIdx.x]] = final;
  }
}

template<typename DType>
__global__ void GlobalReductionKernel(MultiSumSqKernelParam<DType> param,
                                     float* block_reductions,
                                     float* output) {
  __shared__ float vals[512];
  float* reductions_this_tensor = block_reductions + blockIdx.x * param.max_chunks_per_tensor;
  float val = 0;
  for (int i = threadIdx.x; i < param.max_chunks_per_tensor; i += blockDim.x)
    val += reductions_this_tensor[i];

  float final = ReduceBlockIntoLanes(vals, val);

  if (threadIdx.x == 0)
    output[blockIdx.x] = final;
}

template<>
size_t GetRequiredStorageMultiSumSq<gpu>(const std::vector<TBlob> &inputs,
                                         int* param_max_chunks_per_tensor) {
  // find max num of chunks in tensors
  int max_chunks_per_tensor = -1;
  for (size_t t = 0; t < inputs.size(); t++) {
    int chunks_this_tensor = (inputs[t].shape_.Size() + chunk_size - 1) / chunk_size;
    if (chunks_this_tensor > max_chunks_per_tensor)
      max_chunks_per_tensor = chunks_this_tensor;
  }
  if (param_max_chunks_per_tensor != nullptr)
    *param_max_chunks_per_tensor = max_chunks_per_tensor;
  return inputs.size() * max_chunks_per_tensor * sizeof(float);
}

template<>
void MultiSumSqRun<gpu>(const std::vector<TBlob> &inputs, int n_inputs,
                        float *out_ptr, const OpContext &ctx, float scale) {
  const int block_size = 512;
  using namespace mxnet_op;
  auto s = ctx.get_stream<gpu>();
  auto stream = mshadow::Stream<gpu>::GetStream(s);

  MSHADOW_REAL_TYPE_SWITCH(inputs[0].type_flag_, DType, {
    MultiSumSqKernelParam<DType> param;
    size_t workspace_size = GetRequiredStorageMultiSumSq<gpu>(inputs,
                                                              &param.max_chunks_per_tensor);
    Tensor<gpu, 1, char> workspace =
      ctx.requested[multi_sum_sq::kTempSpace].get_space_typed<gpu, 1, char>(
        Shape1(workspace_size), s);
    Tensor<gpu, 1, float> block_reductions(reinterpret_cast<float*>(&workspace[0]),
      Shape1(n_inputs * param.max_chunks_per_tensor), s);
    CUDA_CALL(hipMemsetAsync(block_reductions.dptr_, 0,
                              n_inputs * param.max_chunks_per_tensor* sizeof(float),
                              stream));

    int loc_block_info = 0;   // position in param.block_to_tensor and param.block_to_chunck
    int loc_tensor_info = 0;  // position in param.sizes and param.addresses
    int start_tensor_id = 0;
    for (int t = 0; t < n_inputs; t++, loc_tensor_info++) {  // array index in inputs
      param.sizes[loc_tensor_info] = inputs[t].shape_.Size();
      param.addresses[loc_tensor_info] = inputs[t].FlatTo2D<gpu, DType>(s).dptr_;
      const int chunks_this_tensor = (inputs[t].shape_.Size() - 1) / chunk_size;
      for (int chunk = 0; chunk <= chunks_this_tensor; ++chunk) {  // array chunk index
        param.block_to_tensor[loc_block_info] = loc_tensor_info;
        param.block_to_chunk[loc_block_info] = chunk;
        loc_block_info++;

        const bool last_curr_chunk = chunk == chunks_this_tensor;
        const bool tensors_full = last_curr_chunk && loc_tensor_info == (ARRAY_LIMIT-1);
        const bool blocks_full = (loc_block_info == BLOCK_LIMIT);
        const bool last_chunk = last_curr_chunk && t == n_inputs - 1;
        if (!(tensors_full || blocks_full || last_chunk))
          continue;
        MultiSumSqKernel<<<loc_block_info, block_size, 0, stream>>>
          (chunk_size, param, block_reductions.dptr_, start_tensor_id, scale);
        MSHADOW_CUDA_POST_KERNEL_CHECK(MultiSumSqKernel);

        loc_block_info = 0;
        if (last_curr_chunk) {  // if you start from a new tensor
          loc_tensor_info = -1;
          start_tensor_id = t + 1;
        } else {  // if you start from the same tensor
          param.sizes[0] = param.sizes[loc_tensor_info];
          param.addresses[0] = param.addresses[loc_tensor_info];
          loc_tensor_info = 0;
          start_tensor_id = t;
        }
      }
    }
    // Global reduction
    GlobalReductionKernel<<<n_inputs, block_size, 0, stream>>>
      (param, block_reductions.dptr_, out_ptr);
  });
}

NNVM_REGISTER_OP(multi_sum_sq)
.set_attr<FCompute>("FCompute<gpu>", MultiSumSq<gpu>);

}  // namespace op
}  // namespace mxnet
