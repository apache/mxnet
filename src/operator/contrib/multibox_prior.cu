#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2016 by Contributors
 * \file multibox_prior.cu
 * \brief generate multibox prior boxes cuda kernels
 * \author Joshua Zhang
*/

#include "./multibox_prior-inl.h"
#include <mshadow/cuda/tensor_gpu-inl.cuh>

#define MULTIBOXPRIOR_CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    CHECK_EQ(error, hipSuccess) << " " << hipGetErrorString(error); \
  } while (0)

namespace mshadow {
namespace cuda {
template<typename DType>
__global__ void AssignPriors(DType *out, const float size,
                             const float sqrt_ratio, const int in_width,
                             const int in_height, const float step_x,
                             const float step_y, const float center_offy,
                             const float center_offx, const int stride,
                             const int offset) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= in_width * in_height) return;
  int r = index / in_width;
  int c = index % in_width;
  float center_x = (c + center_offx) * step_x;
  float center_y = (r + center_offy) * step_y;
  float w = size * in_height / in_width * sqrt_ratio / 2;  // half width
  float h = size / sqrt_ratio / 2;  // half height
  DType *ptr = out + index * stride + 4 * offset;
  *(ptr++) = center_x - w;  // xmin
  *(ptr++) = center_y - h;  // ymin
  *(ptr++) = center_x + w;  // xmax
  *(ptr++) = center_y + h;  // ymax
}
}  // namespace cuda

template<typename DType>
inline void MultiBoxPriorForward(const Tensor<gpu, 2, DType> &out,
                            const std::vector<float> &sizes,
                            const std::vector<float> &ratios,
                            const int in_width, const int in_height,
                            const std::vector<float> &steps,
                            const std::vector<float> &offsets) {
  CHECK_EQ(out.CheckContiguous(), true);
  hipStream_t stream = Stream<gpu>::GetStream(out.stream_);
  DType *out_ptr = out.dptr_;
  const float step_x = steps[1];
  const float step_y = steps[0];
  const float offset_x = offsets[1];
  const float offset_y = offsets[0];
  const int num_sizes = static_cast<int>(sizes.size());
  const int num_ratios = static_cast<int>(ratios.size());

  const int num_thread = cuda::kMaxThreadsPerBlock;
  dim3 dimBlock(num_thread);
  dim3 dimGrid((in_width * in_height - 1) / num_thread + 1);
  cuda::CheckLaunchParam(dimGrid, dimBlock, "MultiBoxPrior Forward");

  const int stride = 4 * (num_sizes + num_ratios - 1);
  int offset = 0;
  // ratio = first ratio, various sizes
  float ratio = num_ratios > 0? sqrtf(ratios[0]) : 1.f;
  for (int i = 0; i < num_sizes; ++i) {
    cuda::AssignPriors<DType><<<dimGrid, dimBlock, 0, stream>>>(out_ptr,
      sizes[i], ratio, in_width, in_height, step_x, step_y, offset_y, offset_x, stride, offset);
    ++offset;
  }
  MULTIBOXPRIOR_CUDA_CHECK(hipGetLastError());

  // size = sizes[0], various ratios
  for (int j = 1; j < num_ratios; ++j) {
    cuda::AssignPriors<DType><<<dimGrid, dimBlock, 0, stream>>>(out_ptr,
      sizes[0], sqrtf(ratios[j]), in_width, in_height, step_x, step_y,
       offset_y, offset_x, stride, offset);
    ++offset;
  }
  MULTIBOXPRIOR_CUDA_CHECK(hipGetLastError());
}
}  // namespace mshadow

namespace mxnet {
namespace op {
template<>
Operator* CreateOp<gpu>(MultiBoxPriorParam param, int dtype) {
  Operator *op = nullptr;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new MultiBoxPriorOp<gpu, DType>(param);
  });
  return op;
}

}  // namespace op
}  // namespace mxnet
