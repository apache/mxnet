
#include <hip/hip_runtime.h>
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file tensorrt.cu
 * \brief TensorRT GPU operation registration
 * \author Marek Kolodziej, Clement Fuji Tsang, Serge Panev
*/

#if MXNET_USE_TENSORRT

#include <string>
#include <unordered_map>

#include "./tensorrt-inl.h"

namespace mxnet {
namespace op {

#define CHECK_CUDART(x) do { \
  cudaError_t res = (x); \
  if (res != cudaSuccess) { \
    fprintf(stderr, "CUDART: %s = %d (%s) at (%s:%d)\n", \
      #x, res, cudaGetErrorString(res), __FILE__, __LINE__); \
    exit(1); \
  } \
} while (0)

void TRTCompute(const OpStatePtr& state, const OpContext& ctx,
                const std::vector<TBlob>& inputs,
                const std::vector<OpReqType>& req,
                const std::vector<TBlob>& outputs) {
  using namespace mshadow;
  using namespace mshadow::expr;
  cudaStream_t cuda_s = Stream<gpu>::GetStream(ctx.get_stream<gpu>());
  auto& param = state.get_state<TRTEngineParam>();
  if (param.calibration_mode) {
    std::unordered_map<std::string, void*> input_ptr_map;
    for (auto it : param.input_name_to_idx) {
      input_ptr_map.emplace(it.first, inputs[it.second].dptr_);
    }
    param.calibrator->setBatch(input_ptr_map, cuda_s);
  }
  for (size_t i = 0; i < param.binding_order->size(); ++i) {
    auto& p = param.binding_order->at(i);
    if (p.second == true) {
      param.bindings->at(i) = inputs[p.first].dptr_;
    } else {
      param.bindings->at(i) = outputs[p.first].dptr_;
    }
  }
  param.trt_executor->enqueueV2(param.bindings->data(), cuda_s, nullptr);

  if (param.calibration_mode && param.calibrator->lastIter()) {
    param.calibrator->waitAndSetDone();
    // calibrator is fully calibrated, the calibration tables are ready
    cudaStreamSynchronize(cuda_s);
    // create the new engine
    auto int8_engine = param.future_int8_engine.get();
    LOG(INFO) << "[TensorRT op] Calibration done, setting inference engine to INT8.";
    param.ResetEngine(std::move(int8_engine),
                      /* calibration_mode=*/ false);
  }
}

NNVM_REGISTER_OP(_TensorRT)
.set_attr<FStatefulCompute>("FStatefulCompute<gpu>", TRTCompute)
.set_attr<nnvm::FGradient>("FGradient", MakeZeroGradNodes);

}  // namespace op
}  // namespace mxnet

#endif  // MXNET_USE_TENSORRT
