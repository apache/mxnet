
#include <hip/hip_runtime.h>
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2018 by Contributors
 * \file tensorrt.cu
 * \brief TensorRT GPU operation registration
 * \author Marek Kolodziej, Clement Fuji Tsang
*/

#if MXNET_USE_TENSORRT

#include "./tensorrt-inl.h"

namespace mxnet {
namespace op {

#define CHECK_CUDART(x) do { \
  cudaError_t res = (x); \
  if (res != cudaSuccess) { \
    fprintf(stderr, "CUDART: %s = %d (%s) at (%s:%d)\n", \
      #x, res, cudaGetErrorString(res), __FILE__, __LINE__); \
    exit(1); \
  } \
} while (0)

void TRTCompute(const OpStatePtr& state, const OpContext& ctx,
                const std::vector<TBlob>& inputs,
                const std::vector<OpReqType>& req,
                const std::vector<TBlob>& outputs) {
  using namespace mshadow;
  using namespace mshadow::expr;
  cudaStream_t cuda_s = Stream<gpu>::GetStream(ctx.get_stream<gpu>());
  const auto& param = state.get_state<TRTEngineParam>();
  for (size_t i = 0; i < param.binding_order->size(); ++i) {
    auto& p = param.binding_order->at(i);
    if (p.second == true) {
      param.bindings->at(i) = inputs[p.first].dptr_;
    } else {
      param.bindings->at(i) = outputs[p.first].dptr_;
    }
  }
  param.trt_executor->enqueueV2(param.bindings->data(), cuda_s, nullptr);
}

NNVM_REGISTER_OP(_TensorRT)
.set_attr<FStatefulCompute>("FStatefulCompute<gpu>", TRTCompute)
.set_attr<nnvm::FGradient>("FGradient", MakeZeroGradNodes);

}  // namespace op
}  // namespace mxnet

#endif  // MXNET_USE_TENSORRT
