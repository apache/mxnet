#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2015 by Contributors
 * \file roi_pooling.cu
 * \brief roi pooling operator
 * \author Ross Girshick, Kye-Hyeon Kim, Jian Guo
*/
#include "./roi_pooling-inl.h"
#include <mshadow/tensor.h>
#include <mshadow/cuda/reduce.cuh>
#include <algorithm>
#include <vector>

namespace mshadow {
namespace cuda {

template<typename Dtype>
__global__ void ROIPoolForwardKernel(const int count, const Dtype* bottom_data,
                                     const float spatial_scale, const int batch_size,
                                     const int channels, const int height, const int width,
                                     const int pooled_height, const int pooled_width,
                                     const Dtype* bottom_rois, Dtype* top_data,
                                     index_t* argmax_data) {
  for (index_t index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
       index < count;
       index += blockDim.x * gridDim.x * gridDim.y) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    bottom_rois += n * 5;
    int roi_batch_ind = static_cast<int>(bottom_rois[0]);

    if (roi_batch_ind < 0 || roi_batch_ind >= batch_size) {
      top_data[index] = 0;
      argmax_data[index] = -1;
      continue;
    }

    int roi_start_w = round(bottom_rois[1] * spatial_scale);
    int roi_start_h = round(bottom_rois[2] * spatial_scale);
    int roi_end_w = round(bottom_rois[3] * spatial_scale);
    int roi_end_h = round(bottom_rois[4] * spatial_scale);

    // Force malformed ROIs to be 1x1
    int roi_width = max(roi_end_w - roi_start_w + 1, 1);
    int roi_height = max(roi_end_h - roi_start_h + 1, 1);
    Dtype bin_size_h = static_cast<Dtype>(roi_height)
                       / static_cast<Dtype>(pooled_height);
    Dtype bin_size_w = static_cast<Dtype>(roi_width)
                       / static_cast<Dtype>(pooled_width);

    int hstart = static_cast<int>(floor(static_cast<Dtype>(ph)
                                        * bin_size_h));
    int wstart = static_cast<int>(floor(static_cast<Dtype>(pw)
                                        * bin_size_w));
    int hend = static_cast<int>(ceil(static_cast<Dtype>(ph + 1)
                                     * bin_size_h));
    int wend = static_cast<int>(ceil(static_cast<Dtype>(pw + 1)
                                     * bin_size_w));

    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart + roi_start_h, 0), height);
    hend = min(max(hend + roi_start_h, 0), height);
    wstart = min(max(wstart + roi_start_w, 0), width);
    wend = min(max(wend + roi_start_w, 0), width);
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    // Define an empty pooling region to be zero
    Dtype maxval = is_empty ? 0 : -FLT_MAX;
    // If nothing is pooled, argmax = -1 causes nothing to be backprop'd
    index_t maxidx = -1;
    index_t offset_bottom_data = (roi_batch_ind * channels + c) * height * width;
    bottom_data += offset_bottom_data;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        index_t bottom_index = h * width + w;
        if (bottom_data[bottom_index] > maxval) {
          maxval = bottom_data[bottom_index];
          maxidx = offset_bottom_data + bottom_index;
        }
      }
    }
    top_data[index] = maxval;
    argmax_data[index] = maxidx;
  }
}

template<typename Dtype>
inline void ROIPoolForward(const Tensor<gpu, 4, Dtype> &out,
                           const Tensor<gpu, 4, Dtype> &data,
                           const Tensor<gpu, 2, Dtype> &bbox,
                           const Tensor<gpu, 4, index_t> &max_idx,
                           const float spatial_scale) {
  const Dtype *bottom_data = data.dptr_;
  const Dtype *bottom_rois = bbox.dptr_;
  Dtype *top_data = out.dptr_;
  index_t *argmax_data = max_idx.dptr_;
  const index_t count = out.shape_.Size();
  const int batch_size = data.size(0);
  const int channels = data.size(1);
  const int height = data.size(2);
  const int width = data.size(3);
  const int pooled_height = out.size(2);
  const int pooled_width = out.size(3);
  const int gridSize = (count + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock;
  dim3 dimGrid(kMaxGridDim, (gridSize + kMaxGridDim - 1) / kMaxGridDim);
  dim3 dimBlock(kMaxThreadsPerBlock);
  CheckLaunchParam(dimGrid, dimBlock, "ROIPooling Forward");
  hipStream_t stream = Stream<gpu>::GetStream(out.stream_);
  ROIPoolForwardKernel<Dtype><<<dimGrid, dimBlock, 0, stream>>>(
      count, bottom_data, spatial_scale, batch_size, channels, height, width,
      pooled_height, pooled_width, bottom_rois, top_data, argmax_data);
  MSHADOW_CUDA_POST_KERNEL_CHECK(ROIPoolForwardKernel);
}

template<typename Dtype>
__global__ void ROIPoolBackwardAccKernel(const int count, const Dtype* top_diff,
                                         const index_t* argmax_data, Dtype* bottom_diff) {
  for (index_t index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
       index < count;
       index += blockDim.x * gridDim.x * gridDim.y) {
    index_t max_idx = argmax_data[index];
    if (max_idx >= 0) {
      atomicAdd(&bottom_diff[max_idx], top_diff[index]);
    }
  }
}

template<typename Dtype>
inline void ROIPoolBackwardAcc(const Tensor<gpu, 4, Dtype> &in_grad,
                               const Tensor<gpu, 4, Dtype> &out_grad,
                               const Tensor<gpu, 2, Dtype> &bbox,
                               const Tensor<gpu, 4, index_t> &max_idx,
                               const float spatial_scale) {
  const Dtype *top_diff = out_grad.dptr_;
  Dtype *bottom_diff = in_grad.dptr_;
  index_t *argmax_data = max_idx.dptr_;
  const index_t count = out_grad.shape_.Size();
  const int gridSize = (count + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock;
  dim3 dimGrid(kMaxGridDim, (gridSize + kMaxGridDim - 1) / kMaxGridDim);
  dim3 dimBlock(kMaxThreadsPerBlock);
  CheckLaunchParam(dimGrid, dimBlock, "ROIPooling Backward");
  hipStream_t stream = Stream<gpu>::GetStream(in_grad.stream_);
  ROIPoolBackwardAccKernel<Dtype><<<dimGrid, dimBlock, 0, stream>>>(
      count, top_diff, argmax_data, bottom_diff);
  MSHADOW_CUDA_POST_KERNEL_CHECK(ROIPoolBackwardAccKernel);
}

}  // namespace cuda

template<typename Dtype>
inline void ROIPoolForward(const Tensor<gpu, 4, Dtype> &out,
                           const Tensor<gpu, 4, Dtype> &data,
                           const Tensor<gpu, 2, Dtype> &bbox,
                           const Tensor<gpu, 4, index_t> &max_idx,
                           const float spatial_scale) {
  cuda::ROIPoolForward(out, data, bbox, max_idx, spatial_scale);
}

template<typename Dtype>
inline void ROIPoolBackwardAcc(const Tensor<gpu, 4, Dtype> &in_grad,
                               const Tensor<gpu, 4, Dtype> &out_grad,
                               const Tensor<gpu, 2, Dtype> &bbox,
                               const Tensor<gpu, 4, index_t> &max_idx,
                               const float spatial_scale) {
  cuda::ROIPoolBackwardAcc(in_grad, out_grad, bbox, max_idx, spatial_scale);
}

}  // namespace mshadow


namespace mxnet {
namespace op {

template<>
Operator* CreateOp<gpu>(ROIPoolingParam param, int dtype) {
  Operator* op = nullptr;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new ROIPoolingOp<gpu, DType>(param);
  });
  return op;
}

}  // namespace op
}  // namespace mxnet
