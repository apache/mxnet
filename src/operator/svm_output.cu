#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2015 by Contributors
 * \file svm_output.cu
 * \brief
 * \author Jonas Amaro
*/

#include "./svm_output-inl.h"
#include <>
#include "mshadow/tensor.h"


namespace mshadow {

template<int n_bits, typename DType>
__global__  void L1_SVMKernel(const DType margin,
                              const DType reg_coef,
                              Tensor<gpu, 2, DType> dst,
                              const Tensor<gpu, 1, DType> label,
                              const Tensor<gpu, 2, DType> src) {
  const index_t nmax = dst.size(1);
  const unsigned n_size = 1 << n_bits;
  const int y = blockIdx.x;
  const int n = threadIdx.x;
  const index_t k = static_cast<int>(label[y]);
  for (index_t n_index = n; n_index < nmax; n_index += n_size) {
    if (n_index == k) {
      dst[y][k] = -DType(margin > src[y][k]) * reg_coef;
    } else {
      dst[y][n_index] = DType(margin > -src[y][n_index]) * reg_coef;
    }
  }
}

template<typename DType>
inline void L1_SVM(const DType & margin,
                   const DType & reg_coef,
                   Tensor<gpu, 2, DType> dst,
                   const Tensor<gpu, 1, DType> & label,
                   const Tensor<gpu, 2, DType> & src) {
  dim3 dimBlock(cuda::kBaseThreadNum);
  dim3 dimGrid(dst.size(0));
  hipStream_t stream = Stream<gpu>::GetStream(dst.stream_);
  L1_SVMKernel<cuda::kBaseThreadBits, DType> <<<dimGrid, dimBlock, 0, stream >>>
    (margin, reg_coef, dst, label, src);
  MSHADOW_CUDA_POST_KERNEL_CHECK(L1_SVMKernel);
}


template<int n_bits, typename DType>
__global__  void L2_SVMKernel(const DType margin,
                              const DType reg_coef,
                              Tensor<gpu, 2, DType> dst,
                              const Tensor<gpu, 1, DType> label,
                              const Tensor<gpu, 2, DType> src) {
  const index_t nmax = dst.size(1);
  const unsigned n_size = 1 << n_bits;
  const int y = blockIdx.x;
  const int n = threadIdx.x;
  const index_t k = static_cast<int>(label[y]);
  for (index_t n_index = n; n_index < nmax; n_index += n_size) {
    if (n_index == k) {
      dst[y][k] = margin > src[y][k] ? 2 * (margin - src[y][k]) : DType(0.0f);
      dst[y][k] *= -reg_coef;
    } else {
      dst[y][n_index] = margin > -src[y][n_index] ? (-2)*(margin + src[y][n_index]) : DType(0.0f);
      dst[y][n_index] *= -reg_coef;
    }
  }
}

template<typename DType>
inline void L2_SVM(const DType & margin,
                   const DType & reg_coef,
                   Tensor<gpu, 2, DType> dst,
                   const Tensor<gpu, 1, DType> & label,
                   const Tensor<gpu, 2, DType> & src) {
  dim3 dimBlock(cuda::kBaseThreadNum);
  dim3 dimGrid(dst.size(0));
  hipStream_t stream = Stream<gpu>::GetStream(dst.stream_);
  L2_SVMKernel<cuda::kBaseThreadBits, DType> <<<dimGrid, dimBlock, 0, stream >>>
    (margin, reg_coef, dst, label, src);
  MSHADOW_CUDA_POST_KERNEL_CHECK(L2_SVMKernel);
}
}  // namespace mshadow

namespace mxnet {
namespace op {
template<>
Operator *CreateOp<gpu>(SVMOutputParam param, int dtype) {
  Operator *op = nullptr;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new SVMOutputOp<gpu, DType>(param);
  })
  return op;
}

}  // namespace op
}  // namespace mxnet

