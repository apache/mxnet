#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2016 by Contributors
 * \file spatial_transformer.cu
 * \brief
 * \author Wei Wu
*/

#include "./spatial_transformer-inl.h"
#include <algorithm>
#if MXNET_USE_CUDNN == 1
#include "./cudnn_spatial_transformer-inl.h"
#endif  // MXNET_USE_CUDNN

namespace mshadow {
template<typename DType>
__device__ bool between(DType value, int lowerBound, int upperBound) {
  return (value >= lowerBound && value <= upperBound);
}

template<typename DType>
__global__ void
/*
 * In order to not generate the code that uses too many
 * registers (resulting in too many resources requested
 * error) we need to tell the compiler that we will be
 * launching this kernel with cuda::kMaxThreadsPerBlock
 * threads per block. Setting __launch_bounds__ ensures
 * that such configuration can always be launched.
 */
__launch_bounds__(cuda::kMaxThreadsPerBlock, 1)
BilinearSamplingForwardKernel(const int i_c, const int i_h,
                              const int i_w, const DType* data,
                              const DType* grid, const int o_n,
                              const int o_c, const int o_h,
                              const int o_w, DType* out) {
  for (int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
       index < o_n * o_c * o_h * o_w;
       index += blockDim.x * gridDim.x * gridDim.y) {
    // (n, c, h, w) is the element in out
    int w = index % o_w;
    int h = (index / o_w) % o_h;
    int c = (index / o_w / o_h) % o_c;
    int n = index / o_w / o_h / o_c;
    index_t out_index = n * o_c * o_h * o_w + c * o_h * o_w + h * o_w + w;
    index_t grid_index = n * o_h * o_w * 2 + h * o_w + w;
    DType y_real = (*(grid + grid_index + o_h * o_w) + 1) * (i_h - 1) / 2;
    DType x_real = (*(grid + grid_index) + 1) * (i_w - 1) / 2;
    int top_left_y = static_cast<int>(floor(y_real));
    int top_left_x = static_cast<int>(floor(x_real));
    DType top_left_y_w = 1.0 - (y_real - top_left_y);
    DType top_left_x_w = 1.0 - (x_real - top_left_x);
    int data_index = n * i_c * i_h * i_w + c * i_h * i_w + top_left_y * i_w + top_left_x;
    DType top_left_v = 0;
    DType top_right_v = 0;
    DType bottom_left_v = 0;
    DType bottom_right_v = 0;
    if (between(top_left_x, 0, i_w-1) && between(top_left_y, 0, i_h-1))
      top_left_v = *(data + data_index);
    if (between(top_left_x + 1, 0, i_w-1) && between(top_left_y, 0, i_h-1))
      top_right_v = *(data + data_index + 1);
    if (between(top_left_x, 0, i_w-1) && between(top_left_y + 1, 0, i_h-1))
      bottom_left_v = *(data + data_index + i_w);
    if (between(top_left_x+1, 0, i_w-1) && between(top_left_y + 1, 0, i_h-1))
      bottom_right_v = *(data + data_index + i_w + 1);
    *(out+out_index) = top_left_v * top_left_y_w * top_left_x_w +
                        top_right_v * top_left_y_w * (1.0 - top_left_x_w) +
                        bottom_left_v * (1.0 - top_left_y_w) * top_left_x_w +
                        bottom_right_v * (1.0 - top_left_y_w) * (1.0 - top_left_x_w);
    }
}

/*
 * In order to not generate the code that uses too many
 * registers (resulting in too many resources requested
 * error) we need to tell the compiler that we will be
 * launching this kernel with cuda::kMaxThreadsPerBlock
 * threads per block. Setting __launch_bounds__ ensures
 * that such configuration can always be launched.
 */
template<typename DType>
__global__ void
__launch_bounds__(cuda::kMaxThreadsPerBlock, 1)
BilinearSamplingBackwardKernel(const int i_c, const int i_h,
                               const int i_w, const DType* grad,
                               const DType* data, const int o_n,
                               const int o_c, const int o_h,
                               const int o_w, DType* g_input,
                               DType* grid_src) {
  for (int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
       index < o_n * o_h * o_w;
       index += blockDim.x * gridDim.x * gridDim.y) {
    // (n, c, h, w) is the element in grad
    int w = index % o_w;
    int h = (index / o_w) % o_h;
    int n = index / o_w / o_h;
    DType top_left_y_gw = 0.0;
    DType top_left_x_gw = 0.0;
    index_t grid_src_index = n * o_h * o_w * 2 + h * o_w + w;
    DType y_real = (*(grid_src + grid_src_index + o_h * o_w) + 1) * (i_h - 1) / 2;
    DType x_real = (*(grid_src + grid_src_index) + 1) * (i_w - 1) / 2;
    int top_left_y = static_cast<int>(floor(y_real));
    int top_left_x = static_cast<int>(floor(x_real));
    DType top_left_y_w = 1.0 - (y_real - top_left_y);
    DType top_left_x_w = 1.0 - (x_real - top_left_x);
    for (index_t c = 0; c < o_c; ++c) {
      index_t grad_index = n * o_c * o_h * o_w + c * o_h * o_w + h * o_w + w;
      int data_index = n * i_c * i_h * i_w + c * i_h * i_w + top_left_y * i_w + top_left_x;
      // calc 4 vertex value in input data
      DType top_left_v = 0;
      DType top_right_v = 0;
      DType bottom_left_v = 0;
      DType bottom_right_v = 0;
      // calc input grad
      if (between(top_left_x, 0, i_w-1) && between(top_left_y, 0, i_h-1)) {
        atomicAdd((g_input + data_index), *(grad + grad_index) * top_left_y_w * top_left_x_w);
        top_left_v = *(data + data_index);
      }
      if (between(top_left_x+1, 0, i_w-1) && between(top_left_y, 0, i_h-1)) {
        atomicAdd((g_input + data_index + 1),
                  *(grad + grad_index) * top_left_y_w * (1.0 - top_left_x_w));
        top_right_v = *(data + data_index + 1);
      }
      if (between(top_left_x, 0, i_w-1) && between(top_left_y+1, 0, i_h-1)) {
        atomicAdd((g_input + data_index + i_w),
                  *(grad + grad_index) * (1.0 - top_left_y_w) * top_left_x_w);
        bottom_left_v = *(data + data_index + i_w);
      }
      if (between(top_left_x+1, 0, i_w-1) && between(top_left_y+1, 0, i_h-1)) {
        atomicAdd((g_input + data_index + i_w + 1),
                  *(grad + grad_index) * (1.0 - top_left_y_w) * (1.0 - top_left_x_w));
        bottom_right_v = *(data + data_index + i_w + 1);
      }
      // calc weight grad of top_left_w, then multiple -1 is the grad of grid_src
      top_left_y_gw -= *(grad + grad_index) * (top_right_v - bottom_right_v +
                       (top_left_v - top_right_v - bottom_left_v + bottom_right_v)
                       * top_left_x_w);
      top_left_x_gw -= *(grad + grad_index) * (bottom_left_v - bottom_right_v +
                       (top_left_v - top_right_v - bottom_left_v + bottom_right_v)
                       * top_left_y_w);
    }
    // calc grid_src grad
    *(grid_src + grid_src_index + o_h * o_w) = top_left_y_gw * (i_h - 1) / 2;
    *(grid_src + grid_src_index) = top_left_x_gw * (i_w - 1) / 2;
  }
}

template<typename DType>
inline void BilinearSamplingForward(const Tensor<gpu, 4, DType> &output,
                                    const Tensor<gpu, 4, DType> &input,
                                    const Tensor<gpu, 3, DType> grid_src) {
    DType *out = output.dptr_;
    const DType *data = input.dptr_;
    const DType *grid = grid_src.dptr_;
    int o_n = output.size(0), o_c = output.size(1), o_h = output.size(2), o_w = output.size(3);
    int i_c = input.size(1), i_h = input.size(2), i_w = input.size(3);
    using namespace cuda;
    const int max_block = (output.shape_.Size() + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock;
    dim3 num_blocks(kMaxGridDim, (max_block + kMaxGridDim - 1) / kMaxGridDim);
    dim3 threads_per_block(kMaxThreadsPerBlock);
    CheckLaunchParam(num_blocks, threads_per_block, "spatial transformer forward");
    hipStream_t stream = Stream<gpu>::GetStream(output.stream_);
    BilinearSamplingForwardKernel<DType> << <num_blocks, threads_per_block, 0, stream >> >(
      i_c, i_h, i_w, data, grid, o_n, o_c, o_h, o_w, out);
    MSHADOW_CUDA_POST_KERNEL_CHECK(BilinearSamplingForwardKernel);
}

template<typename DType>
inline void BilinearSamplingBackward(const Tensor<gpu, 4, DType> &input_grad,
                                     const Tensor<gpu, 3, DType> &grid_src_data,
                                     const Tensor<gpu, 4, DType> &output_grad,
                                     const Tensor<gpu, 4, DType> &input_data) {
  DType *g_input = input_grad.dptr_;
  DType *grid_src = grid_src_data.dptr_;
  const DType *grad = output_grad.dptr_;
  const DType *data = input_data.dptr_;
  int o_n = output_grad.size(0), o_c = output_grad.size(1),
      o_h = output_grad.size(2), o_w = output_grad.size(3);
  int i_c = input_data.size(1), i_h = input_data.size(2), i_w = input_data.size(3);
  using namespace cuda;
  const int max_block = (output_grad.shape_.Size() / o_c + kMaxThreadsPerBlock - 1)
                        / kMaxThreadsPerBlock;
  dim3 num_blocks(kMaxGridDim, (max_block + kMaxGridDim - 1) / kMaxGridDim);
  dim3 threads_per_block(kMaxThreadsPerBlock);
  CheckLaunchParam(num_blocks, threads_per_block, "spatial transformer backward");
  hipStream_t stream = Stream<gpu>::GetStream(input_grad.stream_);
  BilinearSamplingBackwardKernel<DType> << <num_blocks, threads_per_block, 0, stream >> >(
    i_c, i_h, i_w, grad, data, o_n, o_c, o_h, o_w, g_input, grid_src);
  MSHADOW_CUDA_POST_KERNEL_CHECK(BilinearSamplingBackwardKernel);
}

}  // namespace mshadow

namespace mxnet {
namespace op {
template<>
Operator* CreateOp<gpu>(SpatialTransformerParam param, int dtype) {
  Operator *op = nullptr;
#if MXNET_USE_CUDNN == 1
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    if (param.cudnn_off.has_value() && param.cudnn_off.value()) {
      op = new SpatialTransformerOp<gpu, DType>(param);
    } else {
      op = new CuDNNSpatialTransformerOp<DType>(param);
    }
  })
#else
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new SpatialTransformerOp<gpu, DType>(param);
  })
#endif  // MXNET_USE_CUDNN
  return op;
}

}  // namespace op
}  // namespace mxnet
