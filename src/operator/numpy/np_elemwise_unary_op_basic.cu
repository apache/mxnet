#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file np_elemwise_unary_op_basic.cu
 * \brief GPU Implementation of numpy unary functions.
 */
#include "../tensor/elemwise_unary_op.h"
#include "../tensor/elemwise_binary_op.h"

namespace mxnet {
namespace op {

#define MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(__name$, __kernel$)       \
  NNVM_REGISTER_OP(__name$)                                               \
  .set_attr<FCompute>("FCompute<gpu>", UnaryRTCCompute{#__kernel$})

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npx_relu, relu);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npx_sigmoid, sigmoid);

NNVM_REGISTER_OP(_npi_copy)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::IdentityCompute<gpu>);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_negative, negation);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_reciprocal, reciprocal);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_absolute, abs);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_sign, sign);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_rint, rint);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_ceil, ceil);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_floor, floor);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_bitwise_not, bitwise_not);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_trunc, trunc);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_fix, fix);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_square, square);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_sqrt, sqrt);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_cbrt, cbrt);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_exp, exp);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_log, log);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_log10, log10);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_log2, log2);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_log1p, log1p);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_expm1, expm1);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_logical_not, np_logical_not);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_isnan, isnan);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_isinf, isinf);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_isposinf, isposinf);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_isneginf, isneginf);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_isfinite, isfinite);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_sin, sin);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_cos, cos);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_tan, tan);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_arcsin, arcsin);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_arccos, arccos);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_arctan, arctan);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_degrees, degrees);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_radians, radians);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_sinh, sinh);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_cosh, cosh);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_tanh, tanh);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_arcsinh, arcsinh);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_arccosh, arccosh);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_arctanh, arctanh);

NNVM_REGISTER_OP(_npi_around)
.set_attr<FCompute>("FCompute<gpu>", AroundOpForward<gpu>);

NNVM_REGISTER_OP(_npi_nan_to_num)
.set_attr<FCompute>("FCompute<gpu>", NumpyNanToNumOpForward<gpu>);

NNVM_REGISTER_OP(_npi_backward_nan_to_num)
.set_attr<FCompute>("FCompute<gpu>", NumpyNanToNumOpBackward<gpu>);

NNVM_REGISTER_OP(_backward_npi_exp)
.set_attr<FCompute>("FCompute<gpu>", UnaryBwdInOutRTCCompute{"mul"});

NNVM_REGISTER_OP(_backward_npi_log)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_log"});

NNVM_REGISTER_OP(_backward_npi_log10)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_log10"});

NNVM_REGISTER_OP(_backward_npi_log2)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_log2"});

NNVM_REGISTER_OP(_backward_npi_log1p)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_log1p"});

NNVM_REGISTER_OP(_backward_npi_expm1)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_expm1"});

NNVM_REGISTER_OP(_backward_npi_sqrt)
.set_attr<FCompute>("FCompute<gpu>", UnaryBwdInOutRTCCompute{"backward_sqrt"});

NNVM_REGISTER_OP(_backward_npi_cbrt)
.set_attr<FCompute>("FCompute<gpu>", UnaryBwdInOutRTCCompute{"backward_cbrt"});

NNVM_REGISTER_OP(_backward_npi_sin)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_sin"});

NNVM_REGISTER_OP(_backward_npi_cos)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_cos"});

NNVM_REGISTER_OP(_backward_npi_tan)
.set_attr<FCompute>("FCompute<gpu>", UnaryBwdInOutRTCCompute{"backward_tan"});

NNVM_REGISTER_OP(_backward_npi_arcsin)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_arcsin"});

NNVM_REGISTER_OP(_backward_npi_arccos)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_arccos"});

NNVM_REGISTER_OP(_backward_npi_arctan)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_arctan"});

NNVM_REGISTER_OP(_backward_npi_degrees)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_degrees"});

NNVM_REGISTER_OP(_backward_npi_radians)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_radians"});

NNVM_REGISTER_OP(_backward_npi_cosh)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_cosh"});

NNVM_REGISTER_OP(_backward_npi_sinh)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_sinh"});

NNVM_REGISTER_OP(_backward_npi_tanh)
.set_attr<FCompute>("FCompute<gpu>", UnaryBwdInOutRTCCompute{"backward_tanh"});

NNVM_REGISTER_OP(_backward_npi_arcsinh)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_arcsinh"});

NNVM_REGISTER_OP(_backward_npi_arccosh)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_arccosh"});

NNVM_REGISTER_OP(_backward_npi_arctanh)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryRTCCompute{"backward_arctanh"});

}  // namespace op
}  // namespace mxnet
