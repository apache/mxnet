/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file np_boolean_mask_assign.cu
 * \brief GPU implementation of Boolean Mask Assign
 */

#include <hipcub/hipcub.hpp>
#include "../../common/utils.h"
#include "../contrib/boolean_mask-inl.h"

namespace mxnet {
namespace op {

template<bool scalar>
struct BooleanAssignGPUKernel {
 private:
  static size_t __device__ bin_search(const size_t* idx,
                                      const size_t idx_size,
                                      const size_t i) {
    size_t left = 0, right = idx_size, mid = (left + right) / 2;
    while (left != right) {
      if (idx[mid] == i + 1) {
        if (idx[mid - 1] == i) {
          mid -= 1;
          break;
        } else if (idx[mid - 1] == i + 1) {
          right = mid;
          mid = (left + right) / 2;
        }
      } else if (idx[mid] == i) {
        if (idx[mid + 1] == i + 1) {
          break;
        } else {
          left = mid;
          mid = (left + right + 1) / 2;
        }
      } else if (idx[mid] < i + 1) {
        left = mid;
        mid = (left + right + 1) / 2;
      } else if (idx[mid] > i + 1) {
        right = mid;
        mid = (left + right) / 2;
      }
    }
    return mid;
  }

 public:
  template<typename DType>
  static void __device__ Map(int i,
                             DType* data,
                             const size_t* idx,
                             const size_t idx_size,
                             const size_t leading,
                             const size_t middle,
                             const size_t valid_num,
                             const size_t trailing,
                             const DType val) {
    // binary search for the turning point
    size_t m = i / trailing % valid_num;
    size_t l = i / trailing / valid_num;
    size_t mid = bin_search(idx, idx_size, m);
    // final answer is in mid
    // i = l * valid_num * trailing + m * trailing + t
    // dst = l * middle * trailing + mid * trailing + t
    data[i + (l * (middle - valid_num) + (mid - m)) * trailing] = val;
  }

  template<typename DType>
  static void __device__ Map(int i,
                             DType* data,
                             const size_t* idx,
                             const size_t idx_size,
                             const size_t leading,
                             const size_t middle,
                             const size_t valid_num,
                             const size_t trailing,
                             DType* tensor,
                             const bool broadcast = false) {
    // binary search for the turning point
    size_t m = i / trailing % valid_num;
    size_t l = i / trailing / valid_num;
    size_t mid = bin_search(idx, idx_size, m);
    size_t dst = i + (l * (middle - valid_num) + (mid - m)) * trailing;
    // final answer is in mid
    if (scalar) {
      data[dst] = tensor[0];
    } else {
      data[dst] = broadcast ? tensor[l * trailing + i % trailing] : tensor[i];
    }
  }
};

struct NonZeroWithCast {
  template<typename OType, typename IType>
  static void __device__ Map(int i, OType* out, const IType* in) {
    out[i] = (in[i]) ? OType(1) : OType(0);
  }
};

// completing the prefix_sum vector and return the pointer to it
template<typename DType>
size_t* GetValidNumGPU(const OpContext &ctx, const DType *idx, const size_t idx_size) {
  using namespace mshadow;
  using namespace mxnet_op;
  using namespace mshadow_op;
  size_t* prefix_sum = nullptr;
  void* d_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  Stream<gpu>* s = ctx.get_stream<gpu>();
  hipStream_t stream = Stream<gpu>::GetStream(s);

  // Calculate total temporary memory size
  hipcub::DeviceScan::ExclusiveSum(d_temp_storage,
                                temp_storage_bytes,
                                prefix_sum,
                                prefix_sum,
                                idx_size + 1,
                                stream);
  size_t buffer_size = (idx_size + 1) * sizeof(size_t);
  temp_storage_bytes += buffer_size;
  // Allocate memory on GPU and allocate pointer
  Tensor<gpu, 1, char> workspace =
    ctx.requested[0].get_space_typed<gpu, 1, char>(Shape1(temp_storage_bytes), s);
  prefix_sum = reinterpret_cast<size_t*>(workspace.dptr_);
  d_temp_storage = workspace.dptr_ + buffer_size;

  // Robustly set the bool values in mask
  // TODO(haojin2): Get a more efficient way to preset the buffer
  Kernel<set_zero, gpu>::Launch(s, idx_size + 1, prefix_sum);
  if (!std::is_same<DType, bool>::value) {
    Kernel<NonZeroWithCast, gpu>::Launch(s, idx_size, prefix_sum, idx);
  } else {
    Kernel<identity_with_cast, gpu>::Launch(s, idx_size, prefix_sum, idx);
  }

  // Calculate prefix sum
  hipcub::DeviceScan::ExclusiveSum(d_temp_storage,
                                temp_storage_bytes,
                                prefix_sum,
                                prefix_sum,
                                idx_size + 1,
                                stream);

  return prefix_sum;
}

void NumpyBooleanAssignForwardGPU(const nnvm::NodeAttrs& attrs,
                                  const OpContext &ctx,
                                  const std::vector<TBlob> &inputs,
                                  const std::vector<OpReqType> &req,
                                  const std::vector<TBlob> &outputs) {
  using namespace mshadow;
  using namespace mxnet_op;
  CHECK(inputs.size() == 2U || inputs.size() == 3U);
  CHECK_EQ(outputs.size(), 1U);
  CHECK_EQ(req.size(), 1U);
  CHECK_EQ(req[0], kWriteInplace)
    << "Only WriteInplace is supported for npi_boolean_assign";

  Stream<gpu>* s = ctx.get_stream<gpu>();

  const TBlob& data = inputs[0];
  const TShape& dshape = data.shape_;
  const TBlob& mask = inputs[1];
  const TShape& mshape = mask.shape_;
  const int start_axis = std::stoi(common::attr_value_string(attrs, "start_axis", "0"));

  // Get valid_num
  size_t mask_size = mask.shape_.Size();
  size_t valid_num = 0;
  size_t* prefix_sum = nullptr;
  if (mask_size != 0) {
    MSHADOW_TYPE_SWITCH_WITH_BOOL(mask.type_flag_, MType, {
      prefix_sum = GetValidNumGPU<MType>(ctx, mask.dptr<MType>(), mask_size);
    });
    hipStream_t stream = mshadow::Stream<gpu>::GetStream(s);
    CUDA_CALL(hipMemcpyAsync(&valid_num, &prefix_sum[mask_size], sizeof(size_t),
                              hipMemcpyDeviceToHost, stream));
    CUDA_CALL(hipStreamSynchronize(stream));
  }

  // If there's no True in mask, return directly
  if (valid_num == 0) return;

  const TShape& vshape = inputs[2].shape_;

  if (inputs.size() == 3U) {
    // tensor case
    if (inputs[2].shape_.Size() != 1) {
      auto vndim = vshape.ndim();
      auto dndim = dshape.ndim();
      auto mndim = mshape.ndim();
      CHECK(vndim <= (dndim - mndim + 1));
      if ((vndim == (dndim - mndim + 1)) && (vshape[start_axis] != 1)) {
        // tensor case, check tensor size equal to or broadcastable with valid_num
        CHECK_EQ(static_cast<size_t>(valid_num), vshape[start_axis])
          << "boolean array indexing assignment cannot assign " << vshape
          << " input values to the " << valid_num << " output values where the mask is true"
          << std::endl;
      }
    }
  }

  size_t leading = 1U;
  size_t middle = mask_size;
  size_t trailing = 1U;

  for (int i = 0; i < dshape.ndim(); ++i) {
    if (i < start_axis) {
      leading *= dshape[i];
    }
    if (i >= start_axis + mshape.ndim()) {
      trailing *= dshape[i];
    }
  }

  if (inputs.size() == 3U) {
    if (inputs[2].shape_.Size() == 1) {
      MSHADOW_TYPE_SWITCH_WITH_BOOL(data.type_flag_, DType, {
        Kernel<BooleanAssignGPUKernel<true>, gpu>::Launch(
          s, leading * valid_num * trailing, data.dptr<DType>(), prefix_sum, mask_size + 1,
          leading, middle, valid_num, trailing, inputs[2].dptr<DType>());
      });
    } else {
      bool need_broadcast = (vshape.ndim() == (dshape.ndim() - mshape.ndim() + 1)) ?
                            (vshape[start_axis] == 1) :
                            true;
      MSHADOW_TYPE_SWITCH_WITH_BOOL(data.type_flag_, DType, {
        Kernel<BooleanAssignGPUKernel<false>, gpu>::Launch(
          s, leading * valid_num * trailing, data.dptr<DType>(), prefix_sum, mask_size + 1,
          leading, middle, valid_num, trailing, inputs[2].dptr<DType>(), need_broadcast);
      });
    }
  } else {
    CHECK(attrs.dict.find("value") != attrs.dict.end()) << "value is not provided";
    double value = std::stod(attrs.dict.at("value"));
    MSHADOW_TYPE_SWITCH_WITH_BOOL(data.type_flag_, DType, {
      Kernel<BooleanAssignGPUKernel<true>, gpu>::Launch(
        s, leading * valid_num * trailing, data.dptr<DType>(), prefix_sum, mask_size + 1,
        leading, middle, valid_num, trailing, static_cast<DType>(value));
    });
  }
}

NNVM_REGISTER_OP(_npi_boolean_mask_assign_scalar)
.set_attr<FCompute>("FCompute<gpu>", NumpyBooleanAssignForwardGPU);

NNVM_REGISTER_OP(_npi_boolean_mask_assign_tensor)
.set_attr<FCompute>("FCompute<gpu>", NumpyBooleanAssignForwardGPU);

}  // namespace op
}  // namespace mxnet
