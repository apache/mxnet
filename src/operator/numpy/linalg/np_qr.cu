/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file np_qr.cu
 * \brief GPU implementation of the QR Operator
 */

#include <mxnet/operator_util.h>
#include <vector>
#include "./np_qr-inl.h"

namespace mxnet {
namespace op {

#if MXNET_USE_CUSOLVER == 1

NNVM_REGISTER_OP(_npi_qr)
.set_attr<FCompute>("FCompute<gpu>", NumpyLaQrForward<gpu>);

NNVM_REGISTER_OP(_backward_npi_qr)
.set_attr<FCompute>("FCompute<gpu>", NumpyLaQrBackward<gpu>);

#endif

}  // namespace op
}  // namespace mxnet
