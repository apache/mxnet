#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file np_elemwise_broadcast_op_extended.cu
 * \brief GPU Implementation of extended functions for elementwise binary broadcast operator.
 */

#include "./np_elemwise_broadcast_op.h"

namespace mxnet {
namespace op {

NNVM_REGISTER_OP(_npi_copysign)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastCompute<gpu, mshadow_op::copysign>);

NNVM_REGISTER_OP(_npi_lcm)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastIntCompute<gpu, mshadow_op::lcm>);

NNVM_REGISTER_OP(_npi_bitwise_and)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastIntCompute<gpu, mshadow_op::bitwise_and>);

NNVM_REGISTER_OP(_npi_bitwise_xor)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastIntCompute<gpu, mshadow_op::bitwise_xor>);

NNVM_REGISTER_OP(_npi_bitwise_or)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastIntCompute<gpu, mshadow_op::bitwise_or>);

NNVM_REGISTER_OP(_backward_npi_copysign)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastBackwardUseIn<gpu, mshadow_op::copysign_grad,
                                                                  mshadow_op::copysign_rgrad>);

NNVM_REGISTER_OP(_npi_arctan2)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastCompute<gpu, mshadow_op::arctan2>);

NNVM_REGISTER_OP(_backward_npi_arctan2)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastBackwardUseIn<gpu, mshadow_op::arctan2_grad,
                                                                  mshadow_op::arctan2_rgrad>);
NNVM_REGISTER_OP(_npi_hypot)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastCompute<gpu, mshadow_op::hypot>);

NNVM_REGISTER_OP(_backward_npi_hypot)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastBackwardUseIn<gpu, mshadow_op::hypot_grad_left,
                                                                  mshadow_op::hypot_grad_right>);
NNVM_REGISTER_OP(_npi_copysign_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::copysign>);

NNVM_REGISTER_OP(_npi_rcopysign_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::rcopysign>);

NNVM_REGISTER_OP(_backward_npi_copysign_scalar)
.set_attr<FCompute>("FCompute<gpu>",
                    BinaryScalarOp::Backward<gpu, mshadow_op::copysign_grad>);

NNVM_REGISTER_OP(_backward_npi_rcopysign_scalar)
.set_attr<FCompute>("FCompute<gpu>",
                    BinaryScalarOp::Backward<gpu, mshadow_op::rcopysign_grad>);

NNVM_REGISTER_OP(_npi_arctan2_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::arctan2>);

NNVM_REGISTER_OP(_backward_npi_arctan2_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::arctan2_grad>);

NNVM_REGISTER_OP(_npi_rarctan2_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::rarctan2>);

NNVM_REGISTER_OP(_backward_npi_rarctan2_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::rarctan2_grad>);

NNVM_REGISTER_OP(_npi_lcm_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::ComputeInt<gpu, mshadow_op::lcm>);

NNVM_REGISTER_OP(_npi_bitwise_and_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::ComputeInt<gpu, mshadow_op::bitwise_and>);

NNVM_REGISTER_OP(_npi_bitwise_xor_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::ComputeInt<gpu, mshadow_op::bitwise_xor>);

NNVM_REGISTER_OP(_npi_bitwise_or_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::ComputeInt<gpu, mshadow_op::bitwise_or>);

NNVM_REGISTER_OP(_npi_ldexp)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastCompute<gpu, mshadow_op::ldexp>);

NNVM_REGISTER_OP(_npi_ldexp_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::ldexp>);

NNVM_REGISTER_OP(_npi_rldexp_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::rldexp>);

NNVM_REGISTER_OP(_backward_npi_ldexp)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastBackwardUseIn<gpu, mshadow_op::ldexp_grad,
                                                                  mshadow_op::ldexp_rgrad>);

NNVM_REGISTER_OP(_backward_npi_ldexp_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Backward<gpu, mshadow_op::ldexp_grad>);

NNVM_REGISTER_OP(_backward_npi_rldexp_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Backward<gpu, mshadow_op::rldexp_grad>);

}  // namespace op
}  // namespace mxnet
