#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 *  Copyright (c) 2019 by Contributors
 * \file np_elemwise_broadcast_op_extended.cu
 * \brief GPU Implementation of extended functions for elementwise binary broadcast operator.
 */

#include "./np_elemwise_broadcast_op.h"

namespace mxnet {
namespace op {

NNVM_REGISTER_OP(_npi_copysign)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastRTCCompute{"copysign"});

NNVM_REGISTER_OP(_npi_gcd)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastRTCCompute{"gcd"});

NNVM_REGISTER_OP(_npi_lcm)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastRTCCompute{"lcm"});

NNVM_REGISTER_OP(_npi_bitwise_and)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastRTCCompute{"bitwise_and"});

NNVM_REGISTER_OP(_npi_bitwise_xor)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastRTCCompute{"bitwise_xor"});

NNVM_REGISTER_OP(_npi_bitwise_or)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastRTCCompute{"bitwise_or"});

NNVM_REGISTER_OP(_backward_npi_copysign)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastRTCBackwardUseIn{"copysign_grad",
                                                                     "zero"});

NNVM_REGISTER_OP(_npi_arctan2)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastRTCCompute{"arctan2"});

NNVM_REGISTER_OP(_backward_npi_arctan2)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastRTCBackwardUseIn{"arctan2_grad",
                                                                     "arctan2_rgrad"});

NNVM_REGISTER_OP(_npi_hypot)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastRTCCompute{"hypot"});

NNVM_REGISTER_OP(_backward_npi_hypot)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastRTCBackwardUseIn{"hypot_grad_left",
                                                                     "hypot_grad_right"});
NNVM_REGISTER_OP(_npi_copysign_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarRTCCompute{"copysign"});

NNVM_REGISTER_OP(_npi_rcopysign_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarRTCCompute{"rcopysign"});

NNVM_REGISTER_OP(_backward_npi_copysign_scalar)
.set_attr<FCompute>("FCompute<gpu>",
                    BinaryScalarRTCBackward{"copysign_grad"});

NNVM_REGISTER_OP(_npi_arctan2_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarRTCCompute{"arctan2"});

NNVM_REGISTER_OP(_backward_npi_arctan2_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarRTCBackward{"arctan2_grad"});

NNVM_REGISTER_OP(_npi_rarctan2_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarRTCCompute{"rarctan2"});

NNVM_REGISTER_OP(_backward_npi_rarctan2_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarRTCBackward{"rarctan2_grad"});

NNVM_REGISTER_OP(_npi_gcd_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarRTCCompute{"gcd"});

NNVM_REGISTER_OP(_npi_lcm_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarRTCCompute{"lcm"});

NNVM_REGISTER_OP(_npi_bitwise_and_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarRTCCompute{"bitwise_and"});

NNVM_REGISTER_OP(_npi_bitwise_xor_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarRTCCompute{"bitwise_xor"});

NNVM_REGISTER_OP(_npi_bitwise_or_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarRTCCompute{"bitwise_or"});

NNVM_REGISTER_OP(_npi_ldexp)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastRTCCompute{"ldexp"});

NNVM_REGISTER_OP(_npi_ldexp_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarRTCCompute{"ldexp"});

NNVM_REGISTER_OP(_npi_rldexp_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarRTCCompute{"rldexp"});

NNVM_REGISTER_OP(_backward_npi_ldexp)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastRTCBackwardUseIn{"ldexp_grad",
                                                                     "ldexp_rgrad"});

NNVM_REGISTER_OP(_backward_npi_ldexp_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarRTCBackward{"ldexp_grad"});

NNVM_REGISTER_OP(_backward_npi_rldexp_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarRTCBackward{"rldexp_grad"});

}  // namespace op
}  // namespace mxnet
