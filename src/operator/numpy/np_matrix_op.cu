/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 *  Copyright (c) 2019 by Contributors
 * \file np_matrix_op.cu
 * \brief GPU Implementation of numpy matrix operations
 */

#include "./np_matrix_op-inl.h"
#include "../nn/concat-inl.h"

namespace mxnet {
namespace op {

NNVM_REGISTER_OP(_npi_transpose)
.set_attr<FCompute>("FCompute<gpu>", NumpyTranspose<gpu>);

NNVM_REGISTER_OP(_np_reshape)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::IdentityCompute<gpu>);

NNVM_REGISTER_OP(_npi_squeeze)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::IdentityCompute<gpu>);

NNVM_REGISTER_OP(_npi_concatenate)
.set_attr<FCompute>("FCompute<gpu>", NumpyConcatenateForward<gpu>);

NNVM_REGISTER_OP(_backward_np_concat)
.set_attr<FCompute>("FCompute<gpu>", NumpyConcatenateBackward<gpu>);

NNVM_REGISTER_OP(_npi_stack)
.set_attr<FCompute>("FCompute<gpu>", StackOpForward<gpu>);

NNVM_REGISTER_OP(_npi_vstack)
.set_attr<FCompute>("FCompute<gpu>", NumpyVstackForward<gpu>);

NNVM_REGISTER_OP(_backward_np_vstack)
.set_attr<FCompute>("FCompute<gpu>", NumpyVstackBackward<gpu>);

NNVM_REGISTER_OP(_npi_hstack)
.set_attr<FCompute>("FCompute<gpu>", HStackCompute<gpu>);

NNVM_REGISTER_OP(_backward_np_hstack)
.set_attr<FCompute>("FCompute<gpu>", HStackGradCompute<gpu>);

NNVM_REGISTER_OP(_npi_dstack)
.set_attr<FCompute>("FCompute<gpu>", DStackCompute<gpu>);

NNVM_REGISTER_OP(_backward_np_dstack)
.set_attr<FCompute>("FCompute<gpu>", DStackGradCompute<gpu>);

NNVM_REGISTER_OP(_npi_column_stack)
.set_attr<FCompute>("FCompute<gpu>", NumpyColumnStackForward<gpu>);

NNVM_REGISTER_OP(_backward_np_column_stack)
.set_attr<FCompute>("FCompute<gpu>", NumpyColumnStackBackward<gpu>);

NNVM_REGISTER_OP(_npi_tril_indices)
.set_attr<FCompute>("FCompute<gpu>", TrilindicesOpForward<gpu>);

NNVM_REGISTER_OP(_npi_roll)
.set_attr<FCompute>("FCompute<gpu>", NumpyRollCompute<gpu>);

template<>
void NumpyFlipForwardImpl<gpu>(const OpContext& ctx,
                               const std::vector<TBlob>& inputs,
                               const std::vector<TBlob>& outputs,
                               const std::vector<index_t>& stride_,
                               const std::vector<index_t>& trailing_,
                               const index_t& flip_index) {
  mshadow::Stream<gpu> *s = ctx.get_stream<gpu>();
  mshadow::Tensor<gpu, 1, uint8_t> workspace =
    ctx.requested[0].get_space_typed<gpu, 1, uint8_t>(
      mshadow::Shape1(flip_index * sizeof(index_t) * 2), s);

  auto stride_workspace = workspace.dptr_;
  auto trailing_workspace = workspace.dptr_ + flip_index * sizeof(index_t);

  hipMemcpyAsync(stride_workspace, thrust::raw_pointer_cast(stride_.data()),
                  stride_.size() * sizeof(index_t),
                  hipMemcpyHostToDevice, mshadow::Stream<gpu>::GetStream(s));
  hipMemcpyAsync(trailing_workspace, thrust::raw_pointer_cast(trailing_.data()),
                  trailing_.size() * sizeof(index_t),
                  hipMemcpyHostToDevice, mshadow::Stream<gpu>::GetStream(s));

  MSHADOW_TYPE_SWITCH(outputs[0].type_flag_, DType, {
    mxnet_op::Kernel<reverse, gpu>::Launch(s, inputs[0].Size(), flip_index,
      inputs[0].dptr<DType>(), outputs[0].dptr<DType>(),
      reinterpret_cast<index_t*>(stride_workspace), reinterpret_cast<index_t*>(trailing_workspace));
  });
}

NNVM_REGISTER_OP(_npi_flip)
.set_attr<FCompute>("FCompute<gpu>", NumpyFlipForward<gpu>);

NNVM_REGISTER_OP(_backward_npi_flip)
.set_attr<FCompute>("FCompute<gpu>", NumpyFlipForward<gpu>);

NNVM_REGISTER_OP(_npi_moveaxis)
.set_attr<FCompute>("FCompute<gpu>", NumpyMoveaxisCompute<gpu>);

NNVM_REGISTER_OP(_npi_rollaxis)
.set_attr<FCompute>("FCompute<gpu>", NumpyRollaxisCompute<gpu>);

NNVM_REGISTER_OP(_npi_rollaxis_backward)
.set_attr<FCompute>("FCompute<gpu>", NumpyRollaxisBackward<gpu>);

NNVM_REGISTER_OP(_npi_rot90)
.set_attr<FCompute>("FCompute<gpu>", NumpyRot90Compute<gpu>);

NNVM_REGISTER_OP(_npi_hsplit)
.set_attr<FCompute>("FCompute<gpu>", HSplitOpForward<gpu>);

NNVM_REGISTER_OP(_npi_hsplit_backward)
.set_attr<FCompute>("FCompute<gpu>", HSplitOpBackward<gpu>);

NNVM_REGISTER_OP(_npi_dsplit)
.set_attr<FCompute>("FCompute<gpu>", SplitOpForward<gpu>);

NNVM_REGISTER_OP(_npx_reshape)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::IdentityCompute<gpu>);

NNVM_REGISTER_OP(_npi_diag)
.set_attr<FCompute>("FCompute<gpu>", NumpyDiagOpForward<gpu>);

NNVM_REGISTER_OP(_backward_npi_diag)
.set_attr<FCompute>("FCompute<gpu>", NumpyDiagOpBackward<gpu>);

NNVM_REGISTER_OP(_npi_diagonal)
.set_attr<FCompute>("FCompute<gpu>", NumpyDiagonalOpForward<gpu>);

NNVM_REGISTER_OP(_backward_npi_diagonal)
.set_attr<FCompute>("FCompute<gpu>", NumpyDiagonalOpBackward<gpu>);

NNVM_REGISTER_OP(_npi_diagflat)
.set_attr<FCompute>("FCompute<gpu>", NumpyDiagflatOpForward<gpu>);

NNVM_REGISTER_OP(_backward_npi_diagflat)
.set_attr<FCompute>("FCompute<gpu>", NumpyDiagflatOpBackward<gpu>);

NNVM_REGISTER_OP(_npi_diag_indices_from)
.set_attr<FCompute>("FCompute<gpu>", NumpyDiagIndicesFromForward<gpu>);

}  // namespace op
}  // namespace mxnet
