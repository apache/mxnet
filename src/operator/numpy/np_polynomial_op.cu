#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2020 by Contributors
 * \file np_polynomial_op.cu
 */

#include "np_polynomial_op-inl.h"
#include "../../common/cuda/utils.h"

namespace mxnet {
namespace op {

template<int req>
struct polyval_backward_gpu {
  template<typename DType>
  MSHADOW_XINLINE static void Map(int i, const DType* p_dptr, const DType* x_dptr,
                                  DType* igrad_x_dptr, DType* igrad_p_dptr,
                                  const DType* ograd_dptr, const index_t p_size) {
  DType igrad_p = 1;
  DType igrad_x = 0;
  index_t j = p_size - 1;
  while (j > 0) {
      // atomic add since different threads could update same variable
      atomicAdd(&igrad_p_dptr[j], igrad_p * ograd_dptr[i]);
      igrad_p *= x_dptr[i];
      igrad_x = igrad_x * x_dptr[i] + p_dptr[p_size - j - 1] * j;
      j--;
  }
  atomicAdd(&igrad_p_dptr[j], igrad_p * ograd_dptr[i]);
  KERNEL_ASSIGN(igrad_x_dptr[i], req, igrad_x * ograd_dptr[i]);
  }
};

void NumpyPolyvalBackwardGPU(const nnvm::NodeAttrs& attrs,
                             const OpContext& ctx,
                             const std::vector<TBlob>& inputs,
                             const std::vector<OpReqType>& req,
                             const std::vector<TBlob>& outputs) {
  CHECK_EQ(inputs.size(), 3U);
  CHECK_EQ(outputs.size(), 2U);
  CHECK_NE(req[0], kWriteInplace);

  if (inputs[1].type_flag_ != inputs[2].type_flag_ ||
    !common::is_float(inputs[1].type_flag_) ||
    !common::is_float(inputs[2].type_flag_)) {
      return;
  }

  mshadow::Stream<gpu> *s = ctx.get_stream<gpu>();
  const TBlob& ograd = inputs[0];
  const TBlob& p = inputs[1];
  const TBlob& x = inputs[2];
  const TBlob& igrad_p = outputs[0];
  const TBlob& igrad_x = outputs[1];
  const size_t p_size = p.Size();

  using namespace mxnet_op;
  MSHADOW_REAL_TYPE_SWITCH(ograd.type_flag_, DType, {
    MXNET_ASSIGN_REQ_SWITCH(req[0], req_type, {
      Kernel<polyval_backward_gpu<req_type>, gpu>::Launch(
        s, ograd.Size(), p.dptr<DType>(), x.dptr<DType>(),
        igrad_x.dptr<DType>(), igrad_p.dptr<DType>(),
        ograd.dptr<DType>(), p_size);
    });
  });
}

NNVM_REGISTER_OP(_npi_polyval)
.set_attr<mxnet::FCompute>("FCompute<gpu>", NumpyPolyvalForward<gpu>);

NNVM_REGISTER_OP(_npi_backward_polyval)
.set_attr<mxnet::FCompute>("FCompute<gpu>", NumpyPolyvalBackwardGPU);

}  // namespace op
}  // namespace mxnet
